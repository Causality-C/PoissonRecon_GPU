#include "hip/hip_runtime.h"
#include <cstdio>
#include <bitset>
#include <cstdlib>
#include <unordered_map>
#include "Geometry.cuh"
#include "OctNode.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "PointStream.cuh"
#include "CmdLineParser.cuh"
#include "Debug.cuh"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"
#include "thrust/async/reduce.h"
#include "thrust/scan.h"
#include "thrust/sort.h"
#include "thrust/copy.h"
#include "Hash.cuh"
#include "PPolynomial.cuh"
#include "FunctionData.cuh"
#include "BinaryNode.cuh"
#include "ConfirmedPPolynomial.cuh"
#include "ConfirmedSparseMatrix.cuh"
#include "CG_CUDA.cuh"
#include "MarchingCubes.cuh"
#include "iomanip"

#define FORCE_UNIT_NORMALS 1


__device__ __host__ int qpow(int p,int q) {
    int res=1;
    while(q) {
        if(q&1) res *= p;
        q>>=1;
        p*=p;
    }
    return res;
}

//#define FORCE_UNIT_NORMALS 1
__global__ void outputDeviceArray(Point3D<float> *d_addr,int size) {
    printf("print array:\n");
    for(int i=0;i<size;++i) {
        printf("%f %f %f\n",d_addr[i].coords[0],d_addr[i].coords[1],d_addr[i].coords[2]);
    }
}

__global__ void outputDeviceArray(int *d_addr,int size) {
    printf("print array:\n");
    for(int i=0;i<size;++i) {
        printf("%d\n",d_addr[i]);
    }
}

__global__ void outputDeviceArray(float *d_addr,int size) {
    printf("print array:\n");
    for(int i=0;i<size;++i) {
        printf("%f\n",d_addr[i]);
    }
}

//__constant__ double EPSILON=float(1e-6);
#define EPSILON float(1e-6)
//__constant__ float ROUND_EPS=float(1e-5);
#define ROUND_EPS float(1e-5)
//__constant__ int maxDepth=9;
#define maxDepth 9
//__constant__ int markOffset=31;
#define markOffset 31
//__constant__ int resolution=1023;
#define resolution 1023

#define stackCapacity 2000

#define convTimes 2

__constant__ int LUTparent[8][27]={
        {0,1,1,3,4,4,3,4,4,9,10,10,12,13,13,12,13,13,9,10,10,12,13,13,12,13,13},
        {1,1,2,4,4,5,4,4,5,10,10,11,13,13,14,13,13,14,10,10,11,13,13,14,13,13,14},
        {3,4,4,3,4,4,6,7,7,12,13,13,12,13,13,15,16,16,12,13,13,12,13,13,15,16,16},
        {4,4,5,4,4,5,7,7,8,13,13,14,13,13,14,16,16,17,13,13,14,13,13,14,16,16,17},
        {9,10,10,12,13,13,12,13,13,9,10,10,12,13,13,12,13,13,18,19,19,21,22,22,21,22,22},
        {10,10,11,13,13,14,13,13,14,10,10,11,13,13,14,13,13,14,19,19,20,22,22,23,22,22,23},
        {12,13,13,12,13,13,15,16,16,12,13,13,12,13,13,15,16,16,21,22,22,21,22,22,24,25,25},
        {13,13,14,13,13,14,16,16,17,13,13,14,13,13,14,16,16,17,22,22,23,22,22,23,25,25,26}
};
__constant__ int LUTchild[8][27]={
        {7,6,7,5,4,5,7,6,7,3,2,3,1,0,1,3,2,3,7,6,7,5,4,5,7,6,7},
        {6,7,6,4,5,4,6,7,6,2,3,2,0,1,0,2,3,2,6,7,6,4,5,4,6,7,6},
        {5,4,5,7,6,7,5,4,5,1,0,1,3,2,3,1,0,1,5,4,5,7,6,7,5,4,5},
        {4,5,4,6,7,6,4,5,4,0,1,0,2,3,2,0,1,0,4,5,4,6,7,6,4,5,4},
        {3,2,3,1,0,1,3,2,3,7,6,7,5,4,5,7,6,7,3,2,3,1,0,1,3,2,3},
        {2,3,2,0,1,0,2,3,2,6,7,6,4,5,4,6,7,6,2,3,2,0,1,0,2,3,2},
        {1,0,1,3,2,3,1,0,1,5,4,5,7,6,7,5,4,5,1,0,1,3,2,3,1,0,1},
        {0,1,0,2,3,2,0,1,0,4,5,4,6,7,6,4,5,4,0,1,0,2,3,2,0,1,0}
};

const int markOffset_h=31;
const int maxDepth_h=9;
const int normalize=0;

int LUTparent_h[8][27]={
        {0,1,1,3,4,4,3,4,4,9,10,10,12,13,13,12,13,13,9,10,10,12,13,13,12,13,13},
        {1,1,2,4,4,5,4,4,5,10,10,11,13,13,14,13,13,14,10,10,11,13,13,14,13,13,14},
        {3,4,4,3,4,4,6,7,7,12,13,13,12,13,13,15,16,16,12,13,13,12,13,13,15,16,16},
        {4,4,5,4,4,5,7,7,8,13,13,14,13,13,14,16,16,17,13,13,14,13,13,14,16,16,17},
        {9,10,10,12,13,13,12,13,13,9,10,10,12,13,13,12,13,13,18,19,19,21,22,22,21,22,22},
        {10,10,11,13,13,14,13,13,14,10,10,11,13,13,14,13,13,14,19,19,20,22,22,23,22,22,23},
        {12,13,13,12,13,13,15,16,16,12,13,13,12,13,13,15,16,16,21,22,22,21,22,22,24,25,25},
        {13,13,14,13,13,14,16,16,17,13,13,14,13,13,14,16,16,17,22,22,23,22,22,23,25,25,26}
};
int LUTchild_h[8][27]={
        {7,6,7,5,4,5,7,6,7,3,2,3,1,0,1,3,2,3,7,6,7,5,4,5,7,6,7},
        {6,7,6,4,5,4,6,7,6,2,3,2,0,1,0,2,3,2,6,7,6,4,5,4,6,7,6},
        {5,4,5,7,6,7,5,4,5,1,0,1,3,2,3,1,0,1,5,4,5,7,6,7,5,4,5},
        {4,5,4,6,7,6,4,5,4,0,1,0,2,3,2,0,1,0,4,5,4,6,7,6,4,5,4},
        {3,2,3,1,0,1,3,2,3,7,6,7,5,4,5,7,6,7,3,2,3,1,0,1,3,2,3},
        {2,3,2,0,1,0,2,3,2,6,7,6,4,5,4,6,7,6,2,3,2,0,1,0,2,3,2},
        {1,0,1,3,2,3,1,0,1,5,4,5,7,6,7,5,4,5,1,0,1,3,2,3,1,0,1},
        {0,1,0,2,3,2,0,1,0,4,5,4,6,7,6,4,5,4,0,1,0,2,3,2,0,1,0}
};

struct markCompact{
    __device__ bool operator()(const long long &x){
        return ( x & (1ll<<markOffset) ) > 0;
    }
};

__device__ long long encodePoint(const Point3D<float> &pos,const long long& idx){
    long long key=0ll;
    Point3D<float> myCenter;
    myCenter.coords[0]=float(0.5);
    myCenter.coords[1]=float(0.5);
    myCenter.coords[2]=float(0.5);

    float myWidth=0.25f;
    for(int i=maxDepth-1;i>=0;--i){
        if(pos.coords[0] > myCenter.coords[0]) {
            key |= 1ll << (3 * i + 34);
            myCenter.coords[0] += myWidth;
        }else{
            myCenter.coords[0] -= myWidth;
        }

        if(pos.coords[1] > myCenter.coords[1]) {
            key |= 1ll << (3 * i + 33);
            myCenter.coords[1] += myWidth;
        }else{
            myCenter.coords[1] -= myWidth;
        }

        if(pos.coords[2] > myCenter.coords[2]) {
            key |= 1ll << (3 * i + 32);
            myCenter.coords[2] += myWidth;
        }else{
            myCenter.coords[2] -= myWidth;
        }
        myWidth/=2;
    }
    return key+idx;
}

__global__ void generateCode(Point3D<float> *points,long long *code,int size){
    long long stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    long long blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    long long offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(long long i=offset;i<size;i+=stride){
        code[i]= encodePoint(points[i],i);
    }
}

__global__ void generateStartHash(long long *key,KeyValue *hashTable,int size){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<size;i+=stride){
        // same key[i]>>32 meet, the point at same node meet
        insertMin(hashTable,int(key[i]>>32),i);
    }
}

__global__ void generateCountHash(long long *key,KeyValue *hashTable,int size){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<size;i+=stride){
        keyAdd(hashTable,int(key[i]>>32));
    }
}


__global__ void generateMark(long long *code,int size){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    if(offset==0){
        code[0] &= ~((1ll<<32)-1);
        code[0]|=1ll<<markOffset;
        offset+=stride;
    }
    for(int i=offset;i<size;i+=stride){
        code[i] &= ~((1ll<<32)-1);
        code[i] += i;
        if(code[i]>>32 != code[i-1]>>32) {
            code[i] |= 1ll << markOffset;
        }
    }
}

__global__ void initUniqueNode(long long *uniqueCode, KeyValue *keyStart,KeyValue *keyCount,
                               int *OriginIdx,OctNode *uniqueNode, int size){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<size;i+=stride){
        uniqueNode[i].key= int(uniqueCode[i] >> 32 ) ;
        uniqueNode[i].pidx=find(keyStart,uniqueNode[i].key);
        uniqueNode[i].pnum=find(keyCount,uniqueNode[i].key);
        OriginIdx[i]=int(uniqueCode[i] & ((1ll<<markOffset)-1) );
    }
}

__global__ void generateNodeNums(long long* uniqueCode,int *nodeNums,int size,int depthD){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    if(offset==0){
        offset+=stride;
    }
    for(int i=offset;i<size;i+=stride){
        if( (uniqueCode[i-1]>>(32 + 3 * (maxDepth-depthD+1) ) )  != (uniqueCode[i]>>(32 + 3 * (maxDepth-depthD+1) ) ) ){
            nodeNums[i]=8;
        }
    }
}


__global__ void generateNodeArrayD(int *OriginIdx,OctNode *uniqueNode,int *nodeAddress,int *PointToNodeArrayD,OctNode *NodeArrayD,int size){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<size;i+=stride){
        int idx=nodeAddress[i] + ( uniqueNode[i].key & 7);
        NodeArrayD[idx] = uniqueNode[i];
        PointToNodeArrayD[OriginIdx[i]] = idx;
    }
}

__global__ void processPointToNodeArrayD(int *PointToNodeArrayD,int count) {
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<count;i+=stride) {
        int nowIdx=i;
        int val=PointToNodeArrayD[nowIdx];
        while(val==-1) {
            --nowIdx;
            val=PointToNodeArrayD[nowIdx];
        }
        PointToNodeArrayD[i]=val;
    }
}

__global__ void initNodeArrayD_DIdxDnum(OctNode *NodeArrayD,int size){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<size;i+=stride){
        NodeArrayD[i].dnum = 1;
        NodeArrayD[i].didx = i;
    }

}

__global__ void parallelSet0xff(OctNode *uniqueNode_D_1,int size){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<size;i+=stride) {
        uniqueNode_D_1[i].pidx=0x7fffffff;
        uniqueNode_D_1[i].didx=0x7fffffff;
    }
}

__global__ void generateNodeKeyIndexHash(OctNode *uniqueNode,int *nodeAddress,int uniqueCount,int depthD,KeyValue *keyIndexHash){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<uniqueCount;i+=stride){
        int fatherKey=uniqueNode[i].key & (~ (7<< (3 * (maxDepth-depthD) ) ) );
//        printf("key:%d\n",fatherKey);
        insertMin(keyIndexHash,fatherKey,nodeAddress[i]/8);
    }
}



__global__ void generateUniqueNodeArrayD_1(OctNode *NodeArray_D,int DSize,KeyValue *keyIndexHash,int depthD,OctNode *uniqueNodeArrayD_1,KeyValue *uniqueNode_D_1_Idx_To_NodeArray_D_Idx){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<DSize;i+=stride){
        if(NodeArray_D[i].pnum==0){
            int st = i-i%8;
            int valid_idx;
            for(int j=0;j<8;++j){
                valid_idx = st+j;
                if(NodeArray_D[valid_idx].pnum != 0){
                    break;
                }
            }
            int fatherKey=NodeArray_D[valid_idx].key & (~ (7<< (3 * (maxDepth-depthD) ) ) );
            int idx=find(keyIndexHash,fatherKey);
            if(NodeArray_D[i].dnum!=0) {
                atomicAdd(&uniqueNodeArrayD_1[idx].dnum, NodeArray_D[i].dnum);
                atomicMin(&uniqueNodeArrayD_1[idx].didx, NodeArray_D[i].didx);
            }
            continue;
        }
        int fatherKey=NodeArray_D[i].key & (~ (7<< (3 * (maxDepth-depthD) ) ) );
        int idx=find(keyIndexHash,fatherKey);
        int sonKey = ( NodeArray_D[i].key >> (3 * (maxDepth-depthD)) ) & 7;
        uniqueNodeArrayD_1[idx].key=fatherKey;
        atomicAdd(&uniqueNodeArrayD_1[idx].pnum,NodeArray_D[i].pnum);
        atomicMin(&uniqueNodeArrayD_1[idx].pidx,NodeArray_D[i].pidx);
        atomicAdd(&uniqueNodeArrayD_1[idx].dnum,NodeArray_D[i].dnum);
        atomicMin(&uniqueNodeArrayD_1[idx].didx,NodeArray_D[i].didx);
        insert(uniqueNode_D_1_Idx_To_NodeArray_D_Idx,idx,i);
        uniqueNodeArrayD_1[idx].children[sonKey]=i;
    }
}

__global__ void generateNodeNumsD_1(OctNode *uniqueNodeArrayD_1,int *NodeNums_D_1,int size,int depthD){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    if(offset==0){
        offset+=stride;
    }
    for(int i=offset;i<size;i+=stride){
        if( (uniqueNodeArrayD_1[i-1].key >> (3 * (maxDepth-depthD+1) ) )  != (uniqueNodeArrayD_1[i].key >> ( 3 * (maxDepth-depthD+1) ) ) ){
            NodeNums_D_1[i]=8;
        }
    }
}

__global__ void generateNodeArrayD_1(OctNode *uniqueNodeArrayD_1,int *nodeAddressD_1,OctNode *NodeArrayD_1,int size,int depthD,KeyValue *uniqueNode_D_1_Idx_To_NodeArray_D_Idx,OctNode *NodeArray_D){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<size;i+=stride){
        int newIdx=nodeAddressD_1[i] + ( (uniqueNodeArrayD_1[i].key>>(3*(maxDepth-depthD+1) ) ) & 7);
        NodeArrayD_1[newIdx] = uniqueNodeArrayD_1[i];
        NodeArray_D[find(uniqueNode_D_1_Idx_To_NodeArray_D_Idx,i)].parent=newIdx;
    }
}

__host__ void pipelineUniqueNode_D_1(OctNode *uniqueNode_D,int *nodeAddress_D,int uniqueCount_D,OctNode *NodeArray_D,int allNodeNums_D,int depthD,
                                     OctNode *&uniqueNode_D_1, int &uniqueCount_D_1, KeyValue *&uniqueNode_D_1_Idx_To_NodeArray_D_Idx)
{
    if(uniqueNode_D_1_Idx_To_NodeArray_D_Idx!=NULL){
        destroy_hashtable(uniqueNode_D_1_Idx_To_NodeArray_D_Idx);
    }
    uniqueNode_D_1_Idx_To_NodeArray_D_Idx=create_hashtable();
    uniqueCount_D_1=allNodeNums_D/8;
    long long nByte = 1ll * sizeof(OctNode) * uniqueCount_D_1;
//    CHECK(hipMalloc((OctNode **)&uniqueNode_D_1,nByte));
//    CHECK(hipMemset(uniqueNode_D_1,0,nByte));
    KeyValue *keyIndexHash=create_hashtable();
    dim3 grid=(32,32);
    dim3 block=(32,32);
    parallelSet0xff<<<grid,block>>>(uniqueNode_D_1,uniqueCount_D_1);
    generateNodeKeyIndexHash<<<grid,block>>>(uniqueNode_D,nodeAddress_D,uniqueCount_D,depthD,keyIndexHash);
    hipDeviceSynchronize();
    generateUniqueNodeArrayD_1<<<grid,block>>>(NodeArray_D,allNodeNums_D,keyIndexHash,depthD,uniqueNode_D_1,uniqueNode_D_1_Idx_To_NodeArray_D_Idx);
    hipDeviceSynchronize();
    destroy_hashtable(keyIndexHash);
}

__host__ void pipelineNodeAddress_D_1(OctNode *uniqueNode_D_1,int uniqueCount_D_1,int depthD,
                                      int *&NodeAddress_D_1)
{
    dim3 grid=(32,32);
    dim3 block=(32,32);
    int *NodeNums_D_1=NULL;
    long long nByte = 1ll * sizeof(int)*uniqueCount_D_1;
    CHECK(hipMalloc((int **)&NodeNums_D_1,nByte));
    CHECK(hipMemset(NodeNums_D_1,0,nByte));
//    CHECK(hipMalloc((int **)&NodeAddress_D_1,nByte));
//    CHECK(hipMemset(NodeAddress_D_1,0,nByte));
    generateNodeNumsD_1<<<grid,block>>>(uniqueNode_D_1, NodeNums_D_1, uniqueCount_D_1, depthD-1);
    hipDeviceSynchronize();

    thrust::device_ptr<int> NodeNums_D_1_ptr=thrust::device_pointer_cast<int>(NodeNums_D_1);
    thrust::device_ptr<int> NodeAddress_D_1_ptr=thrust::device_pointer_cast<int>(NodeAddress_D_1);
    thrust::inclusive_scan(NodeNums_D_1_ptr,NodeNums_D_1_ptr+uniqueCount_D_1,NodeAddress_D_1_ptr);
    hipDeviceSynchronize();
    hipFree(NodeNums_D_1);
}

__global__ void updateParentChildren(int *BaseAddressArray_d,OctNode *NodeArray,int size){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<size;i+=stride){
        if(NodeArray[i].pnum == 0) continue;
        int depth;
        for(depth=0;depth<maxDepth_h;++depth){
            if(BaseAddressArray_d[depth] <= i && BaseAddressArray_d[depth+1] > i){
                break;
            }
        }
        if(i == 0){
            NodeArray[i].parent=-1;
#pragma unroll
            for(int child=0;child<8;++child){
                NodeArray[i].children[child] += BaseAddressArray_d[depth+1];
            }
        }else {
            NodeArray[i].parent += BaseAddressArray_d[depth - 1];
#pragma unroll
            for(int child=0;child<8;++child){
                if(NodeArray[i].children[child]!=0)
                    NodeArray[i].children[child] += BaseAddressArray_d[depth+1];
            }
        }
    }
}

__global__ void updateEmptyNodeInfo(int *BaseAddressArray_d,OctNode *NodeArray,int size){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=1+8*offset;i<size;i+=8*stride){
        int nowPIdx;
        int nowDIdx;
        int validIdx;
        int commonParent;
        for(validIdx=0;validIdx<8;++validIdx){
            if(NodeArray[i+validIdx].pnum!=0){
                nowPIdx=NodeArray[i+validIdx].pidx;
                nowDIdx=NodeArray[i+validIdx].didx;
                commonParent=NodeArray[i+validIdx].parent;
                break;
            }
        }
        int depth;
        for(depth=0;depth<maxDepth_h;++depth){
            if(BaseAddressArray_d[depth] <= i && BaseAddressArray_d[depth+1] > i){
                break;
            }
        }
        int baseKey = NodeArray[i+validIdx].key - ( ( NodeArray[i+validIdx].key ) & ( 7 << (3 * (maxDepth-depth)) ) );

        for(int j=0;j<8;++j){
            int idx=i+j;
            if(NodeArray[idx].pnum==0){
                for(int k=0;k<8;++k){
                    NodeArray[idx].children[k]=-1;
                }
            }else{
                int basePos;
                for(int k=0;k<8;++k){
                    if(NodeArray[idx].children[k]>0){
                        basePos=NodeArray[idx].children[k]-k;
                        break;
                    }
                }
                for(int k=0;k<8;++k){
                    NodeArray[idx].children[k]=basePos+k;
                }
            }
            NodeArray[idx].key = baseKey + ( j << (3 * (maxDepth-depth)) );

            NodeArray[idx].pidx = nowPIdx;
            nowPIdx += NodeArray[idx].pnum;

            if(depth != maxDepth) {
                NodeArray[idx].didx = nowDIdx;
                nowDIdx += NodeArray[idx].dnum;
            }

            NodeArray[idx].parent=commonParent;

        }
    }
}

__global__ void computeNeighbor(OctNode *NodeArray,int left,int right,int depthD){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    for(int i=offset;i<right;i+=stride){
        for(int j=0;j<27;++j){
            int sonKey = ( NodeArray[i].key >> (3 * (maxDepth-depthD)) ) & 7;
            int parentIdx = NodeArray[i].parent;
            int neighParent = NodeArray[ parentIdx ].neighs[LUTparent[sonKey][j]];
            if(neighParent != -1){
                NodeArray[i].neighs[j] = NodeArray[ neighParent ].children[LUTchild[sonKey][j]];
            }else{
                NodeArray[i].neighs[j]= -1;
            }
        }
    }
}

__host__ void pipelineBuildNodeArray(char *fileName,Point3D<float> &center,float &scale,int &count,int &NodeArray_sz,
                                     int NodeArrayCount_h[maxDepth_h+1],int BaseAddressArray_h[maxDepth_h+1], //host
                                     Point3D<float> *&samplePoints_d,Point3D<float> *&sampleNormals_d,int *&PointToNodeArrayD,OctNode *&NodeArray)    //device
{
    count=0;
    PointStream<float>* pointStream;
    char* ext = GetFileExtension(fileName);
    if      (!strcasecmp(ext,"bnpts"))      pointStream = new BinaryPointStream<float>(fileName);
    else if (!strcasecmp(ext,"ply"))        pointStream = new PLYPointStream<float>(fileName);
    else                                    pointStream = new ASCIIPointStream<float>(fileName);

    Point3D<float> position,normal;
    Point3D<float> mx,mn;

    scale=1;
    float scaleFactor=1.25;

    double st=cpuSecond();

    /**     Step 1: compute bounding box     */
    while(pointStream->nextPoint(position,normal)){
        for(int i=0;i<DIMENSION;++i){
            if(!count || position.coords[i]<mn.coords[i]) mn.coords[i]=position.coords[i];
            if(!count || position.coords[i]>mx.coords[i]) mx.coords[i]=position.coords[i];
        }
        ++count;
    }

    for(int i=0;i<DIMENSION;++i){
        if(!i || scale<mx.coords[i]-mn.coords[i]) scale=float(mx.coords[i]-mn.coords[i]);
        center.coords[i]=float(mx.coords[i]+mn.coords[i])/2;
    }
    scale*=scaleFactor;
    for(int i=0;i<DIMENSION;++i)
        center.coords[i]-=scale/2;

    Point3D<float> *p_h=(Point3D<float> *)malloc(sizeof(Point3D<float>) * count);
    Point3D<float> *n_h=(Point3D<float> *)malloc(sizeof(Point3D<float>) * count);

    pointStream->reset();
    int idx=0;
    while(pointStream->nextPoint(position,normal)){
        int i;
        for(i=0;i<DIMENSION;++i)
            position.coords[i]=(position.coords[i]-center.coords[i])/scale;
        for(i=0;i<DIMENSION;++i)
            if(position.coords[i]<0 || position.coords[i]>1)
                break;
        p_h[idx]=position;

#if FORCE_UNIT_NORMALS
        float len=float(Length(normal));
        if(len>EPSILON)
            len=1.0f/len;
        len*=(2<<maxDepth);
        for(i=0;i<DIMENSION;++i)
            normal.coords[i]*=len;
#endif
        n_h[idx]=normal;
        ++idx;
    }

    //  input process may can be optimized as GPU parallel
    double mid=cpuSecond();
    printf("Total points number:%d ,Read takes:%lfs\n",count,mid-st);

    CHECK(hipMalloc((Point3D<float> **)&samplePoints_d,sizeof(Point3D<float>) * count));
    CHECK(hipMemcpy(samplePoints_d,p_h,sizeof(Point3D<float>) * count, hipMemcpyHostToDevice));

    CHECK(hipMalloc((Point3D<float> **)&sampleNormals_d,sizeof(Point3D<float>) * count));
    CHECK(hipMemcpy(sampleNormals_d,n_h,sizeof(Point3D<float>) * count, hipMemcpyHostToDevice));

    /**     Step 2: compute shuffled xyz key and sorting code   */
    long long *key=NULL;
    long long nByte = 1ll * sizeof(long long)*count;
    CHECK(hipMalloc((long long **)&key, nByte));
    dim3 grid=(32,32);
    dim3 block=(32,32);
    generateCode<<<grid,block>>>(samplePoints_d,key,count);
    hipDeviceSynchronize();

    long long *key_backup=NULL;
    CHECK(hipMalloc((long long **)&key_backup, nByte));
    CHECK(hipMemcpy(key_backup,key,nByte,hipMemcpyDeviceToDevice));

    /**     Step 3: sort all sample points      */
    thrust::device_ptr<long long> key_ptr=thrust::device_pointer_cast<long long>(key);
    thrust::sort_by_key(key_ptr,key_ptr+count,samplePoints_d);
    hipDeviceSynchronize();

    key_ptr=thrust::device_pointer_cast<long long>(key_backup);
    thrust::sort_by_key(key_ptr,key_ptr+count,sampleNormals_d);
    hipDeviceSynchronize();

    hipFree(key_backup);
    key_ptr=thrust::device_pointer_cast<long long>(key);

    KeyValue* start_hashTable=create_hashtable();
    KeyValue* count_hashTable=create_hashtable();

    generateStartHash<<<grid,block>>>(key, start_hashTable,count);
    generateCountHash<<<grid,block>>>(key, count_hashTable,count);


    /**     Step 4: find the unique nodes       */
    generateMark<<<grid,block>>>(key,count);
    hipDeviceSynchronize();
    long long *uniqueCode=NULL;
    CHECK(hipMalloc((long long **)&uniqueCode,sizeof(long long) * count));
    thrust::device_ptr<long long> uniqueCode_ptr=thrust::device_pointer_cast<long long>(uniqueCode);
    thrust::device_ptr<long long> uniqueCode_end=thrust::copy_if(key_ptr,key_ptr+count,uniqueCode_ptr,markCompact());
    hipDeviceSynchronize();
    hipFree(key);

    int uniqueCount_h=uniqueCode_end-uniqueCode_ptr;

    /**     Create uniqueNode according to uniqueCode   */
    OctNode *uniqueNode=NULL;
    nByte = 1ll * sizeof(OctNode)*uniqueCount_h;
    CHECK(hipMalloc((OctNode **)&uniqueNode,nByte));
    CHECK(hipMemset(uniqueNode,0,nByte));
    int *OriginIdx=NULL;
    nByte = 1ll * sizeof(int)*uniqueCount_h;
    CHECK(hipMalloc((int**)&OriginIdx,nByte));
//    CHECK(hipMemset(OriginIdx,-1,nByte));
    initUniqueNode<<<grid,block>>>(uniqueCode,start_hashTable,count_hashTable,
                                                   OriginIdx,uniqueNode,uniqueCount_h);
    hipDeviceSynchronize();

    destroy_hashtable(start_hashTable);
    destroy_hashtable(count_hashTable);

    /**     Step 5: augment uniqueNode      */
    int *nodeNums=NULL;
    int *nodeAddress=NULL;
    nByte = 1ll * sizeof(int)*uniqueCount_h;
    CHECK(hipMalloc((int **)&nodeNums,nByte));
    CHECK(hipMemset(nodeNums,0,nByte));

    CHECK(hipMalloc((int **)&nodeAddress,nByte));
    CHECK(hipMemset(nodeAddress,0,nByte));

    generateNodeNums<<<grid,block>>>(uniqueCode,nodeNums,uniqueCount_h,maxDepth_h);
    hipDeviceSynchronize();

    hipFree(uniqueCode);

    thrust::device_ptr<int> nodeNums_ptr=thrust::device_pointer_cast<int>(nodeNums);
    thrust::device_ptr<int> nodeAddress_ptr=thrust::device_pointer_cast<int>(nodeAddress);

    thrust::inclusive_scan(nodeNums_ptr,nodeNums_ptr+uniqueCount_h,nodeAddress_ptr);
    hipDeviceSynchronize();

    hipFree(nodeNums);


    /**     Step 6: create NodeArrayD       */
    int lastAddr;
    CHECK(hipMemcpy(&lastAddr,nodeAddress+uniqueCount_h-1,sizeof(int),hipMemcpyDeviceToHost));

    int allNodeNums=lastAddr+8;
    OctNode *NodeArrayD=NULL;
    nByte = 1ll * sizeof(OctNode) * allNodeNums;
    CHECK(hipMalloc((OctNode **)&NodeArrayD, nByte));
    CHECK(hipMemset(NodeArrayD,0,nByte));
//    int *PointToNodeArrayD=NULL;
    nByte = 1ll * sizeof(int) * count;
    CHECK(hipMalloc((int**)&PointToNodeArrayD,nByte));
    CHECK(hipMemset(PointToNodeArrayD,-1,nByte));
    generateNodeArrayD<<<grid,block>>>(OriginIdx,uniqueNode,nodeAddress,PointToNodeArrayD,NodeArrayD,uniqueCount_h);
    hipDeviceSynchronize();
    initNodeArrayD_DIdxDnum<<<grid,block>>>(NodeArrayD,allNodeNums);
    processPointToNodeArrayD<<<grid,block>>>(PointToNodeArrayD,count);
    hipDeviceSynchronize();


//    OctNode *a=(OctNode *)malloc(sizeof(OctNode)*allNodeNums);
//    hipMemcpy(a,NodeArrayD,sizeof(OctNode)*(allNodeNums),hipMemcpyDeviceToHost);
//    for (int i = 0; i < 50; ++i) {
////            if(a[i].pnum==0) continue;
//        std::cout << i << " " <<std::bitset<32>(a[i].key) << " pidx:" << a[i].pidx << " pnum:" << a[i].pnum << " parent:"
//                  << a[i].parent << " didx:"<< a[i].didx << " dnum:" << a[i].dnum << std::endl;
////            for(int k=0;k<8;++k){
////                printf("children[%d]:%d ",k,a[i].children[k]);
////            }
////            puts("");
////            for(int k=0;k<27;++k){
////                printf("neigh[%d]:%d ",k,a[i].neighs[k]);
////            }
////            puts("");
//    }

    /**     D-1     */
    memset(BaseAddressArray_h,0,sizeof(int) * (maxDepth_h+1));
    OctNode **NodeArrays=(OctNode **)malloc(sizeof(OctNode *) * (maxDepth_h+1));
    NodeArrays[maxDepth_h]=NodeArrayD;

    OctNode *uniqueNode_D=uniqueNode;
    int *NodeAddress_D=nodeAddress;
    int uniqueCount_D=uniqueCount_h;
    int allNodeNums_D=allNodeNums;
    OctNode *NodeArray_D=NodeArrayD;
    for(int depthD=maxDepth_h;depthD>=1;--depthD){
//        printf("depthD:%d %d\n",depthD,allNodeNums_D);
        NodeArrayCount_h[depthD]=allNodeNums_D;

        OctNode *uniqueNode_D_1=NULL,*NodeArray_D_1=NULL;
        int D_1Nums=allNodeNums_D/8;
        nByte = 1ll * sizeof(OctNode) * D_1Nums;
        CHECK(hipMalloc((OctNode **)&uniqueNode_D_1,nByte));
        CHECK(hipMemset(uniqueNode_D_1,0,nByte));
        int *NodeAddress_D_1=NULL;
        nByte = 1ll * sizeof(int) * D_1Nums;
        CHECK(hipMalloc((int **)&NodeAddress_D_1,nByte));
        CHECK(hipMemset(NodeAddress_D_1,0,nByte));
        int uniqueCount_D_1;
        int allNodeNums_D_1;
        KeyValue *uniqueNode_D_1_Idx_To_NodeArray_D_Idx=NULL;
        pipelineUniqueNode_D_1(uniqueNode_D,NodeAddress_D,uniqueCount_D,NodeArray_D,allNodeNums_D,depthD,
                               uniqueNode_D_1,uniqueCount_D_1,uniqueNode_D_1_Idx_To_NodeArray_D_Idx);
        pipelineNodeAddress_D_1(uniqueNode_D_1,uniqueCount_D_1,depthD,
                                NodeAddress_D_1);

        if(depthD>1) {
            int lastAddrD_1;
            CHECK(hipMemcpy(&lastAddrD_1, NodeAddress_D_1 + uniqueCount_D_1 - 1, sizeof(int), hipMemcpyDeviceToHost));
            allNodeNums_D_1 = lastAddrD_1 + 8;

            nByte = 1ll * sizeof(OctNode) * allNodeNums_D_1;
            CHECK(hipMalloc((OctNode **) &NodeArray_D_1, nByte));
            CHECK(hipMemset(NodeArray_D_1, 0, nByte));

            // update NodeArray_D's parent in this global function
            generateNodeArrayD_1<<<grid, block>>>(uniqueNode_D_1, NodeAddress_D_1, NodeArray_D_1, uniqueCount_D_1, depthD,uniqueNode_D_1_Idx_To_NodeArray_D_Idx,NodeArray_D);
            hipDeviceSynchronize();
        }else{
            // D=1, D_1=0
            // the parent of NodeArray_D = 0, don't need to update
            allNodeNums_D_1 = 1;
            NodeArray_D_1=uniqueNode_D_1;
        }

        NodeArrays[depthD-1]=NodeArray_D_1;
        NodeArray_D=NodeArray_D_1;

//        nByte = 1ll * sizeof(OctNode) *uniqueCount_D_1;
//        OctNode *h=(OctNode*)malloc(nByte);
//        hipMemcpy(h,uniqueNode_D_1,nByte,hipMemcpyDeviceToHost);
//        for(int i=0;i<uniqueCount_D_1;++i){
//            std::cout<<std::bitset<32>(h[i].key)<<" pidx:"<<h[i].pidx<<" pnum:"<<h[i].pnum<<std::endl;
//        }
//        printf("depth:%d uniqueNode:%d NodeArray:%d\n",depthD,uniqueCount_D_1, allNodeNums_D_1);

        hipFree(uniqueNode_D);
        uniqueNode_D=uniqueNode_D_1;
        hipFree(NodeAddress_D);
        NodeAddress_D=NodeAddress_D_1;
        uniqueCount_D=uniqueCount_D_1;
        allNodeNums_D=allNodeNums_D_1;
    }

    NodeArrayCount_h[0]=1;
    for(int i=1;i<=maxDepth_h;++i){
        BaseAddressArray_h[i]=BaseAddressArray_h[i-1]+NodeArrayCount_h[i-1];
//        printf("%d %d\n",BaseAddressArray_h[i],NodeArrayCount_h[i]);
    }

    nByte = 1ll * sizeof(int)*(maxDepth_h+1);
//    int *NodeArrayCount_d=NULL;
//    CHECK(hipMalloc((int **)&NodeArrayCount_d,nByte));
//    CHECK(hipMemcpy(NodeArrayCount_d,NodeArrayCount_h,nByte,hipMemcpyHostToDevice));
    int *BaseAddressArray_d=NULL;
    CHECK(hipMalloc((int **)&BaseAddressArray_d,nByte));
    CHECK(hipMemcpy(BaseAddressArray_d,BaseAddressArray_h,nByte,hipMemcpyHostToDevice));

    nByte = 1ll * sizeof(OctNode)*(BaseAddressArray_h[maxDepth_h]+NodeArrayCount_h[maxDepth_h]);
//    printf("%d\n",BaseAddressArray[maxDepth_h]+NodeArrayCount_h[maxDepth_h]);
    CHECK(hipMalloc((OctNode **)&NodeArray,nByte));
    for(int i=0;i<=maxDepth_h;++i){
        CHECK(hipMemcpy(NodeArray+BaseAddressArray_h[i],NodeArrays[i],sizeof(OctNode) * NodeArrayCount_h[i], hipMemcpyDeviceToDevice ));
        hipFree(NodeArrays[i]);
    }

    NodeArray_sz=(BaseAddressArray_h[maxDepth_h]+NodeArrayCount_h[maxDepth_h]);
    updateParentChildren<<<grid,block>>>(BaseAddressArray_d,NodeArray,NodeArray_sz);
    hipDeviceSynchronize();

    updateEmptyNodeInfo<<<grid,block>>>(BaseAddressArray_d,NodeArray,NodeArray_sz);
    hipDeviceSynchronize();

    hipFree(BaseAddressArray_d);

    int Node_0_Neighs[27];
    for(int i=0;i<27;++i)
        Node_0_Neighs[i]=-1;
    Node_0_Neighs[13]=0;

    CHECK(hipMemcpy(NodeArray[0].neighs,Node_0_Neighs,sizeof(int) * 27,hipMemcpyHostToDevice));


    for(int depth=1;depth<=maxDepth_h;++depth){
        computeNeighbor<<<grid,block>>>(NodeArray,BaseAddressArray_h[depth],BaseAddressArray_h[depth]+NodeArrayCount_h[depth],depth);
        hipDeviceSynchronize();
    }

//    OctNode *a=(OctNode *)malloc(sizeof(OctNode)*NodeArray_sz);
//    hipMemcpy(a,NodeArray,sizeof(OctNode)*(BaseAddressArray_h[maxDepth_h]+NodeArrayCount_h[maxDepth_h]),hipMemcpyDeviceToHost);
//    for(int j=0;j<=2;++j) {
//        int all=0;
//        for (int i = BaseAddressArray_h[j]; i < BaseAddressArray_h[j]+10; ++i) {
////            if(a[i].pnum==0) continue;
//            all+=a[i].dnum;
//            std::cout << i << " " <<std::bitset<32>(a[i].key) << " pidx:" << a[i].pidx << " pnum:" << a[i].pnum << " parent:"
//                      << a[i].parent << " didx:"<< a[i].didx << " dnum:" << a[i].dnum << std::endl;
//            for(int k=0;k<8;++k){
//                printf("children[%d]:%d ",k,a[i].children[k]);
//            }
//            puts("");
//            for(int k=0;k<27;++k){
//                printf("neigh[%d]:%d ",k,a[i].neighs[k]);
//            }
//            puts("");
//        }
//        printf("allD:%d\n",all);
//        std::cout<<std::endl;
//    }

    double ed=cpuSecond();
    printf("GPU NodeArray build takes:%lfs\n",ed-mid);

}

__host__ __device__ int getDepth(const int& idxOfNodeArray,int *&BaseAddressArray){
    int depth=0;
#if defined(__CUDA_ARCH__)
    for(depth=0;depth<maxDepth;++depth){
        if(BaseAddressArray[depth] <= idxOfNodeArray && BaseAddressArray[depth+1] > idxOfNodeArray){
            break;
        }
    }
#elif !defined(__CUDA_ARCH__)
    for(depth=0;depth<maxDepth_h;++depth){
        if(BaseAddressArray[depth] <= idxOfNodeArray && BaseAddressArray[depth+1] > idxOfNodeArray){
            break;
        }
    }
#endif
    return depth;
}

__host__ __device__ void getFunctionIdxOfNode(const int& key,const int &depthD,int idx[3]){
    idx[0]=(1<<depthD)-1;
    idx[1]=idx[0];
    idx[2]=idx[1];
#if defined(__CUDA_ARCH__)
    for(int depth=depthD;depth >= 1;--depth){
        int sonKeyX = ( key >> (3 * (maxDepth-depth) + 2) ) & 1;
        int sonKeyY = ( key >> (3 * (maxDepth-depth) + 1) ) & 1;
        int sonKeyZ = ( key >> (3 * (maxDepth-depth)) ) & 1;
        idx[0] += sonKeyX * (1<<(depthD-depth));
        idx[1] += sonKeyY * (1<<(depthD-depth));
        idx[2] += sonKeyZ * (1<<(depthD-depth));
    }
//    if(depthD==2) {
//        printf("%d %d %d\n",idx[0],idx[1],idx[2]);
//    }
#elif !defined(__CUDA_ARCH__)
    for(int depth=depthD;depth >= 1;--depth){
        int sonKeyX = ( key >> (3 * (maxDepth_h-depth) + 2) ) & 1;
        int sonKeyY = ( key >> (3 * (maxDepth_h-depth) + 1) ) & 1;
        int sonKeyZ = ( key >> (3 * (maxDepth_h-depth)) ) & 1;
        idx[0] += sonKeyX * (1<<(depthD-depth));
        idx[1] += sonKeyY * (1<<(depthD-depth));
        idx[2] += sonKeyZ * (1<<(depthD-depth));
    }
#endif
}

__host__ __device__ void getEncodedFunctionIdxOfNode(const int& key,const int &depthD,int *idx){
#if defined(__CUDA_ARCH__)
    *idx = ((1<<depthD)-1)*(1+(1<<(maxDepth+1))+(1<<(2*(maxDepth+1))) );
    for(int depth=depthD;depth >= 1;--depth){
        int sonKeyX = ( key >> (3 * (maxDepth-depth) + 2) ) & 1;
        int sonKeyY = ( key >> (3 * (maxDepth-depth) + 1) ) & 1;
        int sonKeyZ = ( key >> (3 * (maxDepth-depth)) ) & 1;
        *idx += sonKeyX * (1<<(depthD-depth)) +
                sonKeyY * (1<<(depthD-depth)) * (1<<(maxDepth+1)) +
                sonKeyZ * (1<<(depthD-depth)) * (1<<(2*(maxDepth+1)));
    }
#elif !defined(__CUDA_ARCH__)
    *idx = ((1<<depthD)-1)*(1+(1<<maxDepth_h)+(1<<(2*maxDepth_h)) );
    for(int depth=depthD;depth >= 1;--depth){
        int sonKeyX = ( key >> (3 * (maxDepth_h-depth) + 2) ) & 1;
        int sonKeyY = ( key >> (3 * (maxDepth_h-depth) + 1) ) & 1;
        int sonKeyZ = ( key >> (3 * (maxDepth_h-depth)) ) & 1;
        *idx += sonKeyX * (1<<(depthD-depth)) +
                sonKeyY * (1<<(depthD-depth)) * (1<<maxDepth_h) +
                sonKeyZ * (1<<(depthD-depth)) * (1<<(2*maxDepth_h));
    }
#endif
}

__device__ float F_center_width_Point(const ConfirmedPPolynomial<convTimes,convTimes+2> &BaseFunctionMaxDepth_d,const Point3D<float> &center,const float &width,const Point3D<float> &point){
    ConfirmedPPolynomial<convTimes,convTimes+2> thisFunction_X = BaseFunctionMaxDepth_d.shift(center.coords[0]);
    ConfirmedPPolynomial<convTimes,convTimes+2> thisFunction_Y = BaseFunctionMaxDepth_d.shift(center.coords[1]);
    ConfirmedPPolynomial<convTimes,convTimes+2> thisFunction_Z = BaseFunctionMaxDepth_d.shift(center.coords[2]);
    float x=value(thisFunction_X,point.coords[0]);
    float y=value(thisFunction_Y,point.coords[1]);
    float z=value(thisFunction_Z,point.coords[2]);
    return x*y*z;
}

__global__ void computeVectorField(ConfirmedPPolynomial<convTimes,convTimes+2> *BaseFunctionMaxDepth_d,Point3D<float> *samplePoints_d,Point3D<float> *sampleNormals_d,OctNode *NodeArray,int left,int right,Point3D<float> *VectorField){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    for(int i=offset;i<right;i+=stride){
        int idx[3];
        float width;
        getFunctionIdxOfNode(NodeArray[i].key,maxDepth,idx);
        Point3D<float> o_c;
        BinaryNode<float>::CenterAndWidth(idx[0],o_c.coords[0],width);
        BinaryNode<float>::CenterAndWidth(idx[1],o_c.coords[1],width);
        BinaryNode<float>::CenterAndWidth(idx[2],o_c.coords[2],width);
        int IdxInMaxDepth=i-left;
        Point3D<float> val;
        for(int j=0;j<27;++j){
            int neigh=NodeArray[i].neighs[j];
            if(neigh!=-1){
                for(int k=0;k<NodeArray[neigh].pnum;++k){
                    int pointIdx=NodeArray[neigh].pidx+k;
                    float weight= F_center_width_Point(*BaseFunctionMaxDepth_d,samplePoints_d[pointIdx],width,o_c);
                    val.coords[0] += weight * sampleNormals_d[pointIdx].coords[0];
                    val.coords[1] += weight * sampleNormals_d[pointIdx].coords[1];
                    val.coords[2] += weight * sampleNormals_d[pointIdx].coords[2];
//                    VectorField[IdxInMaxDepth].coords[0] += weight * sampleNormals_d[pointIdx].coords[0];
//                    VectorField[IdxInMaxDepth].coords[1] += weight * sampleNormals_d[pointIdx].coords[1];
//                    VectorField[IdxInMaxDepth].coords[2] += weight * sampleNormals_d[pointIdx].coords[2];
                }
            }
        }
        VectorField[IdxInMaxDepth].coords[0] += val.coords[0];
        VectorField[IdxInMaxDepth].coords[1] += val.coords[1];
        VectorField[IdxInMaxDepth].coords[2] += val.coords[2];
//        printf("%d %f\n",IdxInMaxDepth,VectorField[IdxInMaxDepth].coords[0]);
    }
}

__host__ __device__ float DotProduct(const Point3D<float> &p1,const Point3D<float> &p2){
    float res=0;
    res += p1.coords[0]*p2.coords[0];
    res += p1.coords[1]*p2.coords[1];
    res += p1.coords[2]*p2.coords[2];
    return res;
}

//deprecated
__global__ void precomputeFunctionIdxOfNode(int *BaseAddressArray_d,OctNode *NodeArray,int NodeArray_sz,int *NodeIdxInFunction){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<NodeArray_sz;i+=stride){
        int depthD= getDepth(i,BaseAddressArray_d);
        getFunctionIdxOfNode(NodeArray[i].key,depthD,NodeIdxInFunction+3*i);
    }
}

__global__ void precomputeEncodedFunctionIdxOfNode(int *BaseAddressArray_d,OctNode *NodeArray,int NodeArray_sz,int *NodeIdxInFunction){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<NodeArray_sz;i+=stride){
        int depthD= getDepth(i,BaseAddressArray_d);
        getEncodedFunctionIdxOfNode(NodeArray[i].key,depthD,NodeIdxInFunction+i);
    }
}

__global__ void precomputeEncodedFunctionIdxOfNode(OctNode *NodeArray,int NodeArray_sz,
                                                   int *DepthBuffer,int *NodeIdxInFunction)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<NodeArray_sz;i+=stride){
        int depthD= DepthBuffer[i];
        getEncodedFunctionIdxOfNode(NodeArray[i].key,depthD,NodeIdxInFunction+i);
    }
}

__global__ void computeEncodedFinerNodesDivergence(int *BaseAddressArray_d, int *EncodedNodeIdxInFunction,
                                                   OctNode *NodeArray, int left, int right,
                                                   Point3D<float> *VectorField,const double *dot_F_DF,
                                                   float *Divergence) {
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    int maxD=maxDepth;
    int start_D=BaseAddressArray_d[maxD];
    int res=resolution;
    int decode_offset1=(1<<(maxD+1));
    int decode_offset2=(1<<(2*(maxD+1)));
    for(int i=offset;i<right;i+=stride) {
        double val=0;
#pragma unroll
        for(int j=0;j<27;++j){
            int neighIdx=NodeArray[i].neighs[j];
            if(neighIdx == -1) continue;
            for(int k=0;k<NodeArray[neighIdx].dnum;++k){
                int Node_D_Idx=NodeArray[neighIdx].didx + k ;
                const Point3D<float> &vo = VectorField[Node_D_Idx];

                int idxO_1[3],idxO_2[3];

                int encode_idx=EncodedNodeIdxInFunction[i];
                idxO_1[0]=encode_idx%decode_offset1;
                idxO_1[1]=(encode_idx/decode_offset1)%decode_offset1;
                idxO_1[2]=encode_idx/decode_offset2;

                encode_idx=EncodedNodeIdxInFunction[start_D + Node_D_Idx];
                idxO_2[0]=encode_idx%decode_offset1;
                idxO_2[1]=(encode_idx/decode_offset1)%decode_offset1;
                idxO_2[2]=encode_idx/decode_offset2;

                int scratch[3];
//                scratch[0] = idxO_1[0] * res + idxO_2[0];
//                scratch[1] = idxO_1[1] * res + idxO_2[1];
//                scratch[2] = idxO_1[2] * res + idxO_2[2];
                scratch[0] = idxO_1[0] + idxO_2[0] * res;
                scratch[1] = idxO_1[1] + idxO_2[1] * res;
                scratch[2] = idxO_1[2] + idxO_2[2] * res;

                Point3D<float> uo;
                uo.coords[0]=dot_F_DF[scratch[0]];
                uo.coords[1]=dot_F_DF[scratch[1]];
                uo.coords[2]=dot_F_DF[scratch[2]];
                val += DotProduct(vo,uo);
            }
        }
        Divergence[i] = val;
//        printf("%d %f\n",i,val);
    }
}

__global__ void computeCoverNums(OctNode *NodeArray,int idx,int *coverNums){
    *coverNums=0;
    for(int i=0;i<27;++i){
        int neigh=NodeArray[idx].neighs[i];
        if(neigh != -1){
            *(coverNums+i+1) = NodeArray[neigh].dnum + *(coverNums+i);
        }else{
            *(coverNums+i+1) = *(coverNums+i);
        }
    }
}

__device__ int getNeighIdx(int *&coverNums,int threadId){
    int neighIdx=0;
    for(neighIdx=0;neighIdx<27;++neighIdx){
        if(coverNums[neighIdx] <= threadId && coverNums[neighIdx+1] > threadId){
            break;
        }
    }
    return neighIdx;
}

__global__ void generateDIdxArray(OctNode *NodeArray,int idx,int *coverNums,int *DIdxArray){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int size=coverNums[27];
    for(int i=offset;i<size;i+=stride){
        int neighIdx= getNeighIdx(coverNums,i);
        int st=NodeArray[ NodeArray[idx].neighs[neighIdx] ].didx;
        DIdxArray[i]= st + i - coverNums[neighIdx];
    }
}


__global__ void computeEncodedCoarserNodesDivergence(int *DIdxArray,int coverNums,int *BaseAddressArray_d,
                                                     int *NodeIdxInFunction,
                                                     Point3D<float> *VectorField,const double *dot_F_DF,
                                                     int idx,float *divg){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int maxD=maxDepth;
    int start_D=BaseAddressArray_d[maxD];
    int res=resolution;
    int decode_offset1=(1<<(maxD+1));
    int decode_offset2=(1<<(2*(maxD+1)));

    for(int i=offset;i<coverNums;i+=stride){
        int DIdx=DIdxArray[i];
        const Point3D<float> &vo = VectorField[DIdx];

        int idxO_1[3],idxO_2[3];

        int encode_idx=NodeIdxInFunction[idx];
        idxO_1[0]=encode_idx%decode_offset1;
        idxO_1[1]=(encode_idx/decode_offset1)%decode_offset1;
        idxO_1[2]=encode_idx/decode_offset2;

        encode_idx=NodeIdxInFunction[start_D+DIdx];
        idxO_2[0]=encode_idx%decode_offset1;
        idxO_2[1]=(encode_idx/decode_offset1)%decode_offset1;
        idxO_2[2]=encode_idx/decode_offset2;

        int scratch[3];
//        scratch[0] = idxO_1[0] * res + idxO_2[0];
//        scratch[1] = idxO_1[1] * res + idxO_2[1];
//        scratch[2] = idxO_1[2] * res + idxO_2[2];
        scratch[0] = idxO_1[0] + idxO_2[0] * res;
        scratch[1] = idxO_1[1] + idxO_2[1] * res;
        scratch[2] = idxO_1[2] + idxO_2[2] * res;

        Point3D<float> uo;
        uo.coords[0]=dot_F_DF[scratch[0]];
        uo.coords[1]=dot_F_DF[scratch[1]];
        uo.coords[2]=dot_F_DF[scratch[2]];

        divg[i] = DotProduct(vo,uo);
    }
}

__device__ double GetLaplacianEntry(double *&dot_F_DF,double *&dot_F_D2F,
                                    const int (&idx)[3])
{
    double dot[3];
    dot[0]=dot_F_DF[idx[0]];
    dot[1]=dot_F_DF[idx[1]];
    dot[2]=dot_F_DF[idx[2]];
    double Entry=(
            dot_F_D2F[idx[0]]*dot[1]*dot[2]+
            dot_F_D2F[idx[1]]*dot[0]*dot[2]+
            dot_F_D2F[idx[2]]*dot[0]*dot[1]
    );
    return Entry;
}

__global__ void GenerateSingleNodeLaplacian(double *dot_F_F,double *dot_F_D2F,
                                            int *EncodedNodeIdxInFunction,OctNode *NodeArray,
                                            int left,int right,
                                            int *rowCount,int *colIdx,float *val)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int res=resolution;
    int maxD=maxDepth;
    offset+=left;
    double eps=EPSILON;

    int decode_offset1=(1<<(maxD+1));
    int decode_offset2=(1<<(2*(maxD+1)));

    for(int i=offset;i<right;i+=stride){
        int cnt=0;
        int rowIdx=i-left;
        int colStart=rowIdx * 27;

        int idxO_1[3];
        int encode_idx=EncodedNodeIdxInFunction[i];
        idxO_1[0]=encode_idx%decode_offset1;
        idxO_1[1]=(encode_idx/decode_offset1)%decode_offset1;
        idxO_1[2]=encode_idx/decode_offset2;

        for(int j=0;j<27;++j){
            int neigh=NodeArray[i].neighs[j];
            if(neigh == -1) continue;

            int colIndex=neigh-left;

            int idxO_2[3];
            encode_idx=EncodedNodeIdxInFunction[neigh];
            idxO_2[0]=encode_idx%decode_offset1;
            idxO_2[1]=(encode_idx/decode_offset1)%decode_offset1;
            idxO_2[2]=encode_idx/decode_offset2;

            int scratch[3];
            scratch[0] = idxO_1[0] * res + idxO_2[0];
            scratch[1] = idxO_1[1] * res + idxO_2[1];
            scratch[2] = idxO_1[2] * res + idxO_2[2];

            double LaplacianEntryValue= GetLaplacianEntry(dot_F_F,dot_F_D2F,scratch);
            if(LaplacianEntryValue > eps) {
                colIdx[colStart + cnt] = colIndex;
                val[colStart + cnt] = LaplacianEntryValue;
                ++cnt;
            }
        }
        rowCount[rowIdx]=cnt;
    }
}

struct validEntry{
    __device__ bool operator()(const int &x){
        return x >= 0;
    }
};



__host__ void LaplacianIteration(int *BaseAddressArray_h, int *NodeArrayCount_h, const int& nowDepth,   //host
                                 int *EncodedNodeIdxInFunction, OctNode *NodeArray, float *Divergence,//device
                                 const int &NodeArray_sz,
                                 double *dot_F_F,double *dot_F_D2F,
                                 float *&d_x)
{
    float total_time=0.0f;
    dim3 grid=(32,32);
    dim3 block(32,32);
    long long nByte;
    nByte = 1ll * sizeof(float) * NodeArray_sz;
    CHECK(hipMallocManaged((float**)&d_x,nByte));

    // run iteration for single depth nodes
    for(int i=0;i<=maxDepth_h;++i){
        printf("Depth %d Itetation...\n",i);
        int nowDepthNodesNum=NodeArrayCount_h[i];

        int *rowCount = NULL;
        nByte = 1ll * sizeof(int) * (nowDepthNodesNum+2);
        CHECK(hipMallocManaged((int**)&rowCount,nByte));
        CHECK(hipMemset(rowCount,0,nByte));

        int *colIdx = NULL;
        nByte = 1ll * sizeof(int) * nowDepthNodesNum * 27;
        CHECK(hipMallocManaged((int**)&colIdx,nByte));
        CHECK(hipMemset(colIdx,-1,nByte));

        float *val = NULL;
        nByte = 1ll * sizeof(float) * nowDepthNodesNum * 27;
        CHECK(hipMallocManaged((float**)&val,nByte));
//        CHECK(hipMemset(val,0,nByte));

        GenerateSingleNodeLaplacian<<<grid,block>>>(dot_F_F,dot_F_D2F,
                                                    EncodedNodeIdxInFunction,NodeArray,
                                                    BaseAddressArray_h[i],BaseAddressArray_h[i]+nowDepthNodesNum,
                                                    rowCount + 1,colIdx,val);
        hipDeviceSynchronize();


        thrust::device_ptr<int> rowCount_ptr=thrust::device_pointer_cast<int>(rowCount);
//        int valNums_test=thrust::reduce(rowCount_ptr + 1,rowCount_ptr+nowDepthNodesNum + 1);

        int *RowBaseAddress = NULL;
        // first address number is meaningless
        nByte = 1ll * sizeof(int) * (nowDepthNodesNum + 2);
        CHECK(hipMallocManaged((int**)&RowBaseAddress,nByte));
        thrust::device_ptr<int> RowBaseAddress_ptr=thrust::device_pointer_cast<int>(RowBaseAddress);
//        int temp=1;
//        CHECK(hipMemcpy(rowCount,&temp,sizeof(int),hipMemcpyHostToDevice));
        thrust::exclusive_scan(rowCount_ptr,rowCount_ptr+nowDepthNodesNum+1,RowBaseAddress_ptr);
        hipDeviceSynchronize();
        int valNums;
        int lastRowNum;
        CHECK(hipMemcpy(&valNums,RowBaseAddress+nowDepthNodesNum,sizeof(int),hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(&lastRowNum,rowCount+nowDepthNodesNum,sizeof(int),hipMemcpyDeviceToHost));
        valNums+=lastRowNum;
        CHECK(hipMemcpy(RowBaseAddress+nowDepthNodesNum+1,&valNums,sizeof(int),hipMemcpyHostToDevice));

//        --valNums;
//        assert(valNums == valNums_test);


        int *MergedColIdx = NULL;
        nByte = 1ll * sizeof(int) * valNums;
        CHECK(hipMallocManaged((int**)&MergedColIdx,nByte));
        thrust::device_ptr<int> colIdx_ptr=thrust::device_pointer_cast<int>(colIdx);
        thrust::device_ptr<int> MergedColIdx_ptr=thrust::device_pointer_cast<int>(MergedColIdx);

        float *MergedVal = NULL;
        nByte = 1ll * sizeof(float) * valNums;
        CHECK(hipMallocManaged((float**)&MergedVal,nByte));
        thrust::device_ptr<float> val_ptr=thrust::device_pointer_cast<float>(val);
        thrust::device_ptr<float> MergedVal_ptr=thrust::device_pointer_cast<float>(MergedVal);

        thrust::device_ptr<float> MergedVal_end=thrust::copy_if(val_ptr,val_ptr+nowDepthNodesNum*27,colIdx_ptr,MergedVal_ptr,validEntry());
        thrust::device_ptr<int> MergedColIdx_end=thrust::copy_if(colIdx,colIdx+nowDepthNodesNum*27,MergedColIdx_ptr,validEntry());

        assert(MergedVal_end-MergedVal_ptr == valNums);
        assert(MergedColIdx_end-MergedColIdx_ptr == valNums);

        printf("valNums:%d\n",valNums);
//        for(int j=0;j<valNums;++j){
//            printf("matrix:%f\n",MergedVal[j]);
//        }
//
//        for(int j=0;j<nowDepthNodesNum;++j){
//            printf("V:%f\n",Divergence[BaseAddressArray_h[i]+j]);
//        }
        total_time += solverCG_DeviceToDevice(nowDepthNodesNum,valNums,
                                              RowBaseAddress+1,
                                              MergedColIdx,
                                              MergedVal,
                                              Divergence+BaseAddressArray_h[i],
                                              d_x+BaseAddressArray_h[i]);

//        for(int j=0;j<nowDepthNodesNum;++j){
//            printf("X:%f\n",d_x[BaseAddressArray_h[i]+j]);
//        }

        hipFree(rowCount);
        hipFree(colIdx);
        hipFree(val);
        hipFree(RowBaseAddress);
        hipFree(MergedColIdx);
        hipFree(MergedVal);
    }

    printf("Pure CG solving process takes:%fms\n",total_time);
}

__global__ void calculatePointsImplicitFunctionValue(Point3D<float> *samplePoints_d,int *PointToNodeArrayD,int count,int start_D,
                                                     OctNode *NodeArray,float *d_x,
                                                     int *EncodedNodeIdxInFunction, ConfirmedPPolynomial<convTimes+1,convTimes+2> *baseFunctions_d,
                                                     float *pointValue)
{
//    printf("count:%d\n",count);
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int maxD=maxDepth;
    int decode_offset1=(1<<(maxD+1));
    int decode_offset2=(1<<(2*(maxD+1)));
    for(int i=offset;i<count;i+=stride){
        int leaveNodeIdx = start_D + PointToNodeArrayD[i];
        int nowNode = leaveNodeIdx;
        float val=0.0f;
        Point3D<float> samplePoint=samplePoints_d[i];
        while(nowNode != -1){
            for(int j=0;j<27;++j){
                int neigh = NodeArray[nowNode].neighs[j];
                if(neigh != -1){

                    int idxO[3];
                    int encode_idx=EncodedNodeIdxInFunction[neigh];
                    idxO[0]=encode_idx%decode_offset1;
                    idxO[1]=(encode_idx/decode_offset1)%decode_offset1;
                    idxO[2]=encode_idx/decode_offset2;

                    ConfirmedPPolynomial<convTimes+1,convTimes+2> funcX=baseFunctions_d[idxO[0]];
                    ConfirmedPPolynomial<convTimes+1,convTimes+2> funcY=baseFunctions_d[idxO[1]];
                    ConfirmedPPolynomial<convTimes+1,convTimes+2> funcZ=baseFunctions_d[idxO[2]];

                    val += d_x[neigh] * value(funcX,samplePoint.coords[0])
                                      * value(funcY,samplePoint.coords[1])
                                      * value(funcZ,samplePoint.coords[2]);
//                    printf("%f %f %f %f\n",d_x[neigh],value(funcX,samplePoint.coords[0]),
//                           value(funcY,samplePoint.coords[1]),
//                           value(funcZ,samplePoint.coords[2]));
//                    printf("val:%f ",val);
//                    printf("d_x:%f ",d_x[neigh]);
                }
            }
            nowNode = NodeArray[nowNode].parent;
        }
        pointValue[i]=val;
//        printf("%d: %f\n",i,val);
    }
}

// deprecated
__host__ __device__ void getNodeCenter(const int &key,Point3D<float> &myCenter){
    myCenter.coords[0]=float(0.5);
    myCenter.coords[1]=float(0.5);
    myCenter.coords[2]=float(0.5);
    float myWidth=0.25f;
    for(int i=maxDepth-1;i>=0;--i){
        if(( key >> (3 * i + 2) ) & 1)
            myCenter.coords[0] += myWidth;
        else myCenter.coords[0] -= myWidth;
        if(( key >> (3 * i + 1) ) & 1)
            myCenter.coords[1] += myWidth;
        else myCenter.coords[1] -=myWidth;
        if(( key >> (3 * i) ) & 1)
            myCenter.coords[2] += myWidth;
        else myCenter.coords[2] -=myWidth;
        myWidth/=2;
    }
}

__host__ __device__ void getNodeCenterAllDepth(const int &key,Point3D<float> &myCenter,int nowDepth){
    myCenter.coords[0]=float(0.5);
    myCenter.coords[1]=float(0.5);
    myCenter.coords[2]=float(0.5);
    float myWidth=0.25f;
    for(int i=maxDepth-1;i>=(maxDepth-nowDepth);--i){
        if(( key >> (3 * i + 2) ) & 1)
            myCenter.coords[0] += myWidth;
        else myCenter.coords[0] -= myWidth;
        if(( key >> (3 * i + 1) ) & 1)
            myCenter.coords[1] += myWidth;
        else myCenter.coords[1] -=myWidth;
        if(( key >> (3 * i) ) & 1)
            myCenter.coords[2] += myWidth;
        else myCenter.coords[2] -=myWidth;
        myWidth/=2;
    }
}


__global__ void precomputeDepthAndCenter(int *BaseAddressArray_d,OctNode *NodeArray,int NodeArray_sz,
                                         int *DepthBuffer,Point3D<float> *CenterBuffer)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<NodeArray_sz;i+=stride){
        int nowDepth = getDepth(i,BaseAddressArray_d);
        DepthBuffer[i]= nowDepth;
        Point3D<float> nowCenter;
        getNodeCenterAllDepth(NodeArray[i].key,nowCenter,nowDepth);
        CenterBuffer[i]=nowCenter;
    }
}

__global__ void initVertexOwner(OctNode *NodeArray,int left,int right,
                                VertexNode *preVertexArray,
                                int *DepthBuffer,Point3D<float> *CenterBuffer){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    int NodeOwnerKey[8];
    int NodeOwnerIdx[8];
    for(int i=offset;i<right;i+=stride){
        int depth = DepthBuffer[i];
        float halfWidth = 1.0f/(1<<(depth+1));
        float Width = 1.0f/(1<<depth);
        float Widthsq = Width * Width;
        Point3D<float> neighCenter[27];
        int neigh[27];
#pragma unroll
        for(int k=0;k<27;++k){
            neigh[k]=NodeArray[i].neighs[k];
            if(neigh[k] != -1){
                neighCenter[k]=CenterBuffer[neigh[k]];
            }
        }
        const Point3D<float> &nodeCenter = neighCenter[13];

        Point3D<float> vertexPos[8];
#pragma unroll
        for(int j=0;j<8;++j) {
            vertexPos[j].coords[0] = nodeCenter.coords[0] + (2 * (j & 1) - 1) * halfWidth;
            vertexPos[j].coords[1] = nodeCenter.coords[1] + (2 * ((j & 2) >> 1) - 1) * halfWidth;
            vertexPos[j].coords[2] = nodeCenter.coords[2] + (2 * ((j & 4) >> 2) - 1) * halfWidth;
        }

#pragma unroll
        for(int j=0;j<8;++j)
            NodeOwnerKey[j]=0x7fffffff;
        for(int j=0;j<8;++j){
            for(int k=0;k<27;++k){
                if(neigh[k] != -1 && SquareDistance(vertexPos[j],neighCenter[k]) < Widthsq){
                    int neighKey=NodeArray[neigh[k]].key;
                    if(NodeOwnerKey[j]>neighKey){
                        NodeOwnerKey[j]=neighKey;
                        NodeOwnerIdx[j]=neigh[k];
                    }
                }
            }
        }
#pragma unroll
        for(int j=0;j<8;++j) {
            if(NodeOwnerIdx[j] == i) {
                int vertexIdx = 8 * (i - left) + j;
                preVertexArray[vertexIdx].ownerNodeIdx = i;
                preVertexArray[vertexIdx].pos.coords[0] = vertexPos[j].coords[0] ;
                preVertexArray[vertexIdx].pos.coords[1] = vertexPos[j].coords[1] ;
                preVertexArray[vertexIdx].pos.coords[2] = vertexPos[j].coords[2] ;
                preVertexArray[vertexIdx].vertexKind = j;
                preVertexArray[vertexIdx].depth = depth;
            }
        }
    }
}

// only process vertex at maxDepth
__global__ void initSubdivideVertexOwner(int NodeArray_sz,
                                         OctNode *SubdivideArray,int left,int right,
                                         VertexNode *SubdividePreVertexArray,
                                         Point3D<float> *SubdivideArrayCenterBuffer){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    int NodeOwnerKey[8];
    int NodeOwnerIdx[8];
    for(int i=offset;i<right;i+=stride){
        int depth = maxDepth;
        float halfWidth = 1.0f/(1<<(depth+1));
        float Width = 1.0f/(1<<depth);
        float Widthsq = Width * Width;
        Point3D<float> neighCenter[27];
        int neigh[27];
#pragma unroll
        for(int k=0;k<27;++k){
            neigh[k]=SubdivideArray[i].neighs[k];
            if(neigh[k] != -1 && neigh[k] >= NodeArray_sz){
                neighCenter[k] = SubdivideArrayCenterBuffer[neigh[k] - NodeArray_sz];
            }
        }
        const Point3D<float> &nodeCenter = neighCenter[13];

        Point3D<float> vertexPos[8];
#pragma unroll
        for(int j=0;j<8;++j) {
            vertexPos[j].coords[0] = nodeCenter.coords[0] + (2 * (j & 1) - 1) * halfWidth;
            vertexPos[j].coords[1] = nodeCenter.coords[1] + (2 * ((j & 2) >> 1) - 1) * halfWidth;
            vertexPos[j].coords[2] = nodeCenter.coords[2] + (2 * ((j & 4) >> 2) - 1) * halfWidth;
        }

#pragma unroll
        for(int j=0;j<8;++j)
            NodeOwnerKey[j]=0x7fffffff;
        for(int j=0;j<8;++j){
            for(int k=0;k<27;++k){
                if(neigh[k] != -1 && SquareDistance(vertexPos[j],neighCenter[k]) < Widthsq){
                    int neighKey;
                    if(neigh[k] < NodeArray_sz) continue;
                    else
                        neighKey=SubdivideArray[neigh[k]-NodeArray_sz].key;
                    if(NodeOwnerKey[j]>neighKey){
                        NodeOwnerKey[j]=neighKey;
                        NodeOwnerIdx[j]=neigh[k];
                    }
                }
            }
        }
#pragma unroll
        for(int j=0;j<8;++j) {
            if(NodeOwnerIdx[j] == NodeArray_sz + i) {
                int vertexIdx = 8 * (i - left) + j;
                SubdividePreVertexArray[vertexIdx].ownerNodeIdx = NodeOwnerIdx[j];
                SubdividePreVertexArray[vertexIdx].pos.coords[0] = vertexPos[j].coords[0] ;
                SubdividePreVertexArray[vertexIdx].pos.coords[1] = vertexPos[j].coords[1] ;
                SubdividePreVertexArray[vertexIdx].pos.coords[2] = vertexPos[j].coords[2] ;
                SubdividePreVertexArray[vertexIdx].vertexKind = j;
                SubdividePreVertexArray[vertexIdx].depth = depth;
            }
        }
    }
}

// only process vertex at maxDepth
__global__ void initSubdivideVertexOwner(int NodeArray_sz,
                                         EasyOctNode *SubdivideArray,int left,int right,
                                         VertexNode *SubdividePreVertexArray,
                                         Point3D<float> *SubdivideArrayCenterBuffer){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    int NodeOwnerKey[8];
    int NodeOwnerIdx[8];
    for(int i=offset;i<right;i+=stride){
        int depth = maxDepth;
        float halfWidth = 1.0f/(1<<(depth+1));
        float Width = 1.0f/(1<<depth);
        float Widthsq = Width * Width;
        Point3D<float> neighCenter[27];
        int neigh[27];
#pragma unroll
        for(int k=0;k<27;++k){
            neigh[k]=SubdivideArray[i].neighs[k];
            if(neigh[k] != -1 && neigh[k] >= NodeArray_sz){
                neighCenter[k] = SubdivideArrayCenterBuffer[neigh[k] - NodeArray_sz];
            }
        }
        const Point3D<float> &nodeCenter = neighCenter[13];

        Point3D<float> vertexPos[8];
#pragma unroll
        for(int j=0;j<8;++j) {
            vertexPos[j].coords[0] = nodeCenter.coords[0] + (2 * (j & 1) - 1) * halfWidth;
            vertexPos[j].coords[1] = nodeCenter.coords[1] + (2 * ((j & 2) >> 1) - 1) * halfWidth;
            vertexPos[j].coords[2] = nodeCenter.coords[2] + (2 * ((j & 4) >> 2) - 1) * halfWidth;
        }

#pragma unroll
        for(int j=0;j<8;++j)
            NodeOwnerKey[j]=0x7fffffff;
        for(int j=0;j<8;++j){
            for(int k=0;k<27;++k){
                if(neigh[k] != -1 && SquareDistance(vertexPos[j],neighCenter[k]) < Widthsq){
                    int neighKey;
                    if(neigh[k] < NodeArray_sz) continue;
                    else
                        neighKey=SubdivideArray[neigh[k]-NodeArray_sz].key;
                    if(NodeOwnerKey[j]>neighKey){
                        NodeOwnerKey[j]=neighKey;
                        NodeOwnerIdx[j]=neigh[k];
                    }
                }
            }
        }
#pragma unroll
        for(int j=0;j<8;++j) {
            if(NodeOwnerIdx[j] == NodeArray_sz + i) {
                int vertexIdx = 8 * (i - left) + j;
                SubdividePreVertexArray[vertexIdx].ownerNodeIdx = NodeOwnerIdx[j];
                SubdividePreVertexArray[vertexIdx].pos.coords[0] = vertexPos[j].coords[0] ;
                SubdividePreVertexArray[vertexIdx].pos.coords[1] = vertexPos[j].coords[1] ;
                SubdividePreVertexArray[vertexIdx].pos.coords[2] = vertexPos[j].coords[2] ;
                SubdividePreVertexArray[vertexIdx].vertexKind = j;
                SubdividePreVertexArray[vertexIdx].depth = depth;
            }
        }
    }
}

struct validVertex{
    __device__ bool operator()(const VertexNode &x){
        return x.ownerNodeIdx > 0;
    }
};

__global__ void maintainVertexNodePointerNonAtomic(VertexNode *VertexArray,int VertexArray_sz,
                                                   OctNode *NodeArray,
                                                   int *DepthBuffer,Point3D<float> *CenterBuffer){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<VertexArray_sz;i+=stride){
        int owner=VertexArray[i].ownerNodeIdx;
        int depth = DepthBuffer[owner];
        float halfWidth = 1.0f/(1<<(depth+1));
        float Width = 1.0f/(1<<depth);
        float Widthsq = Width * Width;
        Point3D<float> neighCenter[27];
        Point3D<float> vertexPos=VertexArray[i].pos;

        int neigh[27];
        for(int k=0;k<27;++k){
            neigh[k]=NodeArray[owner].neighs[k];
            if(neigh[k] != -1){
                neighCenter[k]=CenterBuffer[neigh[k]];
            }
        }
        int cnt=0;
        for(int k=0;k<27;++k){
            if(neigh[k] != -1 && SquareDistance(vertexPos,neighCenter[k]) < Widthsq){
                VertexArray[i].nodes[cnt]=neigh[k];
                ++cnt;
                int idx=0;
                if(neighCenter[k].coords[0]-vertexPos.coords[0] < 0) idx|=1;
                if(neighCenter[k].coords[2]-vertexPos.coords[2] < 0) idx|=4;
                if(neighCenter[k].coords[1]-vertexPos.coords[1] < 0) {
                    if(idx & 1){
                        idx+=1;
                    }else{
                        idx+=3;
                    }
                }
                NodeArray[neigh[k]].vertices[idx] = i+1;
            }
        }
    }
}

// process only the maxDepth subdivide node
__global__ void maintainSubdivideVertexNodePointer(VertexNode *VertexArray,int VertexArray_sz,
                                                   int NodeArray_sz,
                                                   OctNode *SubdivideArray,
                                                   Point3D<float> *CenterBuffer,
                                                   Point3D<float> *SubdivideArrayCenterBuffer){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<VertexArray_sz;i+=stride){
        int owner=VertexArray[i].ownerNodeIdx;
        int depth = maxDepth;
        float halfWidth = 1.0f/(1<<(depth+1));
        float Width = 1.0f/(1<<depth);
        float Widthsq = Width * Width;
        Point3D<float> neighCenter[27];
        Point3D<float> vertexPos=VertexArray[i].pos;

        int neigh[27];
        for (int k = 0; k < 27; ++k) {
            neigh[k]=SubdivideArray[owner-NodeArray_sz].neighs[k];
        }
        for(int k=0;k<27;++k){
            if(neigh[k] != -1){
                if(neigh[k] < NodeArray_sz) {
                    neighCenter[k] = CenterBuffer[neigh[k]];
                } else{
                    neighCenter[k] = SubdivideArrayCenterBuffer[neigh[k] - NodeArray_sz];
                }
            }
        }
        int cnt=0;
        for(int k=0;k<27;++k){
            if(neigh[k] != -1 && SquareDistance(vertexPos,neighCenter[k]) < Widthsq){
                VertexArray[i].nodes[cnt]=neigh[k];
                ++cnt;
                int idx=0;
                if(neighCenter[k].coords[0]-vertexPos.coords[0] < 0) idx|=1;
                if(neighCenter[k].coords[2]-vertexPos.coords[2] < 0) idx|=4;
                if(neighCenter[k].coords[1]-vertexPos.coords[1] < 0) {
                    if(idx & 1){
                        idx+=1;
                    }else{
                        idx+=3;
                    }
                }
                if(neigh[k] >= NodeArray_sz) {
                    if (SubdivideArray[neigh[k] - NodeArray_sz].vertices[idx] != 0) {
                        printf("idx:%d\n%d %d\n%f %f\n%f %f\n%f %f\nrelationship error\n",idx,SubdivideArray[neigh[k] - NodeArray_sz].vertices[idx],i+1,neighCenter[k].coords[0],
                               vertexPos.coords[0],neighCenter[k].coords[1],vertexPos.coords[1],
                               neighCenter[k].coords[2],vertexPos.coords[2]);
                    }
                    SubdivideArray[neigh[k] - NodeArray_sz].vertices[idx] = i + 1;
                }
            }
        }
    }
}

// process only the maxDepth subdivide node
__global__ void maintainSubdivideVertexNodePointer(VertexNode *VertexArray,int VertexArray_sz,
                                                   int NodeArray_sz,
                                                   EasyOctNode *SubdivideArray,
                                                   Point3D<float> *CenterBuffer,
                                                   Point3D<float> *SubdivideArrayCenterBuffer){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<VertexArray_sz;i+=stride){
        int owner=VertexArray[i].ownerNodeIdx;
        int depth = maxDepth;
        float halfWidth = 1.0f/(1<<(depth+1));
        float Width = 1.0f/(1<<depth);
        float Widthsq = Width * Width;
        Point3D<float> neighCenter[27];
        Point3D<float> vertexPos=VertexArray[i].pos;

        int neigh[27];
        for (int k = 0; k < 27; ++k) {
            neigh[k]=SubdivideArray[owner-NodeArray_sz].neighs[k];
        }
        for(int k=0;k<27;++k){
            if(neigh[k] != -1){
                if(neigh[k] < NodeArray_sz) {
                    neighCenter[k] = CenterBuffer[neigh[k]];
                } else{
                    neighCenter[k] = SubdivideArrayCenterBuffer[neigh[k] - NodeArray_sz];
                }
            }
        }
        int cnt=0;
        for(int k=0;k<27;++k){
            if(neigh[k] != -1 && SquareDistance(vertexPos,neighCenter[k]) < Widthsq){
                VertexArray[i].nodes[cnt]=neigh[k];
                ++cnt;
                int idx=0;
                if(neighCenter[k].coords[0]-vertexPos.coords[0] < 0) idx|=1;
                if(neighCenter[k].coords[2]-vertexPos.coords[2] < 0) idx|=4;
                if(neighCenter[k].coords[1]-vertexPos.coords[1] < 0) {
                    if(idx & 1){
                        idx+=1;
                    }else{
                        idx+=3;
                    }
                }
                if(neigh[k] >= NodeArray_sz)
                    SubdivideArray[neigh[k] - NodeArray_sz].vertices[idx] = i + 1;
            }
        }
    }
}

__global__ void initEdgeArray(OctNode *NodeArray,int left,int right,
                              EdgeNode *preEdgeArray,
                              int *DepthBuffer,Point3D<float> *CenterBuffer){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    int NodeOwnerKey[12];
    int NodeOwnerIdx[12];
    for(int i=offset;i<right;i+=stride){
        int depth = DepthBuffer[i];
        float halfWidth = 1.0f/(1<<(depth+1));
        float Width = 1.0f/(1<<depth);
        float Widthsq = Width * Width;
        Point3D<float> neighCenter[27];
        int neigh[27];
#pragma unroll
        for(int k=0;k<27;++k){
            neigh[k]=NodeArray[i].neighs[k];
            if(neigh[k] != -1){
                neighCenter[k]=CenterBuffer[neigh[k]];
            }
        }
        const Point3D<float> &nodeCenter = neighCenter[13];
        Point3D<float> edgeCenterPos[12];
        int orientation[12];
        int off[24];
#pragma unroll
        for(int j=0;j<12;++j) {
            orientation[j] = j>>2 ;
            off[2*j] = j&1;
            off[2*j+1] = (j&2)>>1;
            int multi[3];
            int dim=2*j;
            for(int k=0;k<3;++k){
                if(orientation[j]==k){
                    multi[k]=0;
                }else{
                    multi[k]=(2 * off[dim] - 1);
                    ++dim;
                }
            }
            edgeCenterPos[j].coords[0] = nodeCenter.coords[0] + multi[0] * halfWidth;
            edgeCenterPos[j].coords[1] = nodeCenter.coords[1] + multi[1] * halfWidth;
            edgeCenterPos[j].coords[2] = nodeCenter.coords[2] + multi[2] * halfWidth;
        }

#pragma unroll
        for(int j=0;j<12;++j)
            NodeOwnerKey[j]=0x7fffffff;
        for(int j=0;j<12;++j){
            for(int k=0;k<27;++k){
                if(neigh[k] != -1 && SquareDistance(edgeCenterPos[j],neighCenter[k]) < Widthsq){
                    int neighKey=NodeArray[neigh[k]].key;
                    if(NodeOwnerKey[j]>neighKey){
                        NodeOwnerKey[j]=neighKey;
                        NodeOwnerIdx[j]=neigh[k];
                    }
                }
            }
        }
#pragma unroll
        for(int j=0;j<12;++j) {
            if(NodeOwnerIdx[j] == i) {
                int edgeIdx = 12 * (i - left) + j;
                preEdgeArray[edgeIdx].ownerNodeIdx = i;
                preEdgeArray[edgeIdx].edgeKind = j;
            }
        }
    }
}

// only use for node at maxDepth
__global__ void initSubdivideEdgeArray(OctNode *SubdivideArray,int left,int right,
                                       int NodeArray_sz,
                                       EdgeNode *SubdividePreEdgeArray,
                                       Point3D<float> *SubdivideArrayCenterBuffer){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    int NodeOwnerKey[12];
    int NodeOwnerIdx[12];
    for(int i=offset;i<right;i+=stride){
        int depth = maxDepth;
        float halfWidth = 1.0f/(1<<(depth+1));
        float Width = 1.0f/(1<<depth);
        float Widthsq = Width * Width;
        Point3D<float> neighCenter[27];
        int neigh[27];
#pragma unroll
        for(int k=0;k<27;++k){
            neigh[k]=SubdivideArray[i].neighs[k];
            if(neigh[k] != -1 && neigh[k] >= NodeArray_sz){
                neighCenter[k]=SubdivideArrayCenterBuffer[neigh[k] - NodeArray_sz];
            }
        }
        const Point3D<float> &nodeCenter = neighCenter[13];
        Point3D<float> edgeCenterPos[12];
        int orientation[12];
        int off[24];
#pragma unroll
        for(int j=0;j<12;++j) {
            orientation[j] = j>>2 ;
            off[2*j] = j&1;
            off[2*j+1] = (j&2)>>1;
            int multi[3];
            int dim=2*j;
            for(int k=0;k<3;++k){
                if(orientation[j]==k){
                    multi[k]=0;
                }else{
                    multi[k]=(2 * off[dim] - 1);
                    ++dim;
                }
            }
            edgeCenterPos[j].coords[0] = nodeCenter.coords[0] + multi[0] * halfWidth;
            edgeCenterPos[j].coords[1] = nodeCenter.coords[1] + multi[1] * halfWidth;
            edgeCenterPos[j].coords[2] = nodeCenter.coords[2] + multi[2] * halfWidth;
        }

#pragma unroll
        for(int j=0;j<12;++j)
            NodeOwnerKey[j]=0x7fffffff;
        for(int j=0;j<12;++j){
            for(int k=0;k<27;++k){
                if(neigh[k] != -1 && SquareDistance(edgeCenterPos[j],neighCenter[k]) < Widthsq){
                    int neighKey;
                    if(neigh[k] < NodeArray_sz) continue;
                    else
                        neighKey=SubdivideArray[neigh[k] - NodeArray_sz].key;
                    if(NodeOwnerKey[j]>neighKey){
                        NodeOwnerKey[j]=neighKey;
                        NodeOwnerIdx[j]=neigh[k];
                    }
                }
            }
        }
#pragma unroll
        for(int j=0;j<12;++j) {
            if(NodeOwnerIdx[j] == i + NodeArray_sz) {
                int edgeIdx = 12 * (i - left) + j;
                SubdividePreEdgeArray[edgeIdx].ownerNodeIdx = NodeOwnerIdx[j];
                SubdividePreEdgeArray[edgeIdx].edgeKind = j;
            }
        }
    }
}

// only use for node at maxDepth
__global__ void initSubdivideEdgeArray(EasyOctNode *SubdivideArray,int left,int right,
                                       int NodeArray_sz,
                                       EdgeNode *SubdividePreEdgeArray,
                                       Point3D<float> *SubdivideArrayCenterBuffer){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    int NodeOwnerKey[12];
    int NodeOwnerIdx[12];
    for(int i=offset;i<right;i+=stride){
        int depth = maxDepth;
        float halfWidth = 1.0f/(1<<(depth+1));
        float Width = 1.0f/(1<<depth);
        float Widthsq = Width * Width;
        Point3D<float> neighCenter[27];
        int neigh[27];
#pragma unroll
        for(int k=0;k<27;++k){
            neigh[k]=SubdivideArray[i].neighs[k];
            if(neigh[k] != -1 && neigh[k] >= NodeArray_sz){
                neighCenter[k]=SubdivideArrayCenterBuffer[neigh[k] - NodeArray_sz];
            }
        }
        const Point3D<float> &nodeCenter = neighCenter[13];
        Point3D<float> edgeCenterPos[12];
        int orientation[12];
        int off[24];
#pragma unroll
        for(int j=0;j<12;++j) {
            orientation[j] = j>>2 ;
            off[2*j] = j&1;
            off[2*j+1] = (j&2)>>1;
            int multi[3];
            int dim=2*j;
            for(int k=0;k<3;++k){
                if(orientation[j]==k){
                    multi[k]=0;
                }else{
                    multi[k]=(2 * off[dim] - 1);
                    ++dim;
                }
            }
            edgeCenterPos[j].coords[0] = nodeCenter.coords[0] + multi[0] * halfWidth;
            edgeCenterPos[j].coords[1] = nodeCenter.coords[1] + multi[1] * halfWidth;
            edgeCenterPos[j].coords[2] = nodeCenter.coords[2] + multi[2] * halfWidth;
        }

#pragma unroll
        for(int j=0;j<12;++j)
            NodeOwnerKey[j]=0x7fffffff;
        for(int j=0;j<12;++j){
            for(int k=0;k<27;++k){
                if(neigh[k] != -1 && SquareDistance(edgeCenterPos[j],neighCenter[k]) < Widthsq){
                    int neighKey;
                    if(neigh[k] < NodeArray_sz) continue;
                    else
                        neighKey=SubdivideArray[neigh[k] - NodeArray_sz].key;
                    if(NodeOwnerKey[j]>neighKey){
                        NodeOwnerKey[j]=neighKey;
                        NodeOwnerIdx[j]=neigh[k];
                    }
                }
            }
        }
#pragma unroll
        for(int j=0;j<12;++j) {
            if(NodeOwnerIdx[j] == i + NodeArray_sz) {
                int edgeIdx = 12 * (i - left) + j;
                SubdividePreEdgeArray[edgeIdx].ownerNodeIdx = NodeOwnerIdx[j];
                SubdividePreEdgeArray[edgeIdx].edgeKind = j;
            }
        }
    }
}

__global__ void maintainEdgeNodePointer(EdgeNode *EdgeArray,int EdgeArray_sz,
                                        OctNode *NodeArray,
                                        int *DepthBuffer,Point3D<float> *CenterBuffer){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<EdgeArray_sz;i+=stride){
        EdgeNode nowEdge = EdgeArray[i];
        int owner = nowEdge.ownerNodeIdx;

        int depth = DepthBuffer[owner];
        float halfWidth = 1.0f/(1<<(depth+1));
        float Width = 1.0f/(1<<depth);
        float Widthsq = Width * Width;

        Point3D<float> neighCenter[27];
        int neigh[27];
        for(int k=0;k<27;++k){
            neigh[k]=NodeArray[owner].neighs[k];
            if(neigh[k] != -1){
                neighCenter[k]=CenterBuffer[neigh[k]];
            }
        }

        const Point3D<float> &nodeCenter = neighCenter[13];
        Point3D<float> edgeCenterPos;
        int multi[3];
        int dim=0;
        int orientation = nowEdge.edgeKind>>2;
        int off[2];
        off[0] = nowEdge.edgeKind & 1;
        off[1] = (nowEdge.edgeKind & 2)>>1;
        for(int k=0;k<3;++k){
            if(orientation==k){
                multi[k]=0;
            }else{
                multi[k]=(2 * off[dim] - 1);
                ++dim;
            }
        }
        edgeCenterPos.coords[0] = nodeCenter.coords[0] + multi[0] * halfWidth;
        edgeCenterPos.coords[1] = nodeCenter.coords[1] + multi[1] * halfWidth;
        edgeCenterPos.coords[2] = nodeCenter.coords[2] + multi[2] * halfWidth;

        int cnt=0;
        for(int k=0;k<27;++k){
            if(neigh[k] != -1 && SquareDistance(edgeCenterPos,neighCenter[k]) < Widthsq){
                EdgeArray[i].nodes[cnt] = neigh[k];
                ++cnt;
                int idx=orientation<<2;
                int dim=0;
                for(int j=0;j<3;++j){
                    if(orientation!=j){
                        if(neighCenter[k].coords[j]-edgeCenterPos.coords[j] < 0) idx |= (1<<dim);
                        ++dim;
                    }
                }
                NodeArray[neigh[k]].edges[idx] = i+1;
            }
        }
    }
}

// only use for edge at maxDepth
__global__ void maintainSubdivideEdgeNodePointer(EdgeNode *EdgeArray,int EdgeArray_sz,
                                                 int NodeArray_sz,
                                                 OctNode *SubdivideArray,
                                                 Point3D<float> *CenterBuffer,
                                                 Point3D<float> *SubdivideArrayCenterBuffer){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<EdgeArray_sz;i+=stride){
        EdgeNode nowEdge = EdgeArray[i];
        int owner = nowEdge.ownerNodeIdx;

        int depth = maxDepth;
        float halfWidth = 1.0f/(1<<(depth+1));
        float Width = 1.0f/(1<<depth);
        float Widthsq = Width * Width;

        Point3D<float> neighCenter[27];
        int neigh[27];
        for(int k=0;k<27;++k){
            neigh[k]=SubdivideArray[owner - NodeArray_sz].neighs[k];
            if(neigh[k] != -1){
                if(neigh[k] < NodeArray_sz)
                    neighCenter[k]=CenterBuffer[neigh[k]];
                else
                    neighCenter[k]=SubdivideArrayCenterBuffer[neigh[k] - NodeArray_sz];
            }
        }

        const Point3D<float> &nodeCenter = neighCenter[13];
        Point3D<float> edgeCenterPos;
        int multi[3];
        int dim=0;
        int orientation = nowEdge.edgeKind>>2;
        int off[2];
        off[0] = nowEdge.edgeKind & 1;
        off[1] = (nowEdge.edgeKind & 2)>>1;
        for(int k=0;k<3;++k){
            if(orientation==k){
                multi[k]=0;
            }else{
                multi[k]=(2 * off[dim] - 1);
                ++dim;
            }
        }
        edgeCenterPos.coords[0] = nodeCenter.coords[0] + multi[0] * halfWidth;
        edgeCenterPos.coords[1] = nodeCenter.coords[1] + multi[1] * halfWidth;
        edgeCenterPos.coords[2] = nodeCenter.coords[2] + multi[2] * halfWidth;

        int cnt=0;
        for(int k=0;k<27;++k){
            if(neigh[k] != -1 && SquareDistance(edgeCenterPos,neighCenter[k]) < Widthsq){
                EdgeArray[i].nodes[cnt] = neigh[k];
                ++cnt;
                int idx=orientation<<2;
                int dim=0;
                for(int j=0;j<3;++j){
                    if(orientation!=j){
                        if(neighCenter[k].coords[j]-edgeCenterPos.coords[j] < 0) idx |= (1<<dim);
                        ++dim;
                    }
                }
                if(neigh[k] >= NodeArray_sz)
                    SubdivideArray[neigh[k] - NodeArray_sz].edges[idx] = i+1;
            }
        }
    }
}

// only use for edge at maxDepth
__global__ void maintainSubdivideEdgeNodePointer(EdgeNode *EdgeArray,int EdgeArray_sz,
                                                 int NodeArray_sz,
                                                 EasyOctNode *SubdivideArray,
                                                 Point3D<float> *CenterBuffer,
                                                 Point3D<float> *SubdivideArrayCenterBuffer){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<EdgeArray_sz;i+=stride){
        EdgeNode nowEdge = EdgeArray[i];
        int owner = nowEdge.ownerNodeIdx;

        int depth = maxDepth;
        float halfWidth = 1.0f/(1<<(depth+1));
        float Width = 1.0f/(1<<depth);
        float Widthsq = Width * Width;

        Point3D<float> neighCenter[27];
        int neigh[27];
        for(int k=0;k<27;++k){
            neigh[k]=SubdivideArray[owner - NodeArray_sz].neighs[k];
            if(neigh[k] != -1){
                if(neigh[k] < NodeArray_sz)
                    neighCenter[k]=CenterBuffer[neigh[k]];
                else
                    neighCenter[k]=SubdivideArrayCenterBuffer[neigh[k] - NodeArray_sz];
            }
        }

        const Point3D<float> &nodeCenter = neighCenter[13];
        Point3D<float> edgeCenterPos;
        int multi[3];
        int dim=0;
        int orientation = nowEdge.edgeKind>>2;
        int off[2];
        off[0] = nowEdge.edgeKind & 1;
        off[1] = (nowEdge.edgeKind & 2)>>1;
        for(int k=0;k<3;++k){
            if(orientation==k){
                multi[k]=0;
            }else{
                multi[k]=(2 * off[dim] - 1);
                ++dim;
            }
        }
        edgeCenterPos.coords[0] = nodeCenter.coords[0] + multi[0] * halfWidth;
        edgeCenterPos.coords[1] = nodeCenter.coords[1] + multi[1] * halfWidth;
        edgeCenterPos.coords[2] = nodeCenter.coords[2] + multi[2] * halfWidth;

        int cnt=0;
        for(int k=0;k<27;++k){
            if(neigh[k] != -1 && SquareDistance(edgeCenterPos,neighCenter[k]) < Widthsq){
                EdgeArray[i].nodes[cnt] = neigh[k];
                ++cnt;
                int idx=orientation<<2;
                int dim=0;
                for(int j=0;j<3;++j){
                    if(orientation!=j){
                        if(neighCenter[k].coords[j]-edgeCenterPos.coords[j] < 0) idx |= (1<<dim);
                        ++dim;
                    }
                }
                if(neigh[k] >= NodeArray_sz)
                    SubdivideArray[neigh[k] - NodeArray_sz].edges[idx] = i+1;
            }
        }
    }
}


struct validEdge{
    __device__ bool operator()(const EdgeNode &x){
        return x.ownerNodeIdx > 0;
    }
};

__device__ void pushStack(int *stack,int &top,const int &val){
    if( top == stackCapacity){
        printf("Stack full, error\n");
//        for(int i=stackCapacity-10;i<stackCapacity;++i){
//            printf("%d\n",stack[i]);
//        }
    }else{
        stack[top]=val;
        ++top;
    }
}

__device__ int popStack(int *stack,int &top){
    if(top==0){
        printf("Stack empty,error.\n");
    }
    --top;
    return stack[top];
}

__device__ int findStack(int *stack,int &top,const int &val){
    for(int i=0;i<top;++i){
        if(stack[i]==val)
            return 1;
    }
    return 0;
}

__global__ void computeVertexImplicitFunctionValue(VertexNode *VertexArray,int VertexArray_sz,
                                                   OctNode *NodeArray,float *d_x,
                                                   int *EncodedNodeIdxInFunction,ConfirmedPPolynomial<convTimes+1,convTimes+2> *baseFunctions_d,
                                                   float *vvalue,float isoValue)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int maxD=maxDepth;
    int decode_offset1=(1<<(maxD+1));
    int decode_offset2=(1<<(2*(maxD+1)));
    for(int i=offset;i<VertexArray_sz;i+=stride){
        VertexNode nowVertex = VertexArray[i];
        int depth = nowVertex.depth;
        float val=0.0f;
        int exceedChildrenId = childrenVertexKind[nowVertex.vertexKind];
        int nowNode = nowVertex.ownerNodeIdx;
        if(nowNode>0){
            while(nowNode != -1){
                for(int k=0;k<27;++k){
                    int neigh = NodeArray[nowNode].neighs[k];
                    if(neigh != -1){
                        int idxO[3];
                        int encode_idx=EncodedNodeIdxInFunction[neigh];
                        idxO[0]=encode_idx%decode_offset1;
                        idxO[1]=(encode_idx/decode_offset1)%decode_offset1;
                        idxO[2]=encode_idx/decode_offset2;

                        ConfirmedPPolynomial<convTimes+1,convTimes+2> funcX=baseFunctions_d[idxO[0]];
                        ConfirmedPPolynomial<convTimes+1,convTimes+2> funcY=baseFunctions_d[idxO[1]];
                        ConfirmedPPolynomial<convTimes+1,convTimes+2> funcZ=baseFunctions_d[idxO[2]];

                        val += d_x[neigh] * value(funcX,nowVertex.pos.coords[0])
                               * value(funcY,nowVertex.pos.coords[1])
                               * value(funcZ,nowVertex.pos.coords[2]);

                    }
                }
                nowNode = NodeArray[nowNode].parent;
            }
            nowNode = nowVertex.ownerNodeIdx;
            while(depth < maxDepth){
                ++depth;
                nowNode = NodeArray[nowNode].children[exceedChildrenId];
                if(nowNode == -1) break;
                for(int k=0;k<27;++k){
                    int neigh = NodeArray[nowNode].neighs[k];
                    if(neigh != -1){
                        int idxO[3];
                        int encode_idx=EncodedNodeIdxInFunction[neigh];
                        idxO[0]=encode_idx%decode_offset1;
                        idxO[1]=(encode_idx/decode_offset1)%decode_offset1;
                        idxO[2]=encode_idx/decode_offset2;

                        ConfirmedPPolynomial<convTimes+1,convTimes+2> funcX=baseFunctions_d[idxO[0]];
                        ConfirmedPPolynomial<convTimes+1,convTimes+2> funcY=baseFunctions_d[idxO[1]];
                        ConfirmedPPolynomial<convTimes+1,convTimes+2> funcZ=baseFunctions_d[idxO[2]];

                        val += d_x[neigh] * value(funcX,nowVertex.pos.coords[0])
                               * value(funcY,nowVertex.pos.coords[1])
                               * value(funcZ,nowVertex.pos.coords[2]);
                    }
                }
            }
        }
        vvalue[i]=val-isoValue;
    }
}

__global__ void computeSubdivideVertexImplicitFunctionValue(VertexNode *SubdivideVertexArray,int SubdivideVertexArray_sz,
                                                            EasyOctNode *SubdivideArray, int rootId,
                                                            OctNode *NodeArray,int NodeArray_sz,
                                                            float *d_x, int *EncodedNodeIdxInFunction,
                                                            ConfirmedPPolynomial<convTimes+1,convTimes+2> *baseFunctions_d,
                                                            float *SubdivideVvalue,float isoValue)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int maxD=maxDepth;
    int decode_offset1=(1<<(maxD+1));
    int decode_offset2=(1<<(2*(maxD+1)));
    for(int i=offset;i<SubdivideVertexArray_sz;i+=stride){
        VertexNode nowVertex = SubdivideVertexArray[i];
        int depth = maxDepth;
        float val=0.0f;
        int nowNode = nowVertex.ownerNodeIdx;
        if(nowNode>0){
            while(nowNode != -1){
                for(int k=0;k<27;++k){
                    int neigh;
                    if(nowNode < NodeArray_sz)
                        neigh = NodeArray[nowNode].neighs[k];
                    else
                        neigh = SubdivideArray[nowNode - NodeArray_sz].neighs[k];
                    if(neigh != -1){
                        if(neigh == NodeArray_sz)
                            neigh = rootId;
                        int idxO[3];
                        int encode_idx;
                        if(neigh < NodeArray_sz)
                            encode_idx=EncodedNodeIdxInFunction[neigh];
                        else continue;  // d_x = 0 in Subdivide space
                        idxO[0]=encode_idx%decode_offset1;
                        idxO[1]=(encode_idx/decode_offset1)%decode_offset1;
                        idxO[2]=encode_idx/decode_offset2;

                        ConfirmedPPolynomial<convTimes+1,convTimes+2> funcX=baseFunctions_d[idxO[0]];
                        ConfirmedPPolynomial<convTimes+1,convTimes+2> funcY=baseFunctions_d[idxO[1]];
                        ConfirmedPPolynomial<convTimes+1,convTimes+2> funcZ=baseFunctions_d[idxO[2]];

                        val += d_x[neigh] * value(funcX,nowVertex.pos.coords[0])
                               * value(funcY,nowVertex.pos.coords[1])
                               * value(funcZ,nowVertex.pos.coords[2]);

                    }
                }
                if(nowNode < NodeArray_sz)
                    nowNode = NodeArray[nowNode].parent;
                else
                    nowNode = SubdivideArray[nowNode - NodeArray_sz].parent;
            }
        }
        SubdivideVvalue[i]=val-isoValue;
    }
}

__global__ void computeSubdivideVertexImplicitFunctionValue(VertexNode *SubdivideVertexArray,int SubdivideVertexArray_sz,
                                                            EasyOctNode *SubdivideArray, int *ReplacedNodeId,int *IsRoot,
                                                            OctNode *NodeArray,int NodeArray_sz,
                                                            float *d_x, int *EncodedNodeIdxInFunction,
                                                            ConfirmedPPolynomial<convTimes+1,convTimes+2> *baseFunctions_d,
                                                            float *SubdivideVvalue,float isoValue)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int maxD=maxDepth;
    int decode_offset1=(1<<(maxD+1));
    int decode_offset2=(1<<(2*(maxD+1)));
    for(int i=offset;i<SubdivideVertexArray_sz;i+=stride){
        VertexNode nowVertex = SubdivideVertexArray[i];
        int depth = maxDepth;
        float val=0.0f;
        int nowNode = nowVertex.ownerNodeIdx;
        if(nowNode>0){
            while(nowNode != -1){
                for(int k=0;k<27;++k){
                    int neigh;
                    if(nowNode < NodeArray_sz)
                        neigh = NodeArray[nowNode].neighs[k];
                    else
                        neigh = SubdivideArray[nowNode - NodeArray_sz].neighs[k];
                    if(neigh != -1){
                        if(neigh >= NodeArray_sz && IsRoot[neigh - NodeArray_sz])
                            neigh = ReplacedNodeId[neigh - NodeArray_sz];
                        int idxO[3];
                        int encode_idx;
                        if(neigh < NodeArray_sz)
                            encode_idx=EncodedNodeIdxInFunction[neigh];
                        else continue;  // d_x = 0 in Subdivide space
                        idxO[0]=encode_idx%decode_offset1;
                        idxO[1]=(encode_idx/decode_offset1)%decode_offset1;
                        idxO[2]=encode_idx/decode_offset2;

                        ConfirmedPPolynomial<convTimes+1,convTimes+2> funcX=baseFunctions_d[idxO[0]];
                        ConfirmedPPolynomial<convTimes+1,convTimes+2> funcY=baseFunctions_d[idxO[1]];
                        ConfirmedPPolynomial<convTimes+1,convTimes+2> funcZ=baseFunctions_d[idxO[2]];

                        val += d_x[neigh] * value(funcX,nowVertex.pos.coords[0])
                               * value(funcY,nowVertex.pos.coords[1])
                               * value(funcZ,nowVertex.pos.coords[2]);

                    }
                }
                if(nowNode < NodeArray_sz)
                    nowNode = NodeArray[nowNode].parent;
                else
                    nowNode = SubdivideArray[nowNode - NodeArray_sz].parent;
            }
        }
        SubdivideVvalue[i]=val-isoValue;
    }
}

__device__ int VertexIndex(const int &x,const int &y,const int &z){
    int ret = x | (z<<2);
    if(y==1){
        if(ret & 1){
            ++ret;
        }else{
            ret+=3;
        }
    }
    return ret;
//    return (z<<2)|(y<<1)|x;
}

__global__ void generateVexNums(EdgeNode *EdgeArray,int EdgeArray_sz,
                                OctNode *NodeArray,float *vvalue,
                                int *vexNums)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<EdgeArray_sz;i+=stride){
        EdgeNode nowEdge=EdgeArray[i];
        int owner=nowEdge.ownerNodeIdx;
        int kind=nowEdge.edgeKind;
        int orientation=kind>>2;
        int idx[2];
        idx[0]=edgeVertex[kind][0];
        idx[1]=edgeVertex[kind][1];

        int v1=NodeArray[owner].vertices[idx[0]]-1;
        int v2=NodeArray[owner].vertices[idx[1]]-1;
        if(vvalue[v1]*vvalue[v2]<=0){
            vexNums[i]=1;
        }
    }
}

__global__ void generateSubdivideVexNums(EdgeNode *SubdivideEdgeArray,int SubdivideEdgeArray_sz,
                                         int NodeArray_sz,
                                         OctNode *SubdivideArray,float *SubdivideVvalue,
                                         int *SubdivideVexNums)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<SubdivideEdgeArray_sz;i+=stride){
        EdgeNode nowEdge=SubdivideEdgeArray[i];
        int owner=nowEdge.ownerNodeIdx - NodeArray_sz;
        int kind=nowEdge.edgeKind;
        int orientation=kind>>2;
        int idx[2];
        idx[0]=edgeVertex[kind][0];
        idx[1]=edgeVertex[kind][1];

        int v1=SubdivideArray[owner].vertices[idx[0]]-1;
        int v2=SubdivideArray[owner].vertices[idx[1]]-1;
//        printf("%d %f %d %f\n",v1,SubdivideVvalue[v1],v2,SubdivideVvalue[v2]);
        if(SubdivideVvalue[v1]*SubdivideVvalue[v2]<=0){
            SubdivideVexNums[i]=1;
        }
    }
}

__global__ void generateSubdivideVexNums(EdgeNode *SubdivideEdgeArray,int SubdivideEdgeArray_sz,
                                         int NodeArray_sz,
                                         EasyOctNode *SubdivideArray,float *SubdivideVvalue,
                                         int *SubdivideVexNums)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<SubdivideEdgeArray_sz;i+=stride){
        EdgeNode nowEdge=SubdivideEdgeArray[i];
        int owner=nowEdge.ownerNodeIdx - NodeArray_sz;
        int kind=nowEdge.edgeKind;
        int orientation=kind>>2;
        int idx[2];
        idx[0]=edgeVertex[kind][0];
        idx[1]=edgeVertex[kind][1];

        int v1=SubdivideArray[owner].vertices[idx[0]]-1;
        int v2=SubdivideArray[owner].vertices[idx[1]]-1;
//        printf("%d %f %d %f\n",v1,SubdivideVvalue[v1],v2,SubdivideVvalue[v2]);
        if(SubdivideVvalue[v1]*SubdivideVvalue[v2]<=0){
            SubdivideVexNums[i]=1;
        }
    }
}

struct validVexNums{
    __device__ bool operator()(const int &x){
        return x != 0;
    }
};


__global__ void generateTriNums(OctNode *NodeArray,
                                int left,int right,
                                float *vvalue,
                                int *triNums,int *cubeCatagory)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    for(int i=offset;i<right;i+=stride){
        OctNode nowNode=NodeArray[i];
        int nowCubeCatagory=0;
        for(int j=0;j<8;++j){
            if(vvalue[nowNode.vertices[j]-1] < 0){
                nowCubeCatagory |= 1<<j;
            }
        }
        triNums[i-left]=trianglesCount[nowCubeCatagory];
        cubeCatagory[i-left]=nowCubeCatagory;
    }
}

__global__ void generateTriNums(EasyOctNode *NodeArray,
                                int left,int right,
                                float *vvalue,
                                int *triNums,int *cubeCatagory)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    for(int i=offset;i<right;i+=stride){
        EasyOctNode nowNode=NodeArray[i];
        int nowCubeCatagory=0;
        for(int j=0;j<8;++j){
            if(vvalue[nowNode.vertices[j]-1] < 0){
                nowCubeCatagory |= 1<<j;
            }
        }
        triNums[i-left]=trianglesCount[nowCubeCatagory];
        cubeCatagory[i-left]=nowCubeCatagory;
    }
}

__device__ void interpolatePoint(const Point3D<float> &p1,const Point3D<float> &p2,
                                 const int &dim,const float &v1,const float &v2,
                                 Point3D<float> & out)
{
    for(int i=0;i<3;++i){
        if(i!=dim){
            out.coords[i]=p1.coords[i];
        }
    }
    float pivot = v1/(v1-v2);
    float another_pivot=1-pivot;
    out.coords[dim]= p2.coords[dim] * pivot + p1.coords[dim] * another_pivot;
//    out.coords[dim]=p1.coords[dim]+(p2.coords[dim]-p1.coords[dim])*pivot;
}

__global__ void generateIntersectionPoint(EdgeNode *validEdgeArray,int validEdgeArray_sz,
                                          VertexNode *VertexArray,OctNode *NodeArray,
                                          int *validVexAddress,float *vvalue,
                                          Point3D<float> *VertexBuffer)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<validEdgeArray_sz;++i){
        int owner=validEdgeArray[i].ownerNodeIdx;
        int kind=validEdgeArray[i].edgeKind;
        int orientation=kind>>2;

        int idx[2];

        idx[0]=edgeVertex[kind][0];
        idx[1]=edgeVertex[kind][1];

        int v1=NodeArray[owner].vertices[idx[0]]-1;
        int v2=NodeArray[owner].vertices[idx[1]]-1;
        Point3D<float> p1=VertexArray[v1].pos,p2=VertexArray[v2].pos;
        float f1=vvalue[v1],f2=vvalue[v2];
        Point3D<float> isoPoint;
        interpolatePoint(p1,p2,
                         orientation,f1,f2,
                         isoPoint);
        VertexBuffer[validVexAddress[i]] = isoPoint;
    }
}

__global__ void generateSubdivideIntersectionPoint(EdgeNode *SubdivideValidEdgeArray,int SubdivideValidEdgeArray_sz,
                                                   VertexNode *SubdivideVertexArray,OctNode *SubdivideArray,
                                                   int NodeArray_sz,
                                                   int *SubdivideValidVexAddress,float *SubdivideVvalue,
                                                   Point3D<float> *SubdivideVertexBuffer)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<SubdivideValidEdgeArray_sz;++i){
        int owner=SubdivideValidEdgeArray[i].ownerNodeIdx - NodeArray_sz;
        int kind=SubdivideValidEdgeArray[i].edgeKind;
        int orientation=kind>>2;

        int idx[2];

        idx[0]=edgeVertex[kind][0];
        idx[1]=edgeVertex[kind][1];

        int v1=SubdivideArray[owner].vertices[idx[0]]-1;
        int v2=SubdivideArray[owner].vertices[idx[1]]-1;
//        printf("v1:%d v2:%d ok\n",v1,v2);
        Point3D<float> p1=SubdivideVertexArray[v1].pos,p2=SubdivideVertexArray[v2].pos;
//        printf("p1 p2 ok\n");
        float f1=SubdivideVvalue[v1],f2=SubdivideVvalue[v2];
//        printf("f1 f2 ok\n");
        Point3D<float> isoPoint;
        interpolatePoint(p1,p2,
                         orientation,f1,f2,
                         isoPoint);
//        printf("interpolate ok\n");
        SubdivideVertexBuffer[SubdivideValidVexAddress[i]] = isoPoint;
//        printf("assign ok\n");
//        printf("%d\n",SubdivideValidVexAddress[i]);
    }
}


__global__ void generateSubdivideIntersectionPoint(EdgeNode *SubdivideValidEdgeArray,int SubdivideValidEdgeArray_sz,
                                                   VertexNode *SubdivideVertexArray,EasyOctNode *SubdivideArray,
                                                   int NodeArray_sz,
                                                   int *SubdivideValidVexAddress,float *SubdivideVvalue,
                                                   Point3D<float> *SubdivideVertexBuffer)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<SubdivideValidEdgeArray_sz;++i){
        int owner=SubdivideValidEdgeArray[i].ownerNodeIdx - NodeArray_sz;
        int kind=SubdivideValidEdgeArray[i].edgeKind;
        int orientation=kind>>2;

        int idx[2];

        idx[0]=edgeVertex[kind][0];
        idx[1]=edgeVertex[kind][1];

        int v1=SubdivideArray[owner].vertices[idx[0]]-1;
        int v2=SubdivideArray[owner].vertices[idx[1]]-1;
        Point3D<float> p1=SubdivideVertexArray[v1].pos,p2=SubdivideVertexArray[v2].pos;
        float f1=SubdivideVvalue[v1],f2=SubdivideVvalue[v2];
        Point3D<float> isoPoint;
        interpolatePoint(p1,p2,
                         orientation,f1,f2,
                         isoPoint);
        SubdivideVertexBuffer[SubdivideValidVexAddress[i]] = isoPoint;
//        printf("%d\n",SubdivideValidVexAddress[i]);
    }
}

__global__ void generateTrianglePos(OctNode *NodeArray,int left,int right,
                                    int *triNums,int *cubeCatagory,
                                    int *vexAddress,
                                    int *triAddress, int *TriangleBuffer,
                                    FaceNode *FaceArray,int *hasSurfaceIntersection)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    for(int i=offset;i<right;i+=stride){
        OctNode nowNode = NodeArray[i];
        int depthDIdx = i-left;
        int nowTriNum = triNums[depthDIdx];
        int nowCubeCatagory = cubeCatagory[depthDIdx];
        int nowTriangleBufferStart = 3 * triAddress[depthDIdx];
        int edgeHasVertex[12]={0};
        for(int j=0;j<12;++j){
            edgeHasVertex[j]=0;
        }
        for(int j=0;j<3*nowTriNum;j+=3){
            int edgeIdx[3];
            edgeIdx[0]=triangles[nowCubeCatagory][j];
            edgeIdx[1]=triangles[nowCubeCatagory][j+1];
            edgeIdx[2]=triangles[nowCubeCatagory][j+2];

            edgeHasVertex[edgeIdx[0]]=1;
            edgeHasVertex[edgeIdx[1]]=1;
            edgeHasVertex[edgeIdx[2]]=1;

            int vertexIdx[3];
            vertexIdx[0] = vexAddress[nowNode.edges[edgeIdx[0]] - 1];
            vertexIdx[1] = vexAddress[nowNode.edges[edgeIdx[1]] - 1];
            vertexIdx[2] = vexAddress[nowNode.edges[edgeIdx[2]] - 1];

            TriangleBuffer[ nowTriangleBufferStart + j ] = vertexIdx[0];
            TriangleBuffer[ nowTriangleBufferStart + j + 1 ] = vertexIdx[1];
            TriangleBuffer[ nowTriangleBufferStart + j + 2 ] = vertexIdx[2];
        }
        int nowFace;
        int parentNodeId;
        for(int j=0;j<6;++j){
            int mark=0;
            for(int k=0;k<4;++k){
                mark |= edgeHasVertex[faceEdges[j][k]];
            }
            if(mark){
                parentNodeId=NodeArray[i].parent;
                nowFace=nowNode.faces[j] - 1;
                hasSurfaceIntersection[nowFace]=1;
                while(FaceArray[nowFace].hasParentFace != -1){
                    nowFace=NodeArray[parentNodeId].faces[j] - 1;
                    parentNodeId = NodeArray[parentNodeId].parent;
                    hasSurfaceIntersection[nowFace]=1;
                }
            }
        }
    }
}

__global__ void generateSubdivideTrianglePos(OctNode *SubdivideArray,int left,int right,
                                             int *SubdivideTriNums,int *SubdivideCubeCatagory,
                                             int *SubdivideVexAddress,
                                             int *SubdivideTriAddress, int *SubdivideTriangleBuffer)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    for(int i=offset;i<right;i+=stride){
        OctNode nowNode = SubdivideArray[i];
        int depthDIdx = i-left;
        int nowTriNum = SubdivideTriNums[depthDIdx];
        int nowCubeCatagory = SubdivideCubeCatagory[depthDIdx];
        int nowTriangleBufferStart = 3 * SubdivideTriAddress[depthDIdx];
//        printf("%d %d\n",depthDIdx,nowTriangleBufferStart);
        for(int j=0;j<3*nowTriNum;j+=3){
            int edgeIdx[3];
            edgeIdx[0]=triangles[nowCubeCatagory][j];
            edgeIdx[1]=triangles[nowCubeCatagory][j+1];
            edgeIdx[2]=triangles[nowCubeCatagory][j+2];

            int vertexIdx[3];
            vertexIdx[0] = SubdivideVexAddress[nowNode.edges[edgeIdx[0]] - 1];
            vertexIdx[1] = SubdivideVexAddress[nowNode.edges[edgeIdx[1]] - 1];
            vertexIdx[2] = SubdivideVexAddress[nowNode.edges[edgeIdx[2]] - 1];

            SubdivideTriangleBuffer[ nowTriangleBufferStart + j ] = vertexIdx[0];
            SubdivideTriangleBuffer[ nowTriangleBufferStart + j + 1 ] = vertexIdx[1];
            SubdivideTriangleBuffer[ nowTriangleBufferStart + j + 2 ] = vertexIdx[2];
        }
    }
}

__global__ void generateSubdivideTrianglePos(EasyOctNode *SubdivideArray,int left,int right,
                                             int *SubdivideTriNums,int *SubdivideCubeCatagory,
                                             int *SubdivideVexAddress,
                                             int *SubdivideTriAddress, int *SubdivideTriangleBuffer)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    for(int i=offset;i<right;i+=stride){
        EasyOctNode nowNode = SubdivideArray[i];
        int depthDIdx = i-left;
        int nowTriNum = SubdivideTriNums[depthDIdx];
        int nowCubeCatagory = SubdivideCubeCatagory[depthDIdx];
        int nowTriangleBufferStart = 3 * SubdivideTriAddress[depthDIdx];
//        printf("%d %d\n",depthDIdx,nowTriangleBufferStart);
        for(int j=0;j<3*nowTriNum;j+=3){
            int edgeIdx[3];
            edgeIdx[0]=triangles[nowCubeCatagory][j];
            edgeIdx[1]=triangles[nowCubeCatagory][j+1];
            edgeIdx[2]=triangles[nowCubeCatagory][j+2];

            int vertexIdx[3];
            vertexIdx[0] = SubdivideVexAddress[nowNode.edges[edgeIdx[0]] - 1];
            vertexIdx[1] = SubdivideVexAddress[nowNode.edges[edgeIdx[1]] - 1];
            vertexIdx[2] = SubdivideVexAddress[nowNode.edges[edgeIdx[2]] - 1];

            SubdivideTriangleBuffer[ nowTriangleBufferStart + j ] = vertexIdx[0];
            SubdivideTriangleBuffer[ nowTriangleBufferStart + j + 1 ] = vertexIdx[1];
            SubdivideTriangleBuffer[ nowTriangleBufferStart + j + 2 ] = vertexIdx[2];
        }
    }
}

__global__ void initFaceArray(OctNode *NodeArray,int left,int right,FaceNode *preFaceArray,int *DepthBuffer,Point3D<float> *CenterBuffer){

    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    int NodeOwnerKey[6];
    int NodeOwnerIdx[6];
    for(int i=offset;i<right;i+=stride){
        int nowDepth = DepthBuffer[i];
        float halfWidth = 1.0f/(1<<(nowDepth+1));
        float Width = 1.0f/(1<<nowDepth);
        float Widthsq = Width * Width;
        Point3D<float> neighCenter[27];
        int neigh[27];
#pragma unroll
        for(int k=0;k<27;++k){
            neigh[k]=NodeArray[i].neighs[k];
            if(neigh[k] != -1){
                neighCenter[k]=CenterBuffer[neigh[k]];
            }
        }
        const Point3D<float> &nodeCenter = neighCenter[13];

        Point3D<float> faceCenterPos[6];
        int orientation;
        int off;
        int multi;
        for(int j=0;j<6;++j){
            orientation = j>>1;
            off = j&1;
            multi = (2*off)-1;
            faceCenterPos[j].coords[0] = nodeCenter.coords[0];
            faceCenterPos[j].coords[1] = nodeCenter.coords[1];
            faceCenterPos[j].coords[2] = nodeCenter.coords[2];
            faceCenterPos[j].coords[orientation] += multi * halfWidth;
        }

        for(int j=0;j<6;++j){
            NodeOwnerKey[j]=0x7fffffff;
        }
        for(int j=0;j<6;++j){
            for(int k=0;k<27;++k){
                if(neigh[k] != -1 && SquareDistance(faceCenterPos[j],neighCenter[k]) < Widthsq){
                    int neighKey = NodeArray[neigh[k]].key;
                    if(NodeOwnerKey[j]>neighKey){
                        NodeOwnerKey[j]=neighKey;
                        NodeOwnerIdx[j]=neigh[k];
                    }
                }
            }
        }

        int parent = NodeArray[i].parent;
        int sonKey = ( NodeArray[i].key >> (3 * (maxDepth-nowDepth)) ) & 7;
        for(int j=0;j<6;++j){
            if(NodeOwnerIdx[j] == i){
                int faceIdx = 6 * (i-left) + j;
                preFaceArray[faceIdx].ownerNodeIdx = i;
                preFaceArray[faceIdx].faceKind = j;
                if(parent == -1){
                    preFaceArray[faceIdx].hasParentFace = -1;
                }else{
                    if(parentFaceKind[sonKey][j] != -1){
                        preFaceArray[faceIdx].hasParentFace = 1;
                    }else{
                        preFaceArray[faceIdx].hasParentFace = -1;
                    }
                }
            }
        }

    }
}

struct validFace{
    __device__ bool operator()(const FaceNode &x){
        return x.ownerNodeIdx >= 0;
    }
};

__global__ void maintainFaceNodePointer(FaceNode *FaceArray,int FaceArray_sz,
                                        OctNode *NodeArray,
                                        int *DepthBuffer,Point3D<float> *CenterBuffer){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<FaceArray_sz;i+=stride){
        int owner = FaceArray[i].ownerNodeIdx;
        int depth = DepthBuffer[owner];
        float halfWidth = 1.0f/(1<<(depth+1));
        float Width = 1.0f/(1<<depth);
        float Widthsq = Width * Width;

        Point3D<float> neighCenter[27];
        int neigh[27];
        for(int k=0;k<27;++k){
            neigh[k]=NodeArray[owner].neighs[k];
            if(neigh[k] != -1){
                neighCenter[k]=CenterBuffer[neigh[k]];
            }
        }

        const Point3D<float> &nodeCenter = neighCenter[13];
        Point3D<float> faceCenterPos;
        int kind = FaceArray[i].faceKind;
        int orientation = kind >> 1;
        int off = kind & 1;
        int multi = (2*off)-1;

        faceCenterPos.coords[0] = nodeCenter.coords[0];
        faceCenterPos.coords[1] = nodeCenter.coords[1];
        faceCenterPos.coords[2] = nodeCenter.coords[2];
        faceCenterPos.coords[orientation] += multi *halfWidth;

        int cnt=0;
        for(int k=0;k<27;++k){
            if(neigh[k] != -1 && SquareDistance(faceCenterPos,neighCenter[k]) < Widthsq){
                FaceArray[i].nodes[cnt] = neigh[k];
                ++cnt;
                int idx = orientation << 1;
                if(neighCenter[k].coords[orientation]-faceCenterPos.coords[orientation]<0)
                    ++idx;

                NodeArray[neigh[k]].faces[idx] = i+1;
            }
        }
    }
}

__global__ void ProcessLeafNodesAtOtherDepth(OctNode *NodeArray,int left,int right,
                                             VertexNode *VertexArray, float *vvalue,
                                             int *hasSurfaceIntersection)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    for(int i=offset;i<right;i+=stride){
        OctNode nowNode = NodeArray[i];
        int hasTri=0;
        int sign = (vvalue[nowNode.vertices[0]-1] < 0 )? -1:1;
        for(int j=1;j<8;++j){
            if(sign * vvalue[nowNode.vertices[j]-1] < 0) {
                hasTri=1;
                break;
            }
        }
        NodeArray[i].hasTriangle=hasTri;

        int hasIntersection=0;
        for(int j=0;j<6;++j){
            if(hasSurfaceIntersection[nowNode.faces[j]-1]){
                hasIntersection=1;
                break;
            }
        }
        NodeArray[i].hasIntersection=hasIntersection;
    }
}

struct ifSubdivide{
    __device__ bool operator()(const OctNode &x){
        return (x.children[0] == -1) && (x.hasTriangle || x.hasIntersection);
    }
};

__global__ void precomputeSubdivideDepth(OctNode *SubdivideNode,int SubdivideNum,
                                         int *DepthBuffer,
                                         int *SubdivideDepthBuffer,
                                         int *SubdivideDepthNum)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<SubdivideNum;i+=stride){
        int nowNodeId = SubdivideNode[i].neighs[13];
        int depth = DepthBuffer[nowNodeId];
        SubdivideDepthBuffer[i] = depth;
        SubdivideDepthNum[i + depth * SubdivideNum] = 1;
    }
}

// correct, but the local memory doesn't support this function to run successfully
// deprecated
__global__ void initFixedDepthNums(OctNode *SubdivideNode,int left,int right,
                                   int *SubdivideDepthBuffer,
                                   int *fixedDepthNums)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset += left;
    for(int i=offset;i<right;i+=stride) {
        int nodeNum=1;
        for(int depth=SubdivideDepthBuffer[i];depth <= maxDepth;++depth){
            fixedDepthNums[ (depth-1) * (right-left) + i - left] = nodeNum;
//            nodeNum *= 8;
            nodeNum <<= 3;
        }
    }
}

// correct, but the local memory doesn't support this function to run successfully
// deprecated
__global__ void wholeRebuildArray(OctNode *SubdivideNode,int left,int right,
                                  OctNode *NodeArray,int NodeArray_sz,
                                  int *SubdivideDepthBuffer,
                                  int *depthNodeAddress_d,int *fixedDepthAddress,
                                  EasyOctNode *RebuildArray,
                                  int *RebuildDepthBuffer,Point3D<float> *RebuildCenterBuffer,
                                  int *ReplaceNodeId,int *IsRoot)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset += left;
    int depthNodeAddress[maxDepth+1];
    for(int i=0;i<=maxDepth;++i){
        depthNodeAddress[i]=depthNodeAddress_d[i];
    }
    int finerSubdivideNum = right - left;
    for(int i=offset;i<right;i+=stride){
        int nowDepth = SubdivideDepthBuffer[i];
        int relativeId = i - left;
        int fixedDepthOffset = fixedDepthAddress[(nowDepth-1) * finerSubdivideNum + relativeId];
        int nowIdx = depthNodeAddress[nowDepth] + fixedDepthOffset;
        OctNode rootNode = SubdivideNode[i];
        int replacedId = rootNode.neighs[13];
        rootNode.neighs[13] = NodeArray_sz + nowIdx;
        RebuildArray[nowIdx] = rootNode;

        ReplaceNodeId[nowIdx] = replacedId;

        RebuildDepthBuffer[nowIdx] = nowDepth;

        IsRoot[nowIdx] = 1;
        Point3D<float> thisNodeCenter;
        getNodeCenterAllDepth(rootNode.key,thisNodeCenter,nowDepth);
        RebuildCenterBuffer[nowIdx] = thisNodeCenter;

        int sonKey = ( rootNode.key >> (3 * (maxDepth-nowDepth)) ) & 7;
        NodeArray[rootNode.parent].children[sonKey] = NodeArray_sz + nowIdx;
        int parentNodeIdx;
        int childrenNums=8;
        while(nowDepth < maxDepth) {
            ++nowDepth;
            fixedDepthOffset = fixedDepthAddress[(nowDepth - 1) * finerSubdivideNum + relativeId];
            nowIdx = depthNodeAddress[nowDepth] + fixedDepthOffset;
            for (int j = 0; j < childrenNums; j+=8) {
                int fatherFixedDepthOffset = fixedDepthAddress[(nowDepth-2) * finerSubdivideNum + relativeId];
                parentNodeIdx = depthNodeAddress[nowDepth-1] + fatherFixedDepthOffset + j/8;
                int parentGlobalIdx=RebuildArray[parentNodeIdx].neighs[13];
                int parentKey=RebuildArray[parentNodeIdx].key;
                for(int k=0;k<8;++k){
                    int thisRoundIdx = nowIdx + j + k;
                    int nowKey = parentKey | (k << (3 * (maxDepth - nowDepth)));
                    RebuildArray[thisRoundIdx].parent = parentGlobalIdx;
                    RebuildArray[thisRoundIdx].key = nowKey;
                    RebuildArray[thisRoundIdx].neighs[13] = NodeArray_sz + thisRoundIdx;

                    ReplaceNodeId[thisRoundIdx] = replacedId;

                    RebuildDepthBuffer[thisRoundIdx] = nowDepth;

                    getNodeCenterAllDepth(nowKey,thisNodeCenter,nowDepth);
                    RebuildCenterBuffer[thisRoundIdx] = thisNodeCenter;

                    RebuildArray[parentNodeIdx].children[k] = NodeArray_sz + thisRoundIdx;
                }
            }
            childrenNums<<=3;
        }
    }
}

__device__ int getSubdivideDepth(const int &rootDepth,const int &idx){
    int up = idx*7+1;
    int base = 8;
    int relativeDepth = 0;
    while(base <= up){
        ++relativeDepth;
        base<<=3;
    }
    return rootDepth + relativeDepth;
}

__global__ void singleRebuildArray(int NodeArray_sz,
                                   OctNode *SubdivideNode,int *SubdivideDepthBuffer,int iterRound,
                                   EasyOctNode *SubdivideArray,int SubdivideArray_sz,
                                   int *SubdivideArrayDepthBuffer,Point3D<float> *SubdivideArrayCenterBuffer)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int rootId = SubdivideNode[iterRound].neighs[13];
    int rootDepth = SubdivideDepthBuffer[iterRound];
    int rootKey = SubdivideNode[iterRound].key;
    for(int i=offset;i<SubdivideArray_sz;i+=stride){
        int thisNodeDepth = getSubdivideDepth(rootDepth,i);
        int relativeDepth = thisNodeDepth - rootDepth;
        int idxOffset = i - (qpow(8,relativeDepth)-1)/7;

        if(thisNodeDepth < maxDepth){
            int nextDepthAddress = (qpow(8,relativeDepth+1)-1)/7;
            for(int k=0;k<8;++k){
                SubdivideArray[i].children[k] = NodeArray_sz + nextDepthAddress + (idxOffset << 3) + k;
            }
        }else{
            for(int k=0;k<8;++k){
                SubdivideArray[i].children[k] = -1;
            }
        }

        if(i != 0){
            int parentDepthAddress = (qpow(8,relativeDepth-1)-1)/7;
            SubdivideArray[i].parent = NodeArray_sz + parentDepthAddress + (idxOffset >> 3);
        }

        int thisKey = rootKey;
        thisKey |= (idxOffset) << (3 * (maxDepth-thisNodeDepth));
        SubdivideArray[i].key = thisKey;

        SubdivideArrayDepthBuffer[i] = thisNodeDepth;
        Point3D<float> thisNodeCenter;
        getNodeCenterAllDepth(thisKey,thisNodeCenter,thisNodeDepth);
        SubdivideArrayCenterBuffer[i]=thisNodeCenter;
    }
}

__global__ void computeRebuildNeighbor(OctNode *SubdivideArray,int left,int right,
                                       OctNode *NodeArray,int NodeArray_sz,
                                       int depthD)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    for(int i=offset;i<right;i+=stride){
        for(int j=0;j<27;++j){
            int sonKey = ( SubdivideArray[i].key >> (3 * (maxDepth-depthD)) ) & 7;
            int parentIdx = SubdivideArray[i].parent;
            int neighParent;
            if(parentIdx < NodeArray_sz){
                neighParent = NodeArray[ parentIdx ].neighs[LUTparent[sonKey][j]];
            }else{
                neighParent = SubdivideArray[ parentIdx - NodeArray_sz ].neighs[LUTparent[sonKey][j]];
            }
            if(neighParent != -1){
                if(neighParent < NodeArray_sz) {
                    SubdivideArray[i].neighs[j] = NodeArray[neighParent].children[LUTchild[sonKey][j]];
                }else{
                    SubdivideArray[i].neighs[j] = SubdivideArray[neighParent - NodeArray_sz ].children[LUTchild[sonKey][j]];
                }
            }else{
                SubdivideArray[i].neighs[j]= -1;
            }
        }
    }
}

__global__ void computeRebuildNeighbor(EasyOctNode *SubdivideArray,int left,int right,
                                       OctNode *NodeArray,int NodeArray_sz,
                                       int depthD)
{
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    offset+=left;
    for(int i=offset;i<right;i+=stride){
        for(int j=0;j<27;++j){
            int sonKey = ( SubdivideArray[i].key >> (3 * (maxDepth-depthD)) ) & 7;
            int parentIdx = SubdivideArray[i].parent;
            int neighParent;
            if(parentIdx < NodeArray_sz){
                neighParent = NodeArray[ parentIdx ].neighs[LUTparent[sonKey][j]];
            }else{
                neighParent = SubdivideArray[ parentIdx - NodeArray_sz ].neighs[LUTparent[sonKey][j]];
            }
            if(neighParent != -1){
                if(neighParent < NodeArray_sz) {
                    SubdivideArray[i].neighs[j] = NodeArray[neighParent].children[LUTchild[sonKey][j]];
                }else{
                    SubdivideArray[i].neighs[j] = SubdivideArray[neighParent - NodeArray_sz ].children[LUTchild[sonKey][j]];
                }
            }else{
                SubdivideArray[i].neighs[j]= -1;
            }
        }
    }
}


__host__ void insertTriangle(Point3D<float> *VertexBuffer,const int &allVexNums,
                             int *TriangleBuffer,const int &allTriNums,
                             CoredVectorMeshData &mesh)
{
    int previousVertex = mesh.inCorePoints.size();
    for(int i=0;i<allVexNums;++i){
        if(abs(VertexBuffer[i].coords[0])<EPSILON)
            printf("error\n");
        mesh.inCorePoints.push_back(VertexBuffer[i]);
    }
    int inCoreFlag=0;
    for(int i=0;i<3;++i){
        inCoreFlag|=CoredMeshData::IN_CORE_FLAG[i];
    }
    for(int i=0;i<allTriNums;++i){
        TriangleIndex tri;
        for(int j=0;j<3;++j) {
            tri.idx[j] = TriangleBuffer[3*i+j] + previousVertex;
            if(tri.idx[j]<0 || tri.idx[j]>=allVexNums + previousVertex){
                printf("%d %d\n",tri.idx[j]-previousVertex,allVexNums);
                printf("tri error\n");
            }
        }
        mesh.addTriangle(tri,inCoreFlag);
    }
}

int main() {
//    char fileName[]="/home/davidxu/horse.npts";
//    char outName[]="/home/davidxu/horse.ply";

    char fileName[]="/home/davidxu/bunny.points.ply";
    char outName[]="/home/davidxu/bunny.ply";

//    char fileName[]="/home/davidxu/eagle.points.ply";
//    char outName[]="/home/davidxu/eagle.ply";

    int NodeArrayCount_h[maxDepth_h+1];
    int BaseAddressArray[maxDepth_h+1];

    Point3D<float> *samplePoints_d=NULL, *sampleNormals_d=NULL;
    int *PointToNodeArrayD;
    OctNode *NodeArray;
    int count=0;
    int NodeArray_sz=0;
    Point3D<float> center;
    float scale;

    double project_st=cpuSecond();

    // the number of nodes at maxDepth is very large, some maintaining of their info is time-consuming
    pipelineBuildNodeArray(fileName,center,scale,count,NodeArray_sz,
                           NodeArrayCount_h,BaseAddressArray,
                           samplePoints_d,sampleNormals_d,PointToNodeArrayD,NodeArray );

    printf("NodeArray_sz:%d\n",NodeArray_sz);

//    outputDeviceArray<<<1,1>>>(PointToNodeArrayD,20);
//    hipDeviceSynchronize();

    int *BaseAddressArray_d=NULL;
    CHECK(hipMalloc((int **)&BaseAddressArray_d,sizeof(int)*(maxDepth_h+1) ));
    CHECK(hipMemcpy(BaseAddressArray_d,BaseAddressArray,sizeof(int)*(maxDepth_h+1),hipMemcpyHostToDevice));
//    for(int i=0;i<=maxDepth_h;++i){
//        printf("%d %d\n",NodeArrayCount_h[i],BaseAddressArray[i]);
//    }


//    OctNode *a=(OctNode *)malloc(sizeof(OctNode)*(BaseAddressArray[maxDepth_h]+NodeArrayCount_h[maxDepth_h]));
//    hipMemcpy(a,NodeArray,sizeof(OctNode)*(BaseAddressArray[maxDepth_h]+NodeArrayCount_h[maxDepth_h]),hipMemcpyDeviceToHost);
//    for(int i=BaseAddressArray[1];i<BaseAddressArray[3];++i){
//        std::cout<<std::bitset<32>(a[i].key)<<" pidx:"<<a[i].pidx<<" pnum:"<<a[i].pnum<<std::endl;
//        int idx[3];
//        getFunctionIdxOfNode(a[i].key, getDepth(i,BaseAddressArray),idx);
//        std::cout<<getDepth(i,BaseAddressArray)<<std::endl;
//        for(int j=0;j<3;++j){
//            printf("idx[%d]:%d ",j,idx[j]);
//        }
//        puts("");
//    }

    // ----------------------------------------------------

    double cpu_st=cpuSecond();

    PPolynomial<convTimes> ReconstructionFunction = PPolynomial<convTimes>::GaussianApproximation();
    FunctionData<convTimes,double> fData;
    fData.set(maxDepth_h,ReconstructionFunction,normalize,0);
    //  precomputed inner product table may can be optimized to GPU parallel
    fData.setDotTables(fData.DOT_FLAG | fData.D_DOT_FLAG | fData.D2_DOT_FLAG);
    PPolynomial<convTimes> &F=ReconstructionFunction;
    switch(normalize){
        case 2:
            F=F/sqrt((F*F).integral(F.polys[0].start,F.polys[F.polyCount-1].start));
            break;
        case 1:
            F=F/F.integral(F.polys[0].start,F.polys[F.polyCount-1].start);
            break;
        default:
            F=F/F(0);
    }

    long long nByte = 1ll * sizeof(double) * fData.res * fData.res;
    double *dot_F_F=NULL;
    CHECK(hipMalloc((double **)&dot_F_F,nByte));
    CHECK(hipMemcpy(dot_F_F,fData.dotTable,nByte,hipMemcpyHostToDevice));

    double *dot_F_DF=NULL;
    CHECK(hipMalloc((double **)&dot_F_DF,nByte));
    CHECK(hipMemcpy(dot_F_DF,fData.dDotTable,nByte,hipMemcpyHostToDevice));

    double *dot_F_D2F=NULL;
    CHECK(hipMalloc((double **)&dot_F_D2F,nByte));
    CHECK(hipMemcpy(dot_F_D2F,fData.d2DotTable,nByte,hipMemcpyHostToDevice));

    fData.clearDotTables(fData.DOT_FLAG | fData.D_DOT_FLAG | fData.D2_DOT_FLAG);

    ConfirmedPPolynomial<convTimes+1,convTimes+2> baseFunctions_h[fData.res];
    for(int i=0;i<fData.res;++i){
        baseFunctions_h[i]=fData.baseFunctions[i];
    }

    ConfirmedPPolynomial<convTimes+1,convTimes+2> *baseFunctions_d=NULL;
    nByte = 1ll * sizeof(ConfirmedPPolynomial<convTimes+1,convTimes+2>) * fData.res;
    CHECK(hipMalloc((ConfirmedPPolynomial<convTimes+1,convTimes+2>**)&baseFunctions_d,nByte));
    CHECK(hipMemcpy(baseFunctions_d,baseFunctions_h,nByte,hipMemcpyHostToDevice));

    double cpu_ed=cpuSecond();
    printf("CPU generate precomputed inner product table takes:%lfs\n",cpu_ed-cpu_st);

    // ----------------------------------------------------

    ConfirmedPPolynomial<convTimes,convTimes+2> BaseFunctionMaxDepth(ReconstructionFunction.scale(1.0/(1<<maxDepth_h)));
    nByte = 1ll * sizeof(BaseFunctionMaxDepth);
    ConfirmedPPolynomial<convTimes,convTimes+2> *BaseFunctionMaxDepth_d= NULL;
    CHECK(hipMalloc((ConfirmedPPolynomial<convTimes,convTimes+2>**)&BaseFunctionMaxDepth_d,nByte));
    CHECK(hipMemcpy(BaseFunctionMaxDepth_d,&BaseFunctionMaxDepth,nByte,hipMemcpyHostToDevice));

    int NodeDNum=NodeArrayCount_h[maxDepth_h];

    Point3D<float> *VectorField=NULL;
    nByte = 1ll * sizeof(Point3D<float>) * NodeArrayCount_h[maxDepth_h];
    CHECK(hipMalloc((Point3D<float> **)&VectorField,nByte));
    CHECK(hipMemset(VectorField,0,nByte));

    double st=cpuSecond();
    dim3 grid=(32,32);
    dim3 block(32,32);
    computeVectorField<<<grid,block>>>(BaseFunctionMaxDepth_d,samplePoints_d,sampleNormals_d,
                                       NodeArray,BaseAddressArray[maxDepth_h],NodeArray_sz,VectorField);
    hipDeviceSynchronize();

//    outputDeviceArray<<<1,1>>>(VectorField,200);
//    hipDeviceSynchronize();

    double mid1=cpuSecond();
    printf("Compute Vector Field takes:%lfs\n",mid1-st);

    // ----------------------------------------------------

    float *Divergence=NULL;
    nByte = 1ll * sizeof(float) * NodeArray_sz;
    CHECK(hipMallocManaged((float **)&Divergence,nByte));
    CHECK(hipMemset(Divergence,0,nByte));

    int *EncodedNodeIdxInFunction=NULL;
    nByte = 1ll * sizeof(int) * NodeArray_sz;
    CHECK(hipMalloc((int **)&EncodedNodeIdxInFunction, nByte));
    precomputeEncodedFunctionIdxOfNode<<<grid,block>>>(BaseAddressArray_d,
                                                       NodeArray, NodeArray_sz,
                                                       EncodedNodeIdxInFunction);
    hipDeviceSynchronize();
    double mid2=cpuSecond();
    printf("Precompute Function index of node takes:%lfs\n",mid2-mid1);

    // memory access is very slow, maybe optimize it by setting faster memory.
    printf("left:%d,right:%d\n",BaseAddressArray[5],NodeArray_sz);
    computeEncodedFinerNodesDivergence<<<grid,block>>>(BaseAddressArray_d, EncodedNodeIdxInFunction,
                                                       NodeArray, BaseAddressArray[5],BaseAddressArray[maxDepth_h]+NodeArrayCount_h[maxDepth_h],
                                                       VectorField, dot_F_DF,
                                                       Divergence);
    hipDeviceSynchronize();

//    float *Divergence_h=(float *)malloc(sizeof(float)*NodeArray_sz);
//    hipMemcpy(Divergence_h,Divergence,sizeof(float)*NodeArray_sz,hipMemcpyDeviceToHost);
//    for(int i=BaseAddressArray[5];i<BaseAddressArray[6];++i){
//        printf("%f\n",Divergence[i]);
//    }

    double mid3=cpuSecond();
    printf("Compute finer depth nodes' divergence takes:%lfs\n",mid3-mid2);

    // ----------------------------------------------------

    // maybe can be optimized by running all nodes at the same time.
//    nByte = 1ll * sizeof(float) * NodeDNum;
    for(int i=4;i>=0;--i){
        for(int j=BaseAddressArray[i];j<BaseAddressArray[i+1];++j){
            int *coverNums=NULL;
            CHECK(hipMalloc((int**)&coverNums,sizeof(int) * 28));
            computeCoverNums<<<1,1>>>(NodeArray,j,coverNums);
            hipDeviceSynchronize();
            int coverNums_h[28];
            CHECK(hipMemcpy(coverNums_h,coverNums,sizeof(int) * 28,hipMemcpyDeviceToHost));
//            printf("%d,%d\n",j,coverNums_h);

            float *divg=NULL;
            nByte = 1ll * sizeof(float)*coverNums_h[27];
            CHECK(hipMalloc((float**)&divg,nByte));
            CHECK(hipMemset(divg,0,nByte));

            int *DIdxArray=NULL;
            nByte = 1ll * sizeof(int)*coverNums_h[27];
            CHECK(hipMalloc((int**)&DIdxArray,nByte));
            CHECK(hipMemset(DIdxArray,0,nByte));

            generateDIdxArray<<<grid,block>>>(NodeArray,j,coverNums,DIdxArray);
            hipDeviceSynchronize();

            computeEncodedCoarserNodesDivergence<<<grid,block>>>(DIdxArray, coverNums_h[27], BaseAddressArray_d,
                                                                 EncodedNodeIdxInFunction,
                                                                 VectorField, dot_F_DF,
                                                                 j, divg);
            hipDeviceSynchronize();
            thrust::device_ptr<float> divg_ptr=thrust::device_pointer_cast<float>(divg);
            float val=thrust::reduce(divg_ptr,divg_ptr+coverNums_h[27]);
            hipDeviceSynchronize();

            CHECK(hipMemcpy(Divergence+j,&val,sizeof(float),hipMemcpyHostToDevice));

            hipFree(DIdxArray);
            hipFree(divg);
        }
    }
    hipFree(VectorField);

    double mid4=cpuSecond();
    printf("Compute coarser depth nodes' divergence takes:%lfs\n",mid4-mid3);

    // ----------------------------------------------------

    // d_x is the Solution
    float *d_x=NULL;
    LaplacianIteration(BaseAddressArray,NodeArrayCount_h,4,
                       EncodedNodeIdxInFunction,NodeArray,Divergence,
                       NodeArray_sz,
                       dot_F_F,dot_F_D2F,
                       d_x);
    hipFree(Divergence);

    double mid5=cpuSecond();
    printf("GPU Laplacian Iteration takes:%lfs\n",mid5-mid4);

    // ----------------------------------------------------

    float *pointValue=NULL;
    nByte = 1ll * sizeof(float)*count;
    CHECK(hipMalloc((float**)&pointValue,nByte));
    CHECK(hipMemset(pointValue,0,nByte));

    grid=(32,32);
    block=(32,32);
    calculatePointsImplicitFunctionValue<<<grid,grid>>>(samplePoints_d,PointToNodeArrayD,count,BaseAddressArray[maxDepth_h],
                                                         NodeArray,d_x,
                                                         EncodedNodeIdxInFunction,baseFunctions_d,
                                                         pointValue);
    hipDeviceSynchronize();

    thrust::device_ptr<float> pointValue_ptr=thrust::device_pointer_cast<float>(pointValue);
    float isoValue=thrust::reduce(pointValue_ptr,pointValue_ptr+count);
    hipDeviceSynchronize();
    isoValue/=count;

    double mid6 = cpuSecond();
    printf("isoValue:%f\nGPU calculate isoValue takes:%lfs\n",isoValue,mid6-mid5);

    // ----------------------------------------------------

    // pre-compute the center of node ?
    int *DepthBuffer=NULL;
    nByte = 1ll * sizeof(int) * NodeArray_sz;
    CHECK(hipMalloc((int**)&DepthBuffer,nByte));

    Point3D<float> *CenterBuffer=NULL;
    nByte = 1ll * sizeof(Point3D<float>) * NodeArray_sz;
    CHECK(hipMallocManaged((Point3D<float>**)&CenterBuffer,nByte));

    precomputeDepthAndCenter<<<grid,block>>>(BaseAddressArray_d,NodeArray,NodeArray_sz,
                                             DepthBuffer,CenterBuffer);
    hipDeviceSynchronize();



    // generate all depth vertex
    VertexNode *preVertexArray=NULL;
    nByte = 1ll * sizeof(VertexNode) * 8 * NodeArray_sz;
    CHECK(hipMalloc((VertexNode**)&preVertexArray,nByte));
    CHECK(hipMemset(preVertexArray,0,nByte));
    grid=(32,32);
    block=(32,32);
    initVertexOwner<<<grid,block>>>(NodeArray,0,NodeArray_sz,
                                    preVertexArray,
                                    DepthBuffer,CenterBuffer);
    hipDeviceSynchronize();

    VertexNode *VertexArray=NULL;
//    nByte = 1ll * sizeof(VertexNode) * 8 * NodeArray_sz;
    CHECK(hipMalloc((VertexNode**)&VertexArray,nByte));
    CHECK(hipMemset(VertexArray,0,nByte));
    thrust::device_ptr<VertexNode> preVertexArray_ptr=thrust::device_pointer_cast<VertexNode>(preVertexArray);
    thrust::device_ptr<VertexNode> VertexArray_ptr=thrust::device_pointer_cast<VertexNode>(VertexArray);
    thrust::device_ptr<VertexNode> VertexArray_end=thrust::copy_if(preVertexArray_ptr,preVertexArray_ptr+8*NodeArray_sz,VertexArray_ptr,validVertex());
    hipDeviceSynchronize();

    hipFree(preVertexArray);

    int VertexArray_sz=VertexArray_end-VertexArray_ptr;

    maintainVertexNodePointerNonAtomic<<<grid,block>>>(VertexArray,VertexArray_sz,
                                                       NodeArray,
                                                       DepthBuffer,CenterBuffer);
    hipDeviceSynchronize();

//    OctNode *a=(OctNode *)malloc(sizeof(OctNode)*NodeArray_sz);
//    hipMemcpy(a,NodeArray,sizeof(OctNode)*(BaseAddressArray[maxDepth_h]+NodeArrayCount_h[maxDepth_h]),hipMemcpyDeviceToHost);
//    for(int j=maxDepth_h;j<=maxDepth_h;++j) {
//        int all=0;
//        for (int i = BaseAddressArray[j]; i < BaseAddressArray[j]+100; ++i) {
////            if(a[i].pnum==0) continue;
//            all+=a[i].dnum;
//            std::cout << i << " " <<std::bitset<32>(a[i].key) << " pidx:" << a[i].pidx << " pnum:" << a[i].pnum << " parent:"
//                      << a[i].parent << " didx:"<< a[i].didx << " dnum:" << a[i].dnum << std::endl;
//            for(int k=0;k<8;++k){
//                printf("children[%d]:%d ",k,a[i].children[k]);
//            }
//            puts("");
//            for(int k=0;k<27;++k){
//                printf("neigh[%d]:%d ",k,a[i].neighs[k]);
//            }
//            puts("");
//            for(int k=0;k<8;++k){
//                printf("vertices[%d]:%d ",k,a[i].vertices[k]);
//            }
//            puts("");
//        }
//        printf("allD:%d\n",all);
//        std::cout<<std::endl;
//    }

    double mid7=cpuSecond();
    printf("VertexArray_sz:%d\nGPU build VertexArray takes:%lfs\n",VertexArray_sz,mid7-mid6);

    // ----------------------------------------------------

    // generate the edge at maxDepth
    EdgeNode *preEdgeArray=NULL;
    nByte = 1ll * sizeof(EdgeNode) * 12 *NodeDNum;
    CHECK(hipMalloc((EdgeNode**)&preEdgeArray,nByte));
    CHECK(hipMemset(preEdgeArray,0,nByte));

    initEdgeArray<<<grid,block>>>(NodeArray,BaseAddressArray[maxDepth_h],NodeArray_sz,
                                  preEdgeArray,
                                  DepthBuffer,CenterBuffer);
    hipDeviceSynchronize();

    EdgeNode *EdgeArray=NULL;
//    nByte = 1ll * sizeof(VertexNode) * 12 * NodeDNum;
    CHECK(hipMalloc((EdgeNode**)&EdgeArray,nByte));
    CHECK(hipMemset(EdgeArray,0,nByte));
    thrust::device_ptr<EdgeNode> preEdgeArray_ptr=thrust::device_pointer_cast<EdgeNode>(preEdgeArray);
    thrust::device_ptr<EdgeNode> EdgeArray_ptr=thrust::device_pointer_cast<EdgeNode>(EdgeArray);
    thrust::device_ptr<EdgeNode> EdgeArray_end=thrust::copy_if(preEdgeArray_ptr,preEdgeArray_ptr+12*NodeDNum,EdgeArray_ptr,validEdge());
    hipDeviceSynchronize();

    hipFree(preEdgeArray);

    int EdgeArray_sz=EdgeArray_end-EdgeArray_ptr;

    maintainEdgeNodePointer<<<grid,block>>>(EdgeArray,EdgeArray_sz,
                                            NodeArray,
                                            DepthBuffer,CenterBuffer);
    hipDeviceSynchronize();

    double mid8=cpuSecond();
    printf("EdgeArray_sz:%d\nGPU build EdgeArray takes:%lfs\n",EdgeArray_sz,mid8-mid7);

    // ----------------------------------------------------


    // generate all depth face
    FaceNode *preFaceArray=NULL;
    nByte = 1ll * sizeof(FaceNode) * 6 * NodeArray_sz;
    CHECK(hipMalloc((FaceNode**)&preFaceArray,nByte));
    CHECK(hipMemset(preFaceArray,-1,nByte));

    initFaceArray<<<grid,block>>>(NodeArray,0,NodeArray_sz,preFaceArray,DepthBuffer,CenterBuffer);
    hipDeviceSynchronize();

    FaceNode *FaceArray = NULL;
//    nByte = 1ll * sizeof(FaceNode) * 6 * NodeArray_sz;
    CHECK(hipMalloc((FaceNode**)&FaceArray,nByte));
    CHECK(hipMemset(FaceArray,-1,nByte));
    thrust::device_ptr<FaceNode> preFaceArray_ptr=thrust::device_pointer_cast<FaceNode>(preFaceArray);
    thrust::device_ptr<FaceNode> FaceArray_ptr=thrust::device_pointer_cast<FaceNode>(FaceArray);
    thrust::device_ptr<FaceNode> FaceArray_end=thrust::copy_if(preFaceArray_ptr,preFaceArray_ptr+6*NodeArray_sz,FaceArray_ptr,validFace());
    hipDeviceSynchronize();

    int FaceArray_sz = FaceArray_end - FaceArray_ptr;

    hipFree(preFaceArray);

    maintainFaceNodePointer<<<grid,block>>>(FaceArray,FaceArray_sz,
                                            NodeArray,
                                            DepthBuffer,CenterBuffer);
    hipDeviceSynchronize();

    double mid_insert=cpuSecond();
    printf("FaceArray_sz:%d\nGPU build FaceArray takes:%lfs\n",FaceArray_sz,mid_insert-mid8);

    int *hasSurfaceIntersection=NULL;
    nByte = 1ll * sizeof(int) * FaceArray_sz;
    CHECK(hipMalloc((int**)&hasSurfaceIntersection,nByte));
    CHECK(hipMemset(hasSurfaceIntersection,0,nByte));


    // ----------------------------------------------------

    // Step 1: compute implicit function values for octree vertices
    float *vvalue = NULL;
    nByte = 1ll * sizeof(float) * VertexArray_sz;
    CHECK(hipMalloc((float**)&vvalue,nByte));
    CHECK(hipMemset(vvalue,0,nByte));

    computeVertexImplicitFunctionValue<<<grid,block>>>(VertexArray,VertexArray_sz,
                                                       NodeArray,d_x,
                                                       EncodedNodeIdxInFunction,baseFunctions_d,
                                                       vvalue,isoValue);
    hipDeviceSynchronize();

    double mid9=cpuSecond();
    printf("Compute vertex implicit function value takes:%lfs\n",mid9-mid_insert);

    // Step 2: compute vertex number and address
    int *vexNums=NULL;
    nByte = 1ll * sizeof(int) * EdgeArray_sz;
    CHECK(hipMalloc((int**)&vexNums,nByte));
    CHECK(hipMemset(vexNums,0,nByte));

    generateVexNums<<<grid,block>>>(EdgeArray,EdgeArray_sz,
                                    NodeArray,vvalue,
                                    vexNums);
    hipDeviceSynchronize();

    int *vexAddress=NULL;
//    nByte = 1ll * sizeof(int) * EdgeArray_sz;
    CHECK(hipMalloc((int**)&vexAddress,nByte));
    CHECK(hipMemset(vexAddress,0,nByte));

    thrust::device_ptr<int> vexNums_ptr=thrust::device_pointer_cast<int>(vexNums);
    thrust::device_ptr<int> vexAddress_ptr=thrust::device_pointer_cast<int>(vexAddress);

    thrust::exclusive_scan(vexNums_ptr,vexNums_ptr+EdgeArray_sz,vexAddress_ptr);
    hipDeviceSynchronize();

    double mid10=cpuSecond();
    printf("Compute vexAddress takes:%lfs\n",mid10-mid9);

    // Step 3: compute triangle number and address
    int *triNums=NULL;
    nByte = 1ll * sizeof(int) * NodeDNum;
    CHECK(hipMalloc((int**)&triNums,nByte));
    CHECK(hipMemset(triNums,0,nByte));

    int *cubeCatagory=NULL;
//    nByte = 1ll * sizeof(int) * NodeDNum;
    CHECK(hipMalloc((int**)&cubeCatagory,nByte));
    CHECK(hipMemset(cubeCatagory,0,nByte));

    generateTriNums<<<grid,block>>>(NodeArray,
                                    BaseAddressArray[maxDepth_h],NodeArray_sz,
                                    vvalue,
                                    triNums,cubeCatagory);
    hipDeviceSynchronize();

    int *triAddress=NULL;
//    nByte = 1ll * sizeof(int) * NodeDNum;
    CHECK(hipMalloc((int**)&triAddress,nByte));
    CHECK(hipMemset(triAddress,0,nByte));

    thrust::device_ptr<int> triNums_ptr=thrust::device_pointer_cast<int>(triNums);
    thrust::device_ptr<int> triAddress_ptr=thrust::device_pointer_cast<int>(triAddress);

    thrust::exclusive_scan(triNums_ptr,triNums_ptr+NodeDNum,triAddress_ptr);
    hipDeviceSynchronize();

    double mid11=cpuSecond();
    printf("Compute triAddress takes:%lfs\n",mid11-mid10);


    // Step 4: generate vertices
    int lastVexAddr;
    int lastVexNums;
    CHECK(hipMemcpy(&lastVexAddr,vexAddress+EdgeArray_sz-1,sizeof(int),hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(&lastVexNums,vexNums+EdgeArray_sz-1,sizeof(int),hipMemcpyDeviceToHost));
    int allVexNums = lastVexAddr + lastVexNums;
    Point3D<float> *VertexBuffer=NULL;
    nByte = 1ll * sizeof(Point3D<float>) * allVexNums;
    CHECK(hipMallocManaged((Point3D<float>**)&VertexBuffer,nByte));
//    CHECK(hipMemset(VertexBuffer,0,nByte));

    EdgeNode * validEdgeArray = NULL;
    nByte = 1ll * sizeof(EdgeNode) * allVexNums;
    CHECK(hipMalloc((EdgeNode**)&validEdgeArray,nByte));
    thrust::device_ptr<EdgeNode> validEdgeArray_ptr = thrust::device_pointer_cast<EdgeNode>(validEdgeArray);
    EdgeArray_ptr = thrust::device_pointer_cast<EdgeNode>(EdgeArray);
    thrust::device_ptr<EdgeNode> validEdgeArray_end = thrust::copy_if(EdgeArray_ptr,EdgeArray_ptr+EdgeArray_sz,vexNums_ptr,validEdgeArray_ptr,validVexNums());
    hipDeviceSynchronize();

    int validEdgeArray_sz = validEdgeArray_end - validEdgeArray_ptr;
    assert(allVexNums == validEdgeArray_sz);

    int *validVexAddress = NULL;
    nByte = 1ll * sizeof(int) * allVexNums;
    CHECK(hipMalloc((int**)&validVexAddress,nByte));
    thrust::device_ptr<int> validVexAddress_ptr = thrust::device_pointer_cast<int>(validVexAddress);
    thrust::device_ptr<int> validVexAddress_end = thrust::copy_if(vexAddress_ptr,vexAddress_ptr + EdgeArray_sz, vexNums_ptr,validVexAddress_ptr,validVexNums());
    hipDeviceSynchronize();

    int validVexAddress_sz = validVexAddress_end - validVexAddress_ptr;
    assert(allVexNums == validVexAddress_sz);

    grid = (std::min(allVexNums / 1024 , 32) ,allVexNums/32768);
    generateIntersectionPoint<<<grid,block>>>(validEdgeArray,allVexNums,
                                              VertexArray,NodeArray,
                                              validVexAddress,vvalue,
                                              VertexBuffer);
    hipDeviceSynchronize();

    grid = (32,32);

    double mid12=cpuSecond();
    printf("Generate interpolate vertices takes:%lfs\n",mid12-mid11);

    // Step 5: generate triangles
    int lastTriAddr;
    int lastTriNums;
    CHECK(hipMemcpy(&lastTriAddr,triAddress+NodeDNum-1,sizeof(int),hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(&lastTriNums,triNums+NodeDNum-1,sizeof(int),hipMemcpyDeviceToHost));
    int allTriNums = lastTriAddr+lastTriNums;

    int *TriangleBuffer=NULL;
    nByte = 1ll * sizeof(int) * 3 * allTriNums;
    CHECK(hipMallocManaged((int**)&TriangleBuffer,nByte));
//    CHECK(hipMemset(TriangleBuffer,0,nByte));

    generateTrianglePos<<<grid,block>>>(NodeArray,BaseAddressArray[maxDepth_h],NodeArray_sz,
                                        triNums,cubeCatagory,
                                        vexAddress,
                                        triAddress,TriangleBuffer,
                                        FaceArray,hasSurfaceIntersection);
    hipDeviceSynchronize();

    double mid13=cpuSecond();
    printf("Process Triangle indices takes:%lfs\n",mid13-mid12);


    CoredVectorMeshData mesh;

    insertTriangle(VertexBuffer,allVexNums,
                   TriangleBuffer,allTriNums,
                   mesh);

    // ----------------------------------------------------

    ProcessLeafNodesAtOtherDepth<<<grid,block>>>(NodeArray,0,BaseAddressArray[maxDepth_h],
                                                 VertexArray,vvalue,
                                                 hasSurfaceIntersection);
    hipDeviceSynchronize();

    hipFree(VertexArray);
    hipFree(EdgeArray);
    hipFree(FaceArray);
    hipFree(hasSurfaceIntersection);
//    hipFree(vvalue);
    hipFree(vexNums);
    hipFree(vexAddress);
    hipFree(triNums);
    hipFree(cubeCatagory);
    hipFree(triAddress);
    hipFree(VertexBuffer);
    hipFree(TriangleBuffer);
    hipFree(validEdgeArray);
    hipFree(validVexAddress);


    // ----------------------------------------------------


    OctNode *SubdivideNode=NULL;
    nByte = 1ll * sizeof(OctNode) * BaseAddressArray[maxDepth_h];
    CHECK(hipMalloc((OctNode**)&SubdivideNode,nByte));
    CHECK(hipMemset(SubdivideNode,0,nByte));
    thrust::device_ptr<OctNode> NodeArray_ptr=thrust::device_pointer_cast<OctNode>(NodeArray);
    thrust::device_ptr<OctNode> SubdivideNode_ptr=thrust::device_pointer_cast<OctNode>(SubdivideNode);
    thrust::device_ptr<OctNode> SubdivideNode_end=thrust::copy_if(NodeArray_ptr,NodeArray_ptr+BaseAddressArray[maxDepth_h],SubdivideNode_ptr,ifSubdivide());
    hipDeviceSynchronize();

    int SubdivideNum = SubdivideNode_end - SubdivideNode_ptr;

    printf("Starting processing the subdivide node\n");

    printf("SubdivideNum:%d\n",SubdivideNum);

    int *SubdivideDepthBuffer=NULL;
    nByte = 1ll * sizeof(int) * SubdivideNum;
    CHECK(hipMalloc((int**)&SubdivideDepthBuffer,nByte));

    int *SubdivideDepthNum = NULL;
    nByte = 1ll * sizeof(int) * (maxDepth_h+1) * SubdivideNum;
    CHECK(hipMalloc((int**)&SubdivideDepthNum,nByte));
    CHECK(hipMemset(SubdivideDepthNum,0,nByte));

    precomputeSubdivideDepth<<<grid,block>>>(SubdivideNode,SubdivideNum,
                                             DepthBuffer,
                                             SubdivideDepthBuffer,
                                             SubdivideDepthNum);
    hipDeviceSynchronize();

    int SubdivideDepthCount[maxDepth_h+1];

    thrust::device_ptr<int> SubdivideDepthNum_ptr = thrust::device_pointer_cast<int>(SubdivideDepthNum);
    for(int i=0;i<=maxDepth_h;++i){
        SubdivideDepthCount[i]=thrust::reduce(SubdivideDepthNum_ptr + i*SubdivideNum,SubdivideDepthNum_ptr + (i+1)*SubdivideNum);
    }
    hipDeviceSynchronize();

    hipFree(SubdivideDepthNum);

    int SubdivideDepthAddress[maxDepth_h+1];
    SubdivideDepthAddress[0]=0;
    for(int i=1;i<=maxDepth_h;++i){
        SubdivideDepthAddress[i]=SubdivideDepthAddress[i-1]+SubdivideDepthCount[i-1];
    }

    int minSubdivideRootDepth;
    CHECK(hipMemcpy(&minSubdivideRootDepth,SubdivideDepthBuffer,sizeof(int),hipMemcpyDeviceToHost));
    int maxNodeNums = (qpow(8,(maxDepth_h-minSubdivideRootDepth+1) )-1 )/7;

    EasyOctNode *SubdivideArray=NULL;
    nByte = 1ll * sizeof(EasyOctNode) * maxNodeNums;
    CHECK(hipMalloc((EasyOctNode**)&SubdivideArray,nByte));

    int *SubdivideArrayDepthBuffer=NULL;
    nByte = 1ll * sizeof(int) * maxNodeNums;
    CHECK(hipMalloc((int**)&SubdivideArrayDepthBuffer,nByte));

    Point3D<float> *SubdivideArrayCenterBuffer=NULL;
    nByte = 1ll * sizeof(Point3D<float>) * maxNodeNums;
    CHECK(hipMalloc((Point3D<float>**)&SubdivideArrayCenterBuffer,nByte));



    int finerDepth = 6;
    for(int i=0;i<SubdivideNum;++i){
//        int rootDepth = SubdivideDepthBuffer[i];
        int rootDepth;
        CHECK(hipMemcpy(&rootDepth,SubdivideDepthBuffer+i,sizeof(int),hipMemcpyDeviceToHost));
        if(rootDepth >= finerDepth){
            break;
        }
        int SubdivideArray_sz = (qpow(8,(maxDepth_h-rootDepth+1) )-1 )/7;
        int fixedDepthNodeNum[maxDepth+1]={0};
        int nowNodeNum=1;
        fixedDepthNodeNum[rootDepth-1]=0;
        for(int j=rootDepth;j<=maxDepth_h;++j){
            fixedDepthNodeNum[j]=nowNodeNum;
            nowNodeNum<<=3;
        }
        int fixedDepthNodeAddress[maxDepth+1]={0};
        fixedDepthNodeAddress[rootDepth-1]=0;
        for(int j=rootDepth;j<=maxDepth_h;++j){
            fixedDepthNodeAddress[j]=fixedDepthNodeAddress[j-1]+fixedDepthNodeNum[j-1];
//            printf("%d %d\n",fixedDepthNodeNum[j-1],fixedDepthNodeAddress[j]);
        }

        OctNode rootNode;
        CHECK(hipMemcpy(&rootNode,&SubdivideNode[i],sizeof(OctNode),hipMemcpyDeviceToHost));
        int rootId;
        rootId = rootNode.neighs[13];
        int rootParent;
        rootParent = rootNode.parent;
        int rootKey = rootNode.key;
        int rootSonKey =( rootKey >> (3 * (maxDepth-rootDepth)) ) & 7 ;

        nByte = 1ll * sizeof(EasyOctNode) * SubdivideArray_sz;
//        CHECK(hipMalloc((EasyOctNode**)&SubdivideArray,nByte));
        CHECK(hipMemset(SubdivideArray,0,nByte));

        CHECK(hipMemcpy(&NodeArray[rootParent].children[rootSonKey],&NodeArray_sz,sizeof(int),hipMemcpyHostToDevice));
        CHECK(hipMemcpy(&SubdivideArray[0].parent,&rootParent,sizeof(int),hipMemcpyHostToDevice));

//        int *SubdivideArrayDepthBuffer=NULL;
//        nByte = 1ll * sizeof(int) * SubdivideArray_sz;
//        CHECK(hipMalloc((int**)&SubdivideArrayDepthBuffer,nByte));

//        Point3D<float> *SubdivideArrayCenterBuffer=NULL;
//        nByte = 1ll * sizeof(Point3D<float>) * SubdivideArray_sz;
//        CHECK(hipMalloc((Point3D<float>**)&SubdivideArrayCenterBuffer,nByte));

        singleRebuildArray<<<grid,block>>>(NodeArray_sz,
                                           SubdivideNode,SubdivideDepthBuffer,i,
                                           SubdivideArray,SubdivideArray_sz,
                                           SubdivideArrayDepthBuffer,SubdivideArrayCenterBuffer);
        hipDeviceSynchronize();


        for(int j=rootDepth;j<=maxDepth_h;++j) {
            computeRebuildNeighbor<<<grid, block>>>(SubdivideArray,fixedDepthNodeAddress[j],
                                             fixedDepthNodeAddress[j] + fixedDepthNodeNum[j],
                                             NodeArray,NodeArray_sz,
                                             j);
            hipDeviceSynchronize();

        }

        // ----------------------------------------------------

        // preVertexArray
        VertexNode *SubdividePreVertexArray = NULL;
        nByte = 1ll * sizeof(VertexNode) * 8 * fixedDepthNodeNum[maxDepth_h];
        CHECK(hipMalloc((VertexNode**)&SubdividePreVertexArray,nByte));
        CHECK(hipMemset(SubdividePreVertexArray,0,nByte));

        initSubdivideVertexOwner<<<grid,block>>>(NodeArray_sz,
                                                 SubdivideArray,fixedDepthNodeAddress[maxDepth_h],SubdivideArray_sz,
                                                 SubdividePreVertexArray,
                                                 SubdivideArrayCenterBuffer);
        hipDeviceSynchronize();


        VertexNode *SubdivideVertexArray = NULL;
//        nByte = 1ll * sizeof(VertexNode) * 8 * fixedDepthNodeNum[maxDepth_h];
        CHECK(hipMalloc((VertexNode**)&SubdivideVertexArray,nByte));
        CHECK(hipMemset(SubdivideVertexArray,0,nByte));
        thrust::device_ptr<VertexNode> SubdividePreVertexArray_ptr = thrust::device_pointer_cast<VertexNode>(SubdividePreVertexArray);
        thrust::device_ptr<VertexNode> SubdivideVertexArray_ptr = thrust::device_pointer_cast<VertexNode>(SubdivideVertexArray);
        thrust::device_ptr<VertexNode> SubdivideVertexArray_end = thrust::copy_if(SubdividePreVertexArray_ptr, SubdividePreVertexArray_ptr + 8 * fixedDepthNodeNum[maxDepth_h],SubdivideVertexArray_ptr,validVertex());
        hipDeviceSynchronize();

        hipFree(SubdividePreVertexArray);

        int SubdivideVertexArray_sz = SubdivideVertexArray_end - SubdivideVertexArray_ptr;

//        for(int j=0;j<10;++j){
//            printf("%d owner:%d\n",j,SubdivideVertexArray[j].ownerNodeIdx);
//        }

        maintainSubdivideVertexNodePointer<<<grid,block>>>(SubdivideVertexArray,SubdivideVertexArray_sz,
                                                           NodeArray_sz,
                                                           SubdivideArray,
                                                           CenterBuffer,
                                                           SubdivideArrayCenterBuffer);
        hipDeviceSynchronize();



        // ----------------------------------------------------

        // preEdgeArray
        EdgeNode *SubdividePreEdgeArray = NULL;
        nByte = 1ll * sizeof(EdgeNode) * 12 * fixedDepthNodeNum[maxDepth_h];
        CHECK(hipMalloc((EdgeNode**)&SubdividePreEdgeArray,nByte));
        CHECK(hipMemset(SubdividePreEdgeArray,0,nByte));

        initSubdivideEdgeArray<<<grid,block>>>(SubdivideArray,fixedDepthNodeAddress[maxDepth_h],SubdivideArray_sz,
                                               NodeArray_sz,
                                               SubdividePreEdgeArray,
                                               SubdivideArrayCenterBuffer);
        hipDeviceSynchronize();

        EdgeNode *SubdivideEdgeArray=NULL;
        CHECK(hipMalloc((EdgeNode**)&SubdivideEdgeArray,nByte));
        CHECK(hipMemset(SubdivideEdgeArray,0,nByte));

        thrust::device_ptr<EdgeNode> SubdividePreEdgeArray_ptr = thrust::device_pointer_cast<EdgeNode>(SubdividePreEdgeArray);
        thrust::device_ptr<EdgeNode> SubdivideEdgeArray_ptr = thrust::device_pointer_cast<EdgeNode>(SubdivideEdgeArray);
        thrust::device_ptr<EdgeNode> SubdivideEdgeArray_end = thrust::copy_if(SubdividePreEdgeArray_ptr,SubdividePreEdgeArray_ptr + 12 * fixedDepthNodeNum[maxDepth_h],SubdivideEdgeArray_ptr,validEdge());
        hipDeviceSynchronize();

        hipFree(SubdividePreEdgeArray);

        int SubdivideEdgeArray_sz = SubdivideEdgeArray_end - SubdivideEdgeArray_ptr;

        maintainSubdivideEdgeNodePointer<<<grid,block>>>(SubdivideEdgeArray,SubdivideEdgeArray_sz,
                                                         NodeArray_sz,
                                                         SubdivideArray,
                                                         CenterBuffer,
                                                         SubdivideArrayCenterBuffer);
        hipDeviceSynchronize();

//        for(int j=0;j<15;++j){
//            printf("%d owner:%d\n",j,SubdivideEdgeArray[j].ownerNodeIdx);
//        }
//
//        for (int j = fixedDepthNodeAddress[maxDepth_h]; j < fixedDepthNodeAddress[maxDepth_h] + 10; ++j) {
//            std::cout << std::bitset<32>(SubdivideArray[j].key) << " parent:" << SubdivideArray[j].parent
//                      << std::endl;
//            for (int k = 0; k < 8; ++k) {
//                std::cout << "children[" << k << "]:" << SubdivideArray[j].children[k] << " ";
//            }
//            std::cout << std::endl;
//            for (int k = 0; k < 27; ++k) {
//                std::cout << "neigh:[" << k << "]:" << SubdivideArray[j].neighs[k] << " ";
//            }
//            std::cout << std::endl;
//            for(int k=0;k<8;++k){
//                std::cout << "vertices["<<k<<"]:"<<SubdivideArray[j].vertices[k]<<" ";
//            }
//            std::cout << std::endl;
//            for(int k=0;k<12;++k){
//                std::cout << "edges["<<k<<"]:"<<SubdivideArray[j].edges[k]<<" ";
//            }
//            std::cout << std::endl;
//        }

        // ----------------------------------------------------

        float *SubdivideVvalue = NULL;
        nByte = 1ll * sizeof(float) * SubdivideVertexArray_sz;
        CHECK(hipMalloc((float**)&SubdivideVvalue,nByte));
        CHECK(hipMemset(SubdivideVvalue,0,nByte));

        computeSubdivideVertexImplicitFunctionValue<<<grid,block>>>(SubdivideVertexArray,SubdivideVertexArray_sz,
                                                                    SubdivideArray,rootId,
                                                                    NodeArray,NodeArray_sz,
                                                                    d_x,EncodedNodeIdxInFunction,
                                                                    baseFunctions_d,
                                                                    SubdivideVvalue,isoValue);
        hipDeviceSynchronize();

        int *SubdivideVexNums=NULL;
        nByte = 1ll * sizeof(int) * SubdivideEdgeArray_sz;
        CHECK(hipMalloc((int**)&SubdivideVexNums,nByte));
        CHECK(hipMemset(SubdivideVexNums,0,nByte));

        generateSubdivideVexNums<<<grid,block>>>(SubdivideEdgeArray,SubdivideEdgeArray_sz,
                                                 NodeArray_sz,
                                                 SubdivideArray,SubdivideVvalue,
                                                 SubdivideVexNums);
        hipDeviceSynchronize();

        int *SubdivideVexAddress=NULL;
        nByte = 1ll * sizeof(int) *SubdivideEdgeArray_sz;
        CHECK(hipMalloc((int**)&SubdivideVexAddress,nByte));
        CHECK(hipMemset(SubdivideVexAddress,0,nByte));

        thrust::device_ptr<int> SubdivideVexNums_ptr = thrust::device_pointer_cast<int>(SubdivideVexNums);
        thrust::device_ptr<int> SubdivideVexAddress_ptr = thrust::device_pointer_cast<int>(SubdivideVexAddress);

        thrust::exclusive_scan(SubdivideVexNums_ptr,SubdivideVexNums_ptr + SubdivideEdgeArray_sz, SubdivideVexAddress_ptr);
        hipDeviceSynchronize();

//        int SubdivideLastVexAddr = SubdivideVexAddress[SubdivideEdgeArray_sz-1];
//        int SubdivideLastVexNums = SubdivideVexNums[SubdivideEdgeArray_sz-1];
        int SubdivideLastVexAddr;
        int SubdivideLastVexNums;
        CHECK(hipMemcpy(&SubdivideLastVexAddr,SubdivideVexAddress+SubdivideEdgeArray_sz-1,sizeof(int),hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(&SubdivideLastVexNums,SubdivideVexNums+SubdivideEdgeArray_sz-1,sizeof(int),hipMemcpyDeviceToHost));

        int SubdivideAllVexNums = SubdivideLastVexAddr + SubdivideLastVexNums;

        if(SubdivideAllVexNums == 0){
            CHECK(hipMemcpy(&NodeArray[rootParent].children[rootSonKey],&rootId,sizeof(int),hipMemcpyHostToDevice));
            hipFree(SubdivideVertexArray);
            hipFree(SubdivideEdgeArray);
            hipFree(SubdivideVvalue);
            hipFree(SubdivideVexNums);
            hipFree(SubdivideVexAddress);
            continue;
        }

        int *SubdivideTriNums=NULL;
        nByte = 1ll * sizeof(int) * fixedDepthNodeNum[maxDepth_h];
        CHECK(hipMalloc((int**)&SubdivideTriNums,nByte));
        CHECK(hipMemset(SubdivideTriNums,0,nByte));

        int *SubdivideCubeCatagory = NULL;
        nByte = 1ll * sizeof(int) * fixedDepthNodeNum[maxDepth_h];
        CHECK(hipMalloc((int**)&SubdivideCubeCatagory,nByte));
        CHECK(hipMemset(SubdivideCubeCatagory,0,nByte));

        generateTriNums<<<grid,block>>>(SubdivideArray,
                                        fixedDepthNodeAddress[maxDepth_h],SubdivideArray_sz,
                                        SubdivideVvalue,
                                        SubdivideTriNums,SubdivideCubeCatagory);
        hipDeviceSynchronize();

        int *SubdivideTriAddress=NULL;
        nByte = 1ll * sizeof(int) * fixedDepthNodeNum[maxDepth_h];
        CHECK(hipMalloc((int**)&SubdivideTriAddress,nByte));
        CHECK(hipMemset(SubdivideTriAddress,0,nByte));

        thrust::device_ptr<int> SubdivideTriNums_ptr = thrust::device_pointer_cast<int>(SubdivideTriNums);
        thrust::device_ptr<int> SubdivideTriAddress_ptr = thrust::device_pointer_cast<int>(SubdivideTriAddress);

        thrust::exclusive_scan(SubdivideTriNums_ptr,SubdivideTriNums_ptr + fixedDepthNodeNum[maxDepth_h],SubdivideTriAddress_ptr);
        hipDeviceSynchronize();

        Point3D<float> *SubdivideVertexBuffer = NULL;
        nByte = 1ll * sizeof(Point3D<float>) * SubdivideAllVexNums;
        CHECK(hipMallocManaged((Point3D<float>**)&SubdivideVertexBuffer,nByte));

        EdgeNode * SubdivideValidEdgeArray = NULL;
        nByte = 1ll * sizeof(EdgeNode) * SubdivideAllVexNums;
        CHECK(hipMalloc((EdgeNode**)&SubdivideValidEdgeArray,nByte));
        thrust::device_ptr<EdgeNode> SubdivideValidEdgeArray_ptr = thrust::device_pointer_cast<EdgeNode>(SubdivideValidEdgeArray);
        SubdivideEdgeArray_ptr = thrust::device_pointer_cast<EdgeNode>(SubdivideEdgeArray);
        thrust::device_ptr<EdgeNode> SubdivideValidEdgeArray_end = thrust::copy_if(SubdivideEdgeArray_ptr,SubdivideEdgeArray_ptr + SubdivideEdgeArray_sz,SubdivideVexNums_ptr,SubdivideValidEdgeArray_ptr,validVexNums());
        hipDeviceSynchronize();

        int *SubdivideValidVexAddress = NULL;
        nByte = 1ll * sizeof(int) * SubdivideAllVexNums;
        CHECK(hipMalloc((int**)&SubdivideValidVexAddress,nByte));
        thrust::device_ptr<int> SubdivideValidVexAddress_ptr = thrust::device_pointer_cast<int>(SubdivideValidVexAddress);
        thrust::device_ptr<int> SubdivideValidVexAddress_end = thrust::copy_if(SubdivideVexAddress_ptr,SubdivideVexAddress_ptr + SubdivideEdgeArray_sz,SubdivideVexNums_ptr,SubdivideValidVexAddress_ptr,validVexNums());
        hipDeviceSynchronize();

//        grid = (std::min(SubdivideAllVexNums / 1024 , 32) ,SubdivideAllVexNums/32768);
        generateSubdivideIntersectionPoint<<<grid,block>>>(SubdivideValidEdgeArray,SubdivideAllVexNums,
                                                           SubdivideVertexArray,SubdivideArray,
                                                           NodeArray_sz,
                                                           SubdivideValidVexAddress,SubdivideVvalue,
                                                           SubdivideVertexBuffer);
        hipDeviceSynchronize();

        hipFree(SubdivideValidEdgeArray);
        hipFree(SubdivideValidVexAddress);


//        int SubdivideLastTriAddr = SubdivideTriAddress[fixedDepthNodeNum[maxDepth_h]-1];
//        int SubdivideLastTriNums = SubdivideTriNums[fixedDepthNodeNum[maxDepth_h]-1];
        int SubdivideLastTriAddr;
        int SubdivideLastTriNums;
        CHECK(hipMemcpy(&SubdivideLastTriAddr,SubdivideTriAddress+fixedDepthNodeNum[maxDepth_h]-1,sizeof(int),hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(&SubdivideLastTriNums,SubdivideTriNums+fixedDepthNodeNum[maxDepth_h]-1,sizeof(int),hipMemcpyDeviceToHost));
        int SubdivideAllTriNums = SubdivideLastTriAddr + SubdivideLastTriNums;


        int *SubdivideTriangleBuffer=NULL;
        nByte = 1ll * sizeof(int) * 3 * SubdivideAllTriNums;
        CHECK(hipMallocManaged((int**)&SubdivideTriangleBuffer,nByte));

        generateSubdivideTrianglePos<<<grid,block>>>(SubdivideArray,fixedDepthNodeAddress[maxDepth_h],SubdivideArray_sz,
                                                     SubdivideTriNums,SubdivideCubeCatagory,
                                                     SubdivideVexAddress,
                                                     SubdivideTriAddress,SubdivideTriangleBuffer);
        hipDeviceSynchronize();


        insertTriangle(SubdivideVertexBuffer,SubdivideAllVexNums,
                       SubdivideTriangleBuffer,SubdivideAllTriNums,
                       mesh);

//        printf("SubdivideAllVexNums:%d SubdivideAllTriNums:%d\n",SubdivideAllVexNums,SubdivideAllTriNums);



        CHECK(hipMemcpy(&NodeArray[rootParent].children[rootSonKey],&rootId,sizeof(int),hipMemcpyHostToDevice));
        hipFree(SubdivideVertexArray);
        hipFree(SubdivideEdgeArray);
        hipFree(SubdivideVvalue);
        hipFree(SubdivideVexNums);
        hipFree(SubdivideVexAddress);
        hipFree(SubdivideTriNums);
        hipFree(SubdivideCubeCatagory);
        hipFree(SubdivideTriAddress);
        hipFree(SubdivideVertexBuffer);
        hipFree(SubdivideTriangleBuffer);
    }

    hipFree(SubdivideArray);
    hipFree(SubdivideArrayCenterBuffer);
    hipFree(SubdivideArrayDepthBuffer);

    double mid14 = cpuSecond();
    printf("GPU processing coarse subdivide nodes takes %lfs\n",mid14-mid13);

    for(int i = finerDepth;i<maxDepth_h;++i) {
        printf("Start the %d depth finer subdivide nodes processing\n",i);
        int *fixedDepthNums = NULL;
        int finerDepthStart = SubdivideDepthAddress[i];
        int finerSubdivideNum = SubdivideDepthCount[i];
        nByte = 1ll * sizeof(int) * finerSubdivideNum * maxDepth_h;
        CHECK(hipMalloc((int **) &fixedDepthNums, nByte));
        CHECK(hipMemset(fixedDepthNums, 0, nByte));

        initFixedDepthNums<<<grid, block>>>(SubdivideNode, finerDepthStart, finerDepthStart + finerSubdivideNum,
                                            SubdivideDepthBuffer,
                                            fixedDepthNums);
        hipDeviceSynchronize();

        thrust::device_ptr<int> fixedDepthNums_ptr = thrust::device_pointer_cast<int>(fixedDepthNums);
        int rebuildNums = thrust::reduce(fixedDepthNums_ptr, fixedDepthNums_ptr + finerSubdivideNum * maxDepth_h);
        hipDeviceSynchronize();

        int depthNodeCount[maxDepth_h + 1];
        depthNodeCount[0] = 0;
        for (int depth = 1; depth <= maxDepth_h; ++depth) {
            depthNodeCount[depth] = thrust::reduce(fixedDepthNums_ptr + (depth - 1) * finerSubdivideNum,
                                                   fixedDepthNums_ptr + depth * finerSubdivideNum);
        }
        hipDeviceSynchronize();

        int depthNodeAddress[maxDepth_h + 1];
        depthNodeAddress[0] = 0;
        for (int depth = 1; depth <= maxDepth_h; ++depth) {
            depthNodeAddress[depth] = depthNodeAddress[depth - 1] + depthNodeCount[depth - 1];
//            printf("%d %d %d\n", depth, depthNodeAddress[depth], depthNodeCount[depth]);
        }
//        printf("rebuildNums:%d\n", rebuildNums);

        int *depthNodeAddress_d = NULL;
        nByte = 1ll * sizeof(int) * (maxDepth_h + 1);
        CHECK(hipMalloc((int **) &depthNodeAddress_d, nByte));
        CHECK(hipMemcpy(depthNodeAddress_d, depthNodeAddress, nByte, hipMemcpyHostToDevice));

        int *fixedDepthAddress = NULL;
        nByte = 1ll * sizeof(int) * finerSubdivideNum * maxDepth_h;
        CHECK(hipMalloc((int **) &fixedDepthAddress, nByte));
        CHECK(hipMemset(fixedDepthAddress, 0, nByte));
        for (int depth = 1; depth <= maxDepth_h; ++depth) {
            thrust::device_ptr<int> fixedDepthAddress_ptr = thrust::device_pointer_cast<int>(
                    fixedDepthAddress + (depth - 1) * finerSubdivideNum);
            thrust::exclusive_scan(fixedDepthNums_ptr + (depth - 1) * finerSubdivideNum,
                                   fixedDepthNums_ptr + depth * finerSubdivideNum, fixedDepthAddress_ptr);
            hipDeviceSynchronize();
        }

        EasyOctNode *RebuildArray = NULL;
        long long nBytell = 1ll * sizeof(EasyOctNode) * rebuildNums;
        CHECK(hipMalloc((EasyOctNode **) &RebuildArray, nBytell));
        CHECK(hipMemset(RebuildArray, 0, nBytell));

        int *RebuildDepthBuffer = NULL;
        nBytell = 1ll * sizeof(int) * rebuildNums;
        CHECK(hipMalloc((int **) &RebuildDepthBuffer, nBytell));

        Point3D<float> *RebuildCenterBuffer = NULL;
        nBytell = 1ll * sizeof(Point3D<float>) * rebuildNums;
        CHECK(hipMalloc((Point3D<float> **) &RebuildCenterBuffer, nBytell));

        int *ReplaceNodeId = NULL;
        nBytell = 1ll * sizeof(int) * rebuildNums;
        CHECK(hipMalloc((int **) &ReplaceNodeId, nBytell));

        int *IsRoot = NULL;
        nBytell = 1ll * sizeof(int) * rebuildNums;
        CHECK(hipMalloc((int **) &IsRoot, nBytell));
        CHECK(hipMemset(IsRoot, 0, nBytell));


        wholeRebuildArray<<<grid, block>>>(SubdivideNode, finerDepthStart, finerDepthStart + finerSubdivideNum,
                                           NodeArray, NodeArray_sz,
                                           SubdivideDepthBuffer,
                                           depthNodeAddress_d, fixedDepthAddress,
                                           RebuildArray,
                                           RebuildDepthBuffer, RebuildCenterBuffer,
                                           ReplaceNodeId, IsRoot);
        hipDeviceSynchronize();


        for (int j = finerDepth; j <= maxDepth_h; ++j) {
            computeRebuildNeighbor<<<grid, block>>>(RebuildArray, depthNodeAddress[j],
                                                    depthNodeAddress[j] + depthNodeCount[j],
                                                    NodeArray, NodeArray_sz,
                                                    j);
            hipDeviceSynchronize();

        }

        // preVertexArray
        VertexNode *RebuildPreVertexArray = NULL;
        nByte = 1ll * sizeof(VertexNode) * 8 * depthNodeCount[maxDepth_h];
        CHECK(hipMalloc((VertexNode **) &RebuildPreVertexArray, nByte));
        CHECK(hipMemset(RebuildPreVertexArray, 0, nByte));

        initSubdivideVertexOwner<<<grid, block>>>(NodeArray_sz,
                                                  RebuildArray, depthNodeAddress[maxDepth_h], rebuildNums,
                                                  RebuildPreVertexArray,
                                                  RebuildCenterBuffer);
        hipDeviceSynchronize();

        VertexNode *RebuildVertexArray = NULL;
        CHECK(hipMalloc((VertexNode **) &RebuildVertexArray, nByte));
        CHECK(hipMemset(RebuildVertexArray, 0, nByte));
        thrust::device_ptr<VertexNode> RebuildPreVertexArray_ptr = thrust::device_pointer_cast<VertexNode>(
                RebuildPreVertexArray);
        thrust::device_ptr<VertexNode> RebuildVertexArray_ptr = thrust::device_pointer_cast<VertexNode>(
                RebuildVertexArray);
        thrust::device_ptr<VertexNode> RebuildVertexArray_end = thrust::copy_if(RebuildPreVertexArray_ptr,
                                                                                RebuildPreVertexArray_ptr +
                                                                                8 * depthNodeCount[maxDepth_h],
                                                                                RebuildVertexArray_ptr, validVertex());
        hipDeviceSynchronize();

        hipFree(RebuildPreVertexArray);

        int RebuildVertexArray_sz = RebuildVertexArray_end - RebuildVertexArray_ptr;

//        printf("RebuildVertexArray_sz:%d\n", RebuildVertexArray_sz);


        maintainSubdivideVertexNodePointer<<<grid, block>>>(RebuildVertexArray, RebuildVertexArray_sz,
                                                            NodeArray_sz,
                                                            RebuildArray,
                                                            CenterBuffer,
                                                            RebuildCenterBuffer);
        hipDeviceSynchronize();

//    for (int t = depthNodeAddress[maxDepth_h]; t < depthNodeAddress[maxDepth_h]+10; ++t) {
//        std::cout << t<<std::endl;
//        std::cout << std::bitset<32>(RebuildArray[t].key) << " parent:" << RebuildArray[t].parent
//                  << std::endl;
//        for (int k = 0; k < 8; ++k) {
//            std::cout << "children[" << k << "]:" << RebuildArray[t].children[k] << " ";
//        }
//        std::cout << std::endl;
//        for (int k = 0; k < 27; ++k) {
//            std::cout << "neigh:[" << k << "]" << RebuildArray[t].neighs[k] << " ";
//        }
//        std::cout << std::endl;
//        for (int k = 0; k < 8; ++k) {
//            std::cout << "vertices:[" << k << "]" << RebuildArray[t].vertices[k] << " ";
//        }
//        std::cout << std::endl;
//    }


        // preEdgeArray
        EdgeNode *RebuildPreEdgeArray = NULL;
        nByte = 1ll * sizeof(EdgeNode) * 12 * depthNodeCount[maxDepth_h];
        CHECK(hipMalloc((EdgeNode **) &RebuildPreEdgeArray, nByte));
        CHECK(hipMemset(RebuildPreEdgeArray, 0, nByte));

        initSubdivideEdgeArray<<<grid, block>>>(RebuildArray, depthNodeAddress[maxDepth_h], rebuildNums,
                                                NodeArray_sz,
                                                RebuildPreEdgeArray,
                                                RebuildCenterBuffer);
        hipDeviceSynchronize();

        EdgeNode *RebuildEdgeArray = NULL;
        CHECK(hipMalloc((EdgeNode **) &RebuildEdgeArray, nByte));
        CHECK(hipMemset(RebuildEdgeArray, 0, nByte));

        thrust::device_ptr<EdgeNode> RebuildPreEdgeArray_ptr = thrust::device_pointer_cast<EdgeNode>(
                RebuildPreEdgeArray);
        thrust::device_ptr<EdgeNode> RebuildEdgeArray_ptr = thrust::device_pointer_cast<EdgeNode>(RebuildEdgeArray);
        thrust::device_ptr<EdgeNode> RebuildEdgeArray_end = thrust::copy_if(RebuildPreEdgeArray_ptr,
                                                                            RebuildPreEdgeArray_ptr +
                                                                            12 * depthNodeCount[maxDepth_h],
                                                                            RebuildEdgeArray_ptr, validEdge());
        hipDeviceSynchronize();

        hipFree(RebuildPreEdgeArray);

        int RebuildEdgeArray_sz = RebuildEdgeArray_end - RebuildEdgeArray_ptr;


        maintainSubdivideEdgeNodePointer<<<grid, block>>>(RebuildEdgeArray, RebuildEdgeArray_sz,
                                                          NodeArray_sz,
                                                          RebuildArray,
                                                          CenterBuffer,
                                                          RebuildCenterBuffer);
        hipDeviceSynchronize();


        float *RebuildVvalue = NULL;
        nByte = 1ll * sizeof(float) * RebuildVertexArray_sz;
        CHECK(hipMalloc((float **) &RebuildVvalue, nByte));
        CHECK(hipMemset(RebuildVvalue, 0, nByte));

        computeSubdivideVertexImplicitFunctionValue<<<grid, block>>>(RebuildVertexArray, RebuildVertexArray_sz,
                                                                     RebuildArray, ReplaceNodeId, IsRoot,
                                                                     NodeArray, NodeArray_sz,
                                                                     d_x, EncodedNodeIdxInFunction,
                                                                     baseFunctions_d,
                                                                     RebuildVvalue, isoValue);
        hipDeviceSynchronize();

        hipFree(ReplaceNodeId);
        hipFree(IsRoot);

        int *RebuildVexNums = NULL;
        nByte = 1ll * sizeof(int) * RebuildEdgeArray_sz;
        CHECK(hipMalloc((int **) &RebuildVexNums, nByte));
        CHECK(hipMemset(RebuildVexNums, 0, nByte));

        generateSubdivideVexNums<<<grid, block>>>(RebuildEdgeArray, RebuildEdgeArray_sz,
                                                  NodeArray_sz,
                                                  RebuildArray, RebuildVvalue,
                                                  RebuildVexNums);
        hipDeviceSynchronize();


        int *RebuildVexAddress = NULL;
        nByte = 1ll * sizeof(int) * RebuildEdgeArray_sz;
        CHECK(hipMalloc((int **) &RebuildVexAddress, nByte));
        CHECK(hipMemset(RebuildVexAddress, 0, nByte));

        thrust::device_ptr<int> RebuildVexNums_ptr = thrust::device_pointer_cast<int>(RebuildVexNums);
        thrust::device_ptr<int> RebuildVexAddress_ptr = thrust::device_pointer_cast<int>(RebuildVexAddress);

        thrust::exclusive_scan(RebuildVexNums_ptr, RebuildVexNums_ptr + RebuildEdgeArray_sz, RebuildVexAddress_ptr);
        hipDeviceSynchronize();

        int RebuildLastVexAddr;
        int RebuildLastVexNums;
        CHECK(hipMemcpy(&RebuildLastVexAddr, RebuildVexAddress + RebuildEdgeArray_sz - 1, sizeof(int),
                         hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(&RebuildLastVexNums, RebuildVexNums + RebuildEdgeArray_sz - 1, sizeof(int),
                         hipMemcpyDeviceToHost));

        int RebuildAllVexNums = RebuildLastVexAddr + RebuildLastVexNums;

//        printf("RebuildAllVexNums:%d\n", RebuildAllVexNums);


        int *RebuildTriNums = NULL;
        nByte = 1ll * sizeof(int) * depthNodeCount[maxDepth_h];
        CHECK(hipMalloc((int **) &RebuildTriNums, nByte));
        CHECK(hipMemset(RebuildTriNums, 0, nByte));

        int *RebuildCubeCatagory = NULL;
        nByte = 1ll * sizeof(int) * depthNodeCount[maxDepth_h];
        CHECK(hipMalloc((int **) &RebuildCubeCatagory, nByte));
        CHECK(hipMemset(RebuildCubeCatagory, 0, nByte));

        generateTriNums<<<grid, block>>>(RebuildArray,
                                         depthNodeAddress[maxDepth_h], rebuildNums,
                                         RebuildVvalue,
                                         RebuildTriNums, RebuildCubeCatagory);
        hipDeviceSynchronize();

        int *RebuildTriAddress = NULL;
        nByte = 1ll * sizeof(int) * depthNodeCount[maxDepth_h];
        CHECK(hipMalloc((int **) &RebuildTriAddress, nByte));
        CHECK(hipMemset(RebuildTriAddress, 0, nByte));

        thrust::device_ptr<int> RebuildTriNums_ptr = thrust::device_pointer_cast<int>(RebuildTriNums);
        thrust::device_ptr<int> RebuildTriAddress_ptr = thrust::device_pointer_cast<int>(RebuildTriAddress);

        thrust::exclusive_scan(RebuildTriNums_ptr, RebuildTriNums_ptr + depthNodeCount[maxDepth_h],
                               RebuildTriAddress_ptr);
        hipDeviceSynchronize();

        Point3D<float> *RebuildVertexBuffer = NULL;
        nByte = 1ll * sizeof(Point3D<float>) * RebuildAllVexNums;
        CHECK(hipMallocManaged((Point3D<float> **) &RebuildVertexBuffer, nByte));


        EdgeNode *RebuildValidEdgeArray = NULL;
        nByte = 1ll * sizeof(EdgeNode) * RebuildAllVexNums;
        CHECK(hipMalloc((EdgeNode **) &RebuildValidEdgeArray, nByte));
        thrust::device_ptr<EdgeNode> RebuildValidEdgeArray_ptr = thrust::device_pointer_cast<EdgeNode>(
                RebuildValidEdgeArray);
        RebuildEdgeArray_ptr = thrust::device_pointer_cast<EdgeNode>(RebuildEdgeArray);
        thrust::device_ptr<EdgeNode> RebuildValidEdgeArray_end = thrust::copy_if(RebuildEdgeArray_ptr,
                                                                                 RebuildEdgeArray_ptr +
                                                                                 RebuildEdgeArray_sz,
                                                                                 RebuildVexNums_ptr,
                                                                                 RebuildValidEdgeArray_ptr,
                                                                                 validVexNums());
        hipDeviceSynchronize();

        int *RebuildValidVexAddress = NULL;
        nByte = 1ll * sizeof(int) * RebuildAllVexNums;
        CHECK(hipMalloc((int **) &RebuildValidVexAddress, nByte));
        thrust::device_ptr<int> RebuildValidVexAddress_ptr = thrust::device_pointer_cast<int>(RebuildValidVexAddress);
        thrust::device_ptr<int> RebuildValidVexAddress_end = thrust::copy_if(RebuildVexAddress_ptr,
                                                                             RebuildVexAddress_ptr +
                                                                             RebuildEdgeArray_sz, RebuildVexNums_ptr,
                                                                             RebuildValidVexAddress_ptr,
                                                                             validVexNums());
        hipDeviceSynchronize();

        generateSubdivideIntersectionPoint<<<grid, block>>>(RebuildValidEdgeArray, RebuildAllVexNums,
                                                            RebuildVertexArray, RebuildArray,
                                                            NodeArray_sz,
                                                            RebuildValidVexAddress, RebuildVvalue,
                                                            RebuildVertexBuffer);
        hipDeviceSynchronize();

        hipFree(RebuildValidEdgeArray);
        hipFree(RebuildValidVexAddress);
        hipFree(RebuildVvalue);


        int RebuildLastTriAddr;
        int RebuildLastTriNums;
//        RebuildLastTriAddr = RebuildTriAddress[depthNodeCount[maxDepth_h] - 1];
//        RebuildLastTriNums = RebuildTriNums[depthNodeCount[maxDepth_h] - 1];
        CHECK(hipMemcpy(&RebuildLastTriAddr,RebuildTriAddress+depthNodeCount[maxDepth_h]-1,sizeof(int),hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(&RebuildLastTriNums,RebuildTriNums+depthNodeCount[maxDepth_h]-1,sizeof(int),hipMemcpyDeviceToHost));
        int RebuildAllTriNums = RebuildLastTriAddr + RebuildLastTriNums;

//        printf("RebuildAllTriNums:%d\n", RebuildAllTriNums);

        hipFree(RebuildVexNums);

        int *RebuildTriangleBuffer = NULL;
        nByte = 1ll * sizeof(int) * 3 * RebuildAllTriNums;
        CHECK(hipMallocManaged((int **) &RebuildTriangleBuffer, nByte));

        generateSubdivideTrianglePos<<<grid, block>>>(RebuildArray, depthNodeAddress[maxDepth_h], rebuildNums,
                                                      RebuildTriNums, RebuildCubeCatagory,
                                                      RebuildVexAddress,
                                                      RebuildTriAddress, RebuildTriangleBuffer);
        hipDeviceSynchronize();

        insertTriangle(RebuildVertexBuffer, RebuildAllVexNums,
                       RebuildTriangleBuffer, RebuildAllTriNums,
                       mesh);

        hipFree(fixedDepthNums);
        hipFree(depthNodeAddress_d);
        hipFree(fixedDepthAddress);
        hipFree(RebuildArray);
        hipFree(RebuildDepthBuffer);
        hipFree(RebuildCenterBuffer);
        hipFree(RebuildVertexArray);
        hipFree(RebuildEdgeArray);
        hipFree(RebuildVexAddress);
        hipFree(RebuildTriNums);
        hipFree(RebuildCubeCatagory);
        hipFree(RebuildTriAddress);
        hipFree(RebuildVertexBuffer);
        hipFree(RebuildTriangleBuffer);
    }

    double mid15 = cpuSecond();
    printf("GPU processing finer subdivide nodes takes %lfs\n",mid15-mid14);

    PlyWriteTriangles(outName,&mesh, PLY_ASCII,center,scale,NULL,0);

    double mid16 = cpuSecond();
    printf("Output ply files takes %lfs\n",mid16-mid15);

    printf("The whole project takes %lfs (including I/O)\n",mid16-project_st);

}