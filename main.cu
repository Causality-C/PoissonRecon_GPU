#include "hip/hip_runtime.h"
#include <cstdio>
#include <bitset>
#include <cstdlib>
#include "Geometry.cuh"
#include "OctNode.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "PointStream.cuh"
#include "CmdLineParser.cuh"
#include "Debug.cuh"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"
#include "thrust/async/reduce.h"
#include "thrust/scan.h"
#include "thrust/sort.h"
#include "thrust/copy.h"


//#define FORCE_UNIT_NORMALS 1

// make readable to device  ?
__constant__ float EPSILON=float(1e-6);
__constant__ float ROUND_EPS=float(1e-5);
__constant__ int maxDepth=10;
__constant__ int markOffset=31;

const int markOffset_h=31;

__device__ long long encodePoint(const Point3D<float>& pos,const long long& idx){
    long long key=0ll;
    Point3D<float> myCenter;
    myCenter.coords[0]=float(0.5);
    myCenter.coords[1]=float(0.5);
    myCenter.coords[2]=float(0.5);

    float myWidth=0.25f;
    for(int i=maxDepth-1;i>=0;--i){
        if(pos.coords[0] > myCenter.coords[0]) {
            key |= 1ll << (3 * i + 34);
            myCenter.coords[0] += myWidth;
        }else{
            myCenter.coords[0] -= myWidth;
        }

        if(pos.coords[1] > myCenter.coords[1]) {
            key |= 1ll << (3 * i + 33);
            myCenter.coords[1] += myWidth;
        }else{
            myCenter.coords[1] -= myWidth;
        }

        if(pos.coords[2] > myCenter.coords[2]) {
            key |= 1ll << (3 * i + 32);
            myCenter.coords[2] += myWidth;
        }else{
            myCenter.coords[2] -= myWidth;
        }
        myWidth/=2;
    }
    return key+idx;
}

__global__ void generateCode(Point3D<float> *points,long long *code,int size){
    long long stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    long long blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    long long offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(long long i=offset;i<size;i+=stride){
        code[i]= encodePoint(points[i],i);
    }
}

__global__ void generateMark(long long *code,int size){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    if(offset==0){
        code[0]|=1ll<<markOffset;
        offset+=stride;
    }
    for(int i=offset;i<size;i+=stride){
        if(code[i]>>32 != code[i-1]>>32) {
            code[i] |= 1ll << markOffset;
        }
    }
}

__global__ void generateNodeNums(long long* uniqueCode,int *nodeNums,int size){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    if(offset==0){
        nodeNums[offset]=8;
        offset+=stride;
    }
    for(int i=offset;i<size;i+=stride){
        if( (uniqueCode[i-1]>>35)  != (uniqueCode[i]>>35) ){
            nodeNums[i]=8;
        }
    }
}

__global__ void initUniqueNode(long long *uniqueCode, OctNode *uniqueNode, int size){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<size;i+=stride){
        uniqueNode[i].key= int(uniqueCode[i] >> 32 ) ;
    }
}


__global__ void generateNodeArray(OctNode *uniqueNode,int *nodeAddress, OctNode *NodeArray,int size){
    int stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    int offset= (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i=offset;i<size;i+=stride){
        NodeArray[nodeAddress[i] + ( (uniqueNode+i)->key & 7) ] = uniqueNode[i];
    }
}


struct markCompact{
    __host__ __device__
    bool operator()(const long long x){
        return ( x & (1ll<<markOffset) ) > 0;
    }
};


int main() {
    char fileName[]="/home/davidxu/horse.npts";

    PointStream<float>* pointStream;
    char* ext = GetFileExtension(fileName);
    if      (!strcasecmp(ext,"bnpts"))      pointStream = new BinaryPointStream<float>(fileName);
    else if (!strcasecmp(ext,"ply"))        pointStream = new PLYPointStream<float>(fileName);
    else                                    pointStream = new ASCIIPointStream<float>(fileName);

    Point3D<float> position,normal;
    Point3D<float> mx,mn;
    Point3D<float> center;
    int count=0;

    float scale=1;
    float scaleFactor=1.25;

    double st=cpuSecond();

    /**     Step 1: compute bounding box     */
    while(pointStream->nextPoint(position,normal)){
        for(int i=0;i<DIMENSION;++i){
            if(!count || position.coords[i]<mn.coords[i]) mn.coords[i]=position.coords[i];
            if(!count || position.coords[i]>mx.coords[i]) mx.coords[i]=position.coords[i];
        }
        ++count;
    }

    for(int i=0;i<DIMENSION;++i){
        if(!i || scale<mx.coords[i]-mn.coords[i]) scale=float(mx.coords[i]-mn.coords[i]);
        center.coords[i]=float(mx.coords[i]+mn.coords[i])/2;
    }
    scale*=scaleFactor;
    for(int i=0;i<DIMENSION;++i)
        center.coords[i]-=scale/2;

    thrust::host_vector<Point3D<float> > p_h(count),n_h(count);

    pointStream->reset();
    int idx=0;
    while(pointStream->nextPoint(position,normal)){
        int i;
        for(i=0;i<DIMENSION;++i)
            position.coords[i]=(position.coords[i]-center.coords[i])/scale;
        for(i=0;i<DIMENSION;++i)
            if(position.coords[i]<0 || position.coords[i]>1)
                break;
        p_h[idx]=position;

#if FORCE_UNIT_NORMALS
        float len=float(Length(normal));
        if(len>EPSILON)
            len=1.0f/len;
        len*=(2<<maxDepth);
        for(i=0;i<DIMENSION;++i)
            normal.coords[i]*=len;
#endif
        n_h[idx]=normal;
        ++idx;
    }

    double mid=cpuSecond();
    printf("Read takes:%lfs\n",mid-st);

    thrust::device_vector<Point3D<float> > p_d=p_h,n_d=n_h;

    Point3D<float> * samplePoints=thrust::raw_pointer_cast(&p_d[0]);
    Point3D<float> * sampleNormals=thrust::raw_pointer_cast(&n_d[0]);

    /**     Step 2: compute shuffled xyz key and sorting code   */
    long long *key=NULL;
    long long nByte=sizeof(long long)*count;
    CHECK(hipMalloc((long long **)&key, nByte));
    dim3 grid=(32,32);
    dim3 block=(32,32);
    generateCode<<<grid,block>>>(samplePoints,key,count);
    hipDeviceSynchronize();

    /**     Step 3: sort all sample points      */
    thrust::device_ptr<long long> key_ptr=thrust::device_pointer_cast<long long>(key);
    thrust::sort_by_key(key_ptr,key_ptr+count,samplePoints);
//    thrust::sort(code_ptr,code_ptr+count,thrust::less<long long>());
    hipDeviceSynchronize();

    /**     Step 4: find the unique nodes       */
    generateMark<<<grid,block>>>(key,count);
    hipDeviceSynchronize();
    thrust::device_vector<long long> uniqueCode(count,-1);
    thrust::copy_if(key_ptr,key_ptr+count,uniqueCode.begin(),markCompact());
    hipDeviceSynchronize();
    int uniqueCount_h=0;
    thrust::host_vector<long long> uniqueCode_h=uniqueCode;
    for(thrust::host_vector<long long>::iterator iter=uniqueCode_h.begin(); iter!=uniqueCode_h.end(); ++iter){
        if(*iter==-1)
            break;
        ++uniqueCount_h;
    }
    uniqueCode.resize(uniqueCount_h);

    /**     Create uniqueN ode according to uniqueCode  */
    OctNode *uniqueNode=NULL;
    nByte=sizeof(OctNode)*uniqueCount_h;
    CHECK(hipMalloc((OctNode **)&uniqueNode,nByte));
    long long *uniqueCode_ptr=thrust::raw_pointer_cast(&uniqueCode[0]);
    initUniqueNode<<<grid,block>>>(uniqueCode_ptr,uniqueNode,uniqueCount_h);
    hipDeviceSynchronize();


    /**     Step 5: augment uniqueNode      */
    int *nodeNums=NULL;
    int *nodeAddress=NULL;
    nByte=sizeof(int)*uniqueCount_h;
    CHECK(hipMalloc((int **)&nodeNums,nByte));
    CHECK(hipMemset(nodeNums,0,nByte));

    CHECK(hipMalloc((int **)&nodeAddress,nByte));
    CHECK(hipMemset(nodeAddress,0,nByte));

    generateNodeNums<<<grid,block>>>(uniqueCode_ptr,nodeNums,uniqueCount_h);
    hipDeviceSynchronize();

    thrust::device_ptr<int> nodeNums_ptr=thrust::device_pointer_cast<int>(nodeNums);
    thrust::device_ptr<int> nodeAddress_ptr=thrust::device_pointer_cast<int>(nodeAddress);

    thrust::exclusive_scan(nodeNums_ptr,nodeNums_ptr+uniqueCount_h,nodeAddress_ptr);
    hipDeviceSynchronize();


    /**     Step 6: create NodeArrayD       */
    int lastAddr,lastNum;
    CHECK(hipMemcpy(&lastAddr,nodeAddress+uniqueCount_h-1,sizeof(int),hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(&lastNum,nodeNums+uniqueCount_h-1,sizeof(int),hipMemcpyDeviceToHost));
    printf("%d %d\n",lastAddr,lastNum);

    int allNodeNums=lastAddr+lastNum;
    OctNode *NodeArray=NULL;
    nByte=sizeof(OctNode) * allNodeNums;
    CHECK(hipMalloc((OctNode **)&NodeArray, nByte));
    CHECK(hipMemset(NodeArray,0,nByte));
    generateNodeArray<<<grid,block>>>(uniqueNode,nodeAddress,NodeArray,uniqueCount_h);

    double ed=cpuSecond();
    printf("Numbers of points:%d\nNumbers of uniqueCode:%d\n",count,uniqueCount_h);
    printf("GPU:%lfs\n",ed-mid);

    hipFree(key);
    hipFree(uniqueNode);
    hipFree(nodeNums);
    hipFree(nodeAddress);
    hipFree(NodeArray);
}
